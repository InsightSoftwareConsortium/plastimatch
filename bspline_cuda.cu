#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
#include "plm_config.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#if defined (_WIN32)
#include <windows.h>
#endif

#include "bspline_opts.h"
#include "bspline.h"
#include "bspline_cuda.h"
#include "bspline_cuda_kernels.h"
#include "cuda_util.h"
#include "mha_io.h"
#include "volume.h"

// Define file-scope textures
texture<float, 1, hipReadModeElementType> tex_moving_image;
texture<float, 1, hipReadModeElementType> tex_coeff;
texture<float, 1, hipReadModeElementType> tex_LUT_Bspline_x;
texture<float, 1, hipReadModeElementType> tex_LUT_Bspline_y;
texture<float, 1, hipReadModeElementType> tex_LUT_Bspline_z;


#define GRID_LIMIT_X 65535
#define GRID_LIMIT_Y 65535


////////////////////////////////////////////////////////////

// Uncomment to include profiling code for MSE CUDA flavor J
//#define PROFILE_J

typedef struct gpu_bspline_data GPU_Bspline_Data;
struct gpu_bspline_data
{
    // bxf items
    int3 rdims;         
    int3 cdims;
    float3 img_origin;      
    float3 img_spacing;
    int3 roi_dim;           
    int3 roi_offset;        
    int3 vox_per_rgn;       

    // fixed volume items
    int3 fix_dim;

    // moving volume items
    int3 mov_dim;       
    float3 mov_offset;
    float3 mov_spacing;
};



// These routine provides a method of more cleanly
// allocating and populating GPU global memory.
// Additionally, zero copy is handled eloquently
// and seamlessly.
//
// James Shacleford
// October 27th, 2010
void
gpu_alloc_copy (
    void** gpu_addr,
    void** cpu_addr,
    size_t mem_size,
    gpu_alloc_copy_mode mode
)
{
    // If zero copying, this will hold the CPU memory address of
    // the new pinned memory address in the CPU memory map.
    // After CPU memory contents is relocated to this new pinned
    // memory, this pointer will overwrite the original CPU
    // pointer (*cpu_addr).
    void* pinned_host_mem;

    if (mode == cudaZeroCopy) {
        // Allocate some pinned CPU memory for zero paging
        hipHostAlloc ((void **)&pinned_host_mem, mem_size, hipHostMallocMapped);
        cuda_utils_check_error ("Failed to allocate pinned memory.");

        // Relocate data to pinned memory
        memcpy (pinned_host_mem, *cpu_addr, mem_size);
        free (*cpu_addr);
        *cpu_addr = pinned_host_mem;

        // Get the address of the pinned page in the GPU memory map.
        hipHostGetDevicePointer ((void **)gpu_addr, (void *)pinned_host_mem, 0);
        cuda_utils_check_error ("Failed to map CPU memory to GPU.");
    } else {
        // Allcoated some global memory on the GPU
        hipMalloc ((void**)gpu_addr, mem_size);
        cuda_utils_check_error ("Out of GPU memory.");

        // Populate the allocated global GPU memory
        hipMemcpy (*gpu_addr, *cpu_addr, mem_size, hipMemcpyHostToDevice);
        cuda_utils_check_error ("Failed to copy data to GPU");
    }
}

// If you plan on using gpu_alloc_vmem() to extend
// the GPU memory, then you must first call this.
void
gpu_init_vmem (dev_pointers_bspline* dev_ptrs)
{
    dev_ptrs->vmem_list = NULL;
}

// This function should only be used to supplement the GPU's
// available "Global Memory" with pinned CPU memory.  Currently,
// the GPU address bus is 32-bit, so using this function we are
// only able to supplement the GPU global memory *up to* 4GB.
// Cards already equiped with 4GB of global memory have a full
// memory map and can therefore be extended no further!
void
gpu_alloc_vmem (
    void** gpu_addr,
    size_t mem_size,
    Dev_Pointers_Bspline* dev_ptrs
)
{
    void* pinned_host_mem;
    Vmem_Entry* new_entry;

    // Allocate some pinned CPU memory for zero paging
    hipHostAlloc ((void **)&pinned_host_mem, mem_size, hipHostMallocMapped);
    cuda_utils_check_error ("Failed to allocate pinned memory.");

    // Clear out new pinned CPU memory
    memset (pinned_host_mem, 0, mem_size);

    // Get the address of the pinned page in the GPU memory map.
    hipHostGetDevicePointer ((void **)gpu_addr, (void *)pinned_host_mem, 0);
    cuda_utils_check_error ("Failed to map CPU memory to GPU.");

    // Now we will register this allocation with my gpu "virtual memory"
    // system.  CUDA requires that we free pinned CPU memory with the CPU
    // pointer; NOT the GPU pointer.  This can be troublesome if you are only
    // tracking GPU pointers and have no need to access the CPU side memory
    // with the CPU.  So, every time we pin CPU memory, we register the pair of
    // pointers (CPU & GPU) in a linked list.  This allows us to only track
    // track one and look up the other.  It also allows us to free all pinned
    // memory without knowing the pointers by simply cycling through the linked
    // list and freeing everything.

    // create a new vmem entry
    new_entry = (Vmem_Entry*) malloc (sizeof(Vmem_Entry));

    // initialize the new entry
    new_entry->gpu_pointer = *gpu_addr;
    new_entry->cpu_pointer = pinned_host_mem;
    new_entry->size = mem_size;

    // insert new entry @ the head
    new_entry->next = dev_ptrs->vmem_list;
    dev_ptrs->vmem_list = new_entry;
}

// Returns the total amount of "virtual global"
// (i.e. pinned CPU) memory. Perhaps useful.
size_t
gpu_tally_vmem (Dev_Pointers_Bspline* dev_ptrs)
{
    size_t total_vmem = 0;
    Vmem_Entry* curr = dev_ptrs->vmem_list;

    while (curr != NULL)
    {
        total_vmem += curr->size;
        curr = curr->next;
    }

    return total_vmem;
}

// For debugging.  Just prints out the virtual
// memory pointer association list.
void
gpu_print_vmem (Dev_Pointers_Bspline* dev_ptrs)
{
    int i = 0;
    Vmem_Entry* curr = dev_ptrs->vmem_list;

    while (curr != NULL)
    {
        printf ("Entry #%i:\n", i);
        printf ("  gpu_pointer: %p\n", curr->gpu_pointer);
        printf ("  cpu_pointer: %p\n\n", curr->cpu_pointer);

        curr = curr->next;
        i++;
    }
}

// Free GPU "virtual memory" via GPU mapped address.
int
gpu_free_vmem (
    void* gpu_pointer,
    Dev_Pointers_Bspline* dev_ptrs
)
{
    Vmem_Entry* curr = dev_ptrs->vmem_list;
    Vmem_Entry* prev = NULL;

    while (curr != NULL)
    {
        if (curr->gpu_pointer == gpu_pointer) {
            hipHostFree (curr->cpu_pointer);
            cuda_utils_check_error ("Failed to free virtual GPU memory.");

            if (prev == NULL) {
                // we are removing the head
                dev_ptrs->vmem_list = curr->next;
                free (curr);
                return 0;
            } else {
                // removing past the head
                prev->next = curr->next;
                free (curr);
                return 0;
            }
        }
        prev = curr;
        curr = curr->next;
    }

    // Failed to free virtual GPU memory.
    return 1;
}

// Frees *ALL* GPU "virtual memory"
// Returns number of freed entries
int
gpu_freeall_vmem (
    Dev_Pointers_Bspline* dev_ptrs
)
{
    int i = 0;
    Vmem_Entry* curr = dev_ptrs->vmem_list;

    while (curr != NULL)
    {
        hipHostFree (curr->cpu_pointer);
        cuda_utils_check_error ("Failed to free virtual GPU memory.");

        dev_ptrs->vmem_list = curr->next;
        free (curr);

        curr = dev_ptrs->vmem_list;
        i++;
    }

    return i;
}

int
gpu_alloc_zero (
    void** gpu_addr,
    size_t mem_size,
    gpu_alloc_fail_mode fail_mode
)
{
    // Allcoated some global memory on the GPU
    hipMalloc ((void**)gpu_addr, mem_size);
    if (fail_mode == cudaAllocStern) {
        cuda_utils_check_error ("Out of GPU memory.");
    } else {
        if (cuda_utils_return_error ("Out of GPU memory.")) {
            return 1;
        }
    }

    // Zero out the allocated global GPU memory
    hipMemset (*gpu_addr, 0, mem_size);
    if (fail_mode == cudaAllocStern) {
        cuda_utils_check_error ("Failed to zero out GPU memory.");
    } else {
        if (cuda_utils_return_error ("Failed to zero out GPU memory.")) {
            return 1;
        }
    }

    // Success
    return 0;
}

int
gpu_zero_copy_check (Bspline_parms* parms)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, parms->gpuid);
    if (props.canMapHostMemory) {
        // GPU supports zero copy
        return 1;
    } else {
        // GPU doest not support zero copy
        return 0;
    }
}



void CUDA_listgpu ()
{
    int num_gpus, i;
    int cores_per_sm;
    hipDeviceProp_t props;

    hipGetDeviceCount(&num_gpus);

    for (i = 0; i < num_gpus; i++) {
        hipGetDeviceProperties(&props, i);
        if (props.major == 1) {
            cores_per_sm = 8;
        } else if (props.major == 2) {
            cores_per_sm = 32;
        } else {
            printf ("GPU Compute Capability: Unknown to Platimatch!\n");
            return;
        }

        printf ("GPU ID %i:\n", i);
        printf ("              Name: %s (%.2f GB)\n", props.name, props.totalGlobalMem / (float)(1024 * 1024 * 1024));
        printf ("Compute Capability: %d.%d\n", props.major, props.minor);
        printf ("     Shared Memory: %.1f MB\n", props.sharedMemPerBlock / (float)1024);
        printf ("         Registers: %i\n", props.regsPerBlock);
        printf ("        Clock Rate: %.2f MHz\n", props.clockRate / (float)(1024));
        printf ("           # Cores: %d\n", props.multiProcessorCount * cores_per_sm);
        printf ("\n");
    }
}

// Selects the best GPU or the user specified
// GPU as defiend on command line
void CUDA_selectgpu (int gpuid)
{
    int num_gpus;
    int cores_per_sm;
    hipDeviceProp_t props;

    hipGetDeviceCount(&num_gpus);

    if (gpuid < num_gpus) {
        hipGetDeviceProperties(&props, gpuid);
        if (props.major == 1) {
            cores_per_sm = 8;
        } else if (props.major == 2) {
            cores_per_sm = 32;
        } else {
            printf ("Compute Capability: Unknown to Platimatch!\n");
            return;
        }

        printf ("Using %s (%.2f GB)\n", props.name, props.totalGlobalMem / (float)(1024 * 1024 * 1024));
        printf ("  - Compute Capability: %d.%d\n", props.major, props.minor);
        printf ("  - # Multi-Processors: %d\n", props.multiProcessorCount);
        printf ("  -    Number of Cores: %d\n", props.multiProcessorCount * cores_per_sm);
        hipSetDevice (gpuid);
    } else {
        printf ("\nInvalid GPU ID specified.  Choices are:\n\n");
        CUDA_listgpu ();
        exit (0);
    }
}

// Returns the value held in __CUDA_ARCH__
// __CUDA_ARCH__ is only accessable in GPU code
// This allows us to use the compute capability
// in CPU code.
int CUDA_getarch (int gpuid)
{
    int num_gpus;
    hipDeviceProp_t props;

    hipGetDeviceCount(&num_gpus);

    if (gpuid < num_gpus) {
        hipGetDeviceProperties(&props, gpuid);

        return 100*props.major + 10*props.minor;

    } else {
        /* Invalid GPU ID specified */
        return -1;
    }
}


// Constructs the GPU Bspline Data structure
void
build_gbd (
    GPU_Bspline_Data* gbd,
    Bspline_xform* bxf,
    Volume* fixed,
    Volume* moving)
{
    if (bxf != NULL) {
        // populate bxf entries
        memcpy (&gbd->rdims, bxf->rdims, 3*sizeof(int));
        memcpy (&gbd->cdims, bxf->cdims, 3*sizeof(int));
        memcpy (&gbd->img_origin, bxf->img_origin, 3*sizeof(float));
        memcpy (&gbd->img_spacing, bxf->img_spacing, 3*sizeof(float));
        memcpy (&gbd->roi_dim, bxf->roi_dim, 3*sizeof(int));
        memcpy (&gbd->roi_offset, bxf->roi_offset, 3*sizeof(int));
        memcpy (&gbd->vox_per_rgn, bxf->vox_per_rgn, 3*sizeof(int));
    }

    if (fixed != NULL) {
        // populate fixed volume entries
        memcpy (&gbd->fix_dim, fixed->dim, 3*sizeof(int));
    }

    if (moving != NULL) {
        // populate moving volume entries
        memcpy (&gbd->mov_dim, moving->dim, 3*sizeof(int));
        memcpy (&gbd->mov_offset, moving->offset, 3*sizeof(float));
        memcpy (&gbd->mov_spacing, moving->pix_spacing, 3*sizeof(float));
    }
    
}


// Builds execution configurations for kernels that
// assign one thread per element (1tpe).
int
build_exec_conf_1tpe (
    dim3 *dimGrid,          // OUTPUT: Grid  dimensions
    dim3 *dimBlock,         // OUTPUT: Block dimensions
    int num_threads,        // INPUT: Total # of threads
    int threads_per_block,  // INPUT: Threads per block
    bool negotiate          // INPUT: Is threads per block negotiable?
)
{
    int i;
    int Grid_x = 0;
    int Grid_y = 0;
    int sqrt_num_blocks;
    int num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;

    if (negotiate) {
        int found_flag = 0;
        int j = 0;

        // Search for a valid execution configuration for the required # of blocks.
        // Block size has been specified as changable.  This helps if the
        // number of blocks required is a prime number > 65535.  Changing the
        // # of threads per block will change the # of blocks... which hopefully
        // won't be prime again.
        for (j = threads_per_block; j > 32; j -= 32) {
            num_blocks = (num_threads + j - 1) / j;
            sqrt_num_blocks = (int)sqrt((float)num_blocks);

            for (i = sqrt_num_blocks; i < GRID_LIMIT_X; i++) {
                if (num_blocks % i == 0) {
                    Grid_x = i;
                    Grid_y = num_blocks / Grid_x;
                    found_flag = 1;
                    break;
                }
            }

            if (found_flag == 1) {
                threads_per_block = j;
                break;
            }
        }

    } else {

        // Search for a valid execution configuration for the required # of blocks.
        // The calling algorithm has specifed that # of threads per block
        // is non negotiable.
        sqrt_num_blocks = (int)sqrt((float)num_blocks);

        for (i = sqrt_num_blocks; i < GRID_LIMIT_X; i++) {
            if (num_blocks % i == 0) {
                Grid_x = i;
                Grid_y = num_blocks / Grid_x;
                break;
            }
        }
    }



    // Were we able to find a valid exec config?
    if (Grid_x == 0) {
        printf ("\n");
        printf ("[GPU KERNEL PANIC] Unable to find suitable execution configuration!");
        printf ("Terminating...\n");
        exit (0);
    } else {
        // callback function could be added
        // to arguments and called here if you need
        // to do something fancy upon success.
#if VERBOSE
        printf ("Grid [%i,%i], %d threads_per_block.\n", 
            Grid_x, Grid_y, threads_per_block);
#endif
    }

    // Pass configuration back by reference
    dimGrid->x = Grid_x;
    dimGrid->y = Grid_y;
    dimGrid->z = 1;

    dimBlock->x = threads_per_block;
    dimBlock->y = 1;
    dimBlock->z = 1;

    // Return the # of blocks we decided on just
    // in case we need it later to allocate shared memory, etc.
    return num_blocks;
}

// Builds execution configurations for kernels that
// assign one block per element (1bpe).
void
build_exec_conf_1bpe (
    dim3 *dimGrid,          // OUTPUT: Grid  dimensions
    dim3 *dimBlock,         // OUTPUT: Block dimensions
    int num_blocks,         // INPUT: Number of blocks
    int threads_per_block)  // INPUT: Threads per block
{
    int i;
    int Grid_x = 0;
    int Grid_y = 0;

    // Search for a valid execution configuration for the required # of blocks.
    int sqrt_num_blocks = (int)sqrt((float)num_blocks);

    for (i = sqrt_num_blocks; i < 65535; i++) {
        if (num_blocks % i == 0) {
            Grid_x = i;
            Grid_y = num_blocks / Grid_x;
            break;
        }
    }


    // Were we able to find a valid exec config?
    if (Grid_x == 0) {
        printf ("\n");
        printf ("[GPU KERNEL PANIC] Unable to find suitable execution configuration!");
        printf ("Terminating...\n");
        exit (0);
    } else {
        // callback function could be added
        // to arguments and called here if you need
        // to do something fancy upon success.
#if VERBOSE
        printf ("Grid [%i,%i], %d threads_per_block.\n", 
            Grid_x, Grid_y, threads_per_block);
#endif
    }

    // Pass configuration back by reference
    dimGrid->x = Grid_x;
    dimGrid->y = Grid_y;
    dimGrid->z = 1;

    dimBlock->x = threads_per_block;
    dimBlock->y = 1;
    dimBlock->z = 1;

}
    



/**
 * A simple kernel used to ensure that CUDA is working correctly.
 *
 * @param dx Stores thread index of every executed thread.
 * @param dy Stores thread index of every executed thread.
 * @param dz Stores thread index of every executed thread.
 */
__global__ void
test_kernel
(
    int3 volume_dim,
    float *dx,
    float *dy,
    float *dz)
{
    // Calculate the index of the thread block in the grid.
    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
    
    // Calculate the total number of threads in each thread block.
    int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

    // Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
    int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

    // Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
    int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

    if (threadIdxInGrid < (volume_dim.x * volume_dim.y * volume_dim.z)) {
        dx[threadIdxInGrid] = (float)threadIdxInGrid;
        dy[threadIdxInGrid] = (float)threadIdxInGrid;
        dz[threadIdxInGrid] = (float)threadIdxInGrid;
    }
}

extern "C" void
bspline_cuda_init_MI_a (
    Dev_Pointers_Bspline* dev_ptrs,
    Volume* fixed,
    Volume* moving,
    Volume* moving_grad,
    Bspline_xform* bxf,
    Bspline_parms* parms)
{
    int out_of_gmem;
    BSPLINE_MI_Hist* mi_hist = &parms->mi_hist;

    // Keep track of how much memory we allocated
    // in the GPU global memory.
    long unsigned GPU_Memory_Bytes = 0;
    long unsigned CPU_Pinned_Bytes = 0;

    printf ("Allocating GPU Memory\n");

    // Get GPU properties (can we zero-copy?)
    parms->gpu_zcpy = gpu_zero_copy_check (parms) && parms->gpu_zcpy;
    if (parms->gpu_zcpy) {
        // Enable GPU to use pinned CPU memory
        hipSetDeviceFlags (hipDeviceMapHost);
        gpu_init_vmem (dev_ptrs);
        printf ("GPU is using zero copy\n");
    }

    // Fixed Image (zero copy if possible)
    // ----------------------------------------------------------
    dev_ptrs->fixed_image_size = fixed->npix * fixed->pix_size;
    if (parms->gpu_zcpy) {
        gpu_alloc_copy ((void **)&dev_ptrs->fixed_image,
                        (void **)&fixed->img,
                        dev_ptrs->fixed_image_size,
                        cudaZeroCopy);
        CPU_Pinned_Bytes += dev_ptrs->fixed_image_size;
        printf ("o");
    } else {
        gpu_alloc_copy ((void **)&dev_ptrs->fixed_image,
                        (void **)&fixed->img,
                        dev_ptrs->fixed_image_size,
                        cudaGlobalMem);
        GPU_Memory_Bytes += dev_ptrs->fixed_image_size;
        printf(".");
    }
    // ----------------------------------------------------------


    // Moving Image (zero copy if possible)
    // ----------------------------------------------------------
    dev_ptrs->moving_image_size = moving->npix * moving->pix_size;
    if (parms->gpu_zcpy) {
        gpu_alloc_copy ((void **)&dev_ptrs->moving_image,
                        (void **)&moving->img,
                        dev_ptrs->moving_image_size,
                        cudaZeroCopy);
        CPU_Pinned_Bytes += dev_ptrs->moving_image_size;
        printf ("o");
    } else {
        gpu_alloc_copy ((void **)&dev_ptrs->moving_image,
                        (void **)&moving->img,
                        dev_ptrs->moving_image_size,
                        cudaGlobalMem);

        GPU_Memory_Bytes += dev_ptrs->moving_image_size;
        printf(".");
    }
    // ----------------------------------------------------------


    // Skipped Voxels
    // ----------------------------------------------------------
    dev_ptrs->skipped_size = sizeof(unsigned int);
    gpu_alloc_zero ((void**)&dev_ptrs->skipped_atomic,
                    dev_ptrs->skipped_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->skipped_size;
    printf(".");
    // ----------------------------------------------------------


    // Histograms
    // ----------------------------------------------------------
    dev_ptrs->f_hist_size = mi_hist->fixed.bins * sizeof(float);
    dev_ptrs->m_hist_size = mi_hist->moving.bins * sizeof(float);
    dev_ptrs->j_hist_size = mi_hist->fixed.bins * mi_hist->moving.bins * sizeof(float);
    hipMalloc ((void**)&dev_ptrs->f_hist, dev_ptrs->f_hist_size);
    hipMalloc ((void**)&dev_ptrs->m_hist, dev_ptrs->m_hist_size);
    hipMalloc ((void**)&dev_ptrs->j_hist, dev_ptrs->j_hist_size);

    GPU_Memory_Bytes += dev_ptrs->f_hist_size;
    GPU_Memory_Bytes += dev_ptrs->m_hist_size;
    GPU_Memory_Bytes += dev_ptrs->j_hist_size;
    printf("...");
    // ----------------------------------------------------------


    // Coefficient LUT
    // ----------------------------------------------------------
    dev_ptrs->coeff_size = sizeof(float) * bxf->num_coeff;
    gpu_alloc_zero ((void **)&dev_ptrs->coeff,
                    dev_ptrs->coeff_size,
                    cudaAllocStern);

    hipBindTexture(0, tex_coeff,
                    dev_ptrs->coeff,
                    dev_ptrs->coeff_size);

    cuda_utils_check_error("Failed to bind dev_ptrs->coeff to texture reference!");
    GPU_Memory_Bytes += dev_ptrs->coeff_size;
    printf(".");
    // ----------------------------------------------------------


    // Score
    // ----------------------------------------------------------
    dev_ptrs->score_size = sizeof(float) * fixed->npix;
    gpu_alloc_zero ((void **)&dev_ptrs->score,
                    dev_ptrs->score_size,
                    cudaAllocStern);

    GPU_Memory_Bytes += dev_ptrs->score_size;
    printf(".");
    // ----------------------------------------------------------
    

    // Gradient (dC_cP)
    // ----------------------------------------------------------
    dev_ptrs->grad_size = sizeof(float) * bxf->num_coeff;
    gpu_alloc_zero ((void **)&dev_ptrs->grad,
                    dev_ptrs->grad_size,
                    cudaAllocStern);

    GPU_Memory_Bytes += dev_ptrs->grad_size;
    printf(".");
    // ----------------------------------------------------------



    // dc_dv_x,  dc_dv_y,  and  dc_dv_z
    //   Try to keep these in GPU memory for speed.  If there
    //   is not enough GPU memory, we will move them to pinned
    //   CPU memory as a last resort.
    //
    // NOTE: If spill over to CPU pinned memory occurs and the
    //       user does not have a significantly large amount
    //       of RAM, the linux kernel may decide that the bspline
    //       process needs to die.  In this event the user will
    //       just see "Killed" in the terminal.  There is no
    //       way to trap this signal, so that is as user friendly
    //       as it can get.
    // ----------------------------------------------------------
    int3 vol_dim;
    vol_dim.x = fixed->dim[0];
    vol_dim.y = fixed->dim[1];
    vol_dim.z = fixed->dim[2];

    int3 tile_dim;
    tile_dim.x = bxf->vox_per_rgn[0];
    tile_dim.y = bxf->vox_per_rgn[1];
    tile_dim.z = bxf->vox_per_rgn[2];

    int4 num_tile;
    num_tile.x = (vol_dim.x+tile_dim.x-1) / tile_dim.x;
    num_tile.y = (vol_dim.y+tile_dim.y-1) / tile_dim.y;
    num_tile.z = (vol_dim.z+tile_dim.z-1) / tile_dim.z;
    num_tile.w = num_tile.x * num_tile.y * num_tile.z;

    int tile_padding = 64 - ((tile_dim.x * tile_dim.y * tile_dim.z) % 64);
    int tile_bytes = (tile_dim.x * tile_dim.y * tile_dim.z);

    dev_ptrs->dc_dv_x_size = ((tile_bytes + tile_padding) * num_tile.w) * sizeof(float);
    dev_ptrs->dc_dv_y_size = dev_ptrs->dc_dv_x_size;
    dev_ptrs->dc_dv_z_size = dev_ptrs->dc_dv_x_size;

    out_of_gmem = 
    gpu_alloc_zero ((void **)&dev_ptrs->dc_dv_x,
                    dev_ptrs->dc_dv_x_size,
                    cudaAllocCasual);

    if (out_of_gmem) {
        if (parms->gpu_zcpy) {
            gpu_alloc_vmem ((void **)&dev_ptrs->dc_dv_x,
                            dev_ptrs->dc_dv_x_size,
                            dev_ptrs);
            CPU_Pinned_Bytes += dev_ptrs->dc_dv_x_size;
            printf ("o");
        } else {
            printf ("Out of memory!\n");
            exit (0);
        }
    } else {
        GPU_Memory_Bytes += dev_ptrs->dc_dv_x_size;
        printf(".");
    }

    out_of_gmem = 
    gpu_alloc_zero ((void **)&dev_ptrs->dc_dv_y,
                    dev_ptrs->dc_dv_y_size,
                    cudaAllocCasual);

    if (out_of_gmem) {
        if (parms->gpu_zcpy) {
            gpu_alloc_vmem ((void **)&dev_ptrs->dc_dv_y,
                            dev_ptrs->dc_dv_y_size,
                            dev_ptrs);
            CPU_Pinned_Bytes += dev_ptrs->dc_dv_y_size;
            printf ("o");
        } else {
            printf ("Out of memory!\n");
            exit (0);
        }
    } else {
        GPU_Memory_Bytes += dev_ptrs->dc_dv_y_size;
        printf(".");
    }

    out_of_gmem = 
    gpu_alloc_zero ((void **)&dev_ptrs->dc_dv_z,
                    dev_ptrs->dc_dv_z_size,
                    cudaAllocCasual);

    if (out_of_gmem) {
        if (parms->gpu_zcpy) {
            gpu_alloc_vmem ((void **)&dev_ptrs->dc_dv_z,
                            dev_ptrs->dc_dv_z_size,
                            dev_ptrs);
            CPU_Pinned_Bytes += dev_ptrs->dc_dv_z_size;
            printf ("o");
        } else {
            printf ("Out of memory!\n");
            exit (0);
        }
    } else {
        GPU_Memory_Bytes += dev_ptrs->dc_dv_z_size;
        printf(".");
    }
    // ----------------------------------------------------------


    // Condensed dc_dv vectors
    // ----------------------------------------------------------
    dev_ptrs->cond_x_size = 64*bxf->num_knots*sizeof(float);
    gpu_alloc_zero ((void **)&dev_ptrs->cond_x,
                    dev_ptrs->cond_x_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->cond_x_size;
    printf(".");

    dev_ptrs->cond_y_size = 64*bxf->num_knots*sizeof(float);
    gpu_alloc_zero ((void **)&dev_ptrs->cond_y,
                    dev_ptrs->cond_y_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->cond_y_size;
    printf(".");

    dev_ptrs->cond_z_size = 64*bxf->num_knots*sizeof(float);
    gpu_alloc_zero ((void **)&dev_ptrs->cond_z,
                    dev_ptrs->cond_z_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->cond_z_size;
    printf(".");
    // ----------------------------------------------------------


    // Tile Offset LUT
    // ----------------------------------------------------------
    int* offsets = calc_offsets(bxf->vox_per_rgn, bxf->cdims);
    int num_tiles = (bxf->cdims[0]-3) * (bxf->cdims[1]-3) * (bxf->cdims[2]-3);

    dev_ptrs->LUT_Offsets_size = num_tiles*sizeof(int);

    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Offsets,
                    (void **)&offsets,
                    dev_ptrs->LUT_Offsets_size,
                    cudaGlobalMem);

    GPU_Memory_Bytes += dev_ptrs->LUT_Offsets_size;
    printf(".");

    free (offsets);
    // ----------------------------------------------------------


    // Control Point (Knot) LUT
    // ----------------------------------------------------------
    dev_ptrs->LUT_Knot_size = 64*num_tiles*sizeof(int);

    int* local_set_of_64 = (int*)malloc(64*sizeof(int));
    int* LUT_Knot = (int*)malloc(dev_ptrs->LUT_Knot_size);

    int i,j;
    for (i = 0; i < num_tiles; i++)
    {
        find_knots(local_set_of_64, i, bxf->rdims, bxf->cdims);
        for (j = 0; j < 64; j++) {
            LUT_Knot[64*i + j] = local_set_of_64[j];
        }
    }

    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Knot,
                    (void **)&LUT_Knot,
                    dev_ptrs->LUT_Knot_size,
                    cudaGlobalMem);

    free (local_set_of_64);
    free (LUT_Knot);

    GPU_Memory_Bytes += dev_ptrs->LUT_Knot_size;
    printf (".");
    // ----------------------------------------------------------



    // B-spline LUT
    // ----------------------------------------------------------
    dev_ptrs->LUT_Bspline_x_size = 4*bxf->vox_per_rgn[0]* sizeof(float);
    dev_ptrs->LUT_Bspline_y_size = 4*bxf->vox_per_rgn[1]* sizeof(float);
    dev_ptrs->LUT_Bspline_z_size = 4*bxf->vox_per_rgn[2]* sizeof(float);
    float* LUT_Bspline_x = (float*)malloc(dev_ptrs->LUT_Bspline_x_size);
    float* LUT_Bspline_y = (float*)malloc(dev_ptrs->LUT_Bspline_y_size);
    float* LUT_Bspline_z = (float*)malloc(dev_ptrs->LUT_Bspline_z_size);

    for (j = 0; j < 4; j++)
    {
        for (i = 0; i < bxf->vox_per_rgn[0]; i++) {
            LUT_Bspline_x[j*bxf->vox_per_rgn[0] + i] = CPU_obtain_spline_basis_function (j, i, bxf->vox_per_rgn[0]);
        }

        for (i = 0; i < bxf->vox_per_rgn[1]; i++) {
            LUT_Bspline_y[j*bxf->vox_per_rgn[1] + i] = CPU_obtain_spline_basis_function (j, i, bxf->vox_per_rgn[1]);
        }

        for (i = 0; i < bxf->vox_per_rgn[2]; i++) {
            LUT_Bspline_z[j*bxf->vox_per_rgn[2] + i] = CPU_obtain_spline_basis_function (j, i, bxf->vox_per_rgn[2]);
        }
    }

    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Bspline_x,
                    (void **)&LUT_Bspline_x,
                    dev_ptrs->LUT_Bspline_x_size,
                    cudaGlobalMem);

    hipBindTexture(0, tex_LUT_Bspline_x, dev_ptrs->LUT_Bspline_x, dev_ptrs->LUT_Bspline_x_size);
    GPU_Memory_Bytes += dev_ptrs->LUT_Bspline_x_size;
    printf(".");


    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Bspline_y,
                    (void **)&LUT_Bspline_y,
                    dev_ptrs->LUT_Bspline_y_size,
                    cudaGlobalMem);

    hipBindTexture(0, tex_LUT_Bspline_y, dev_ptrs->LUT_Bspline_y, dev_ptrs->LUT_Bspline_y_size);
    GPU_Memory_Bytes += dev_ptrs->LUT_Bspline_y_size;
    printf(".");

    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Bspline_z,
                    (void **)&LUT_Bspline_z,
                    dev_ptrs->LUT_Bspline_z_size,
                    cudaGlobalMem);

    hipBindTexture(0, tex_LUT_Bspline_z, dev_ptrs->LUT_Bspline_z, dev_ptrs->LUT_Bspline_z_size);
    GPU_Memory_Bytes += dev_ptrs->LUT_Bspline_z_size;
    printf(".");


    free (LUT_Bspline_x);
    free (LUT_Bspline_y);
    free (LUT_Bspline_z);
    // ----------------------------------------------------------

    // Inform user we are finished.
    printf (" done.\n");

    // Report global memory allocation.
    printf("             Real GPU Memory: %ld MB\n", GPU_Memory_Bytes / 1048576);
    printf("          Virtual GPU Memory: %ld MB\n", gpu_tally_vmem (dev_ptrs) / 1048576);
    printf("Explicitly Pinned CPU Memory: %ld MB\n", CPU_Pinned_Bytes / 1048576);


#if defined (commentout)
    printf ("---------------------------\n");
    gpu_print_vmem (dev_ptrs);
    printf ("---------------------------\n");
    printf ("Skipped Voxels: %i MB\n", dev_ptrs->skipped_size / 1048576);
    printf ("         Score: %i MB\n", dev_ptrs->score_size / 1048576);
    printf ("       dc_dv_x: %i MB\n", dev_ptrs->dc_dv_x_size / 1048576);
    printf ("       dc_dv_y: %i MB\n", dev_ptrs->dc_dv_y_size / 1048576);
    printf ("       dc_dv_z: %i MB\n", dev_ptrs->dc_dv_z_size / 1048576);
    printf ("        cond_x: %i MB\n", dev_ptrs->cond_x_size / 1048576);
    printf ("        cond_y: %i MB\n", dev_ptrs->cond_y_size / 1048576);
    printf ("        cond_z: %i MB\n", dev_ptrs->cond_z_size / 1048576);
    printf ("    Fixed Hist: %i KB\n", dev_ptrs->f_hist_size / 1024);
    printf ("   Moving Hist: %i KB\n", dev_ptrs->m_hist_size / 1024);
    printf ("    Joint Hist: %i KB\n", dev_ptrs->j_hist_size / 1024);
    printf ("         q-lut: %i KB\n", dev_ptrs->q_lut_size / 1024);
    printf ("         c-lut: %i KB\n", dev_ptrs->c_lut_size / 1024);
    printf ("     coeff-lut: %i KB\n", dev_ptrs->coeff_size / 1024);
    printf ("      Gradient: %i KB\n", dev_ptrs->grad_size / 1024);
    printf ("  Tile Offsets: %i KB\n", dev_ptrs->LUT_Offsets_size / 1024);
    printf ("      Knot LUT: %i KB\n", dev_ptrs->LUT_Knot_size / 1024);
    printf ("B-spline LUT-x: %i KB\n", dev_ptrs->LUT_Bspline_x_size / 1024);
    printf ("B-spline LUT-y: %i KB\n", dev_ptrs->LUT_Bspline_y_size / 1024);
    printf ("B-spline LUT-z: %i KB\n", dev_ptrs->LUT_Bspline_z_size / 1024);
    printf ("---------------------------\n");
#endif
}


////////////////////////////////////////////////////////////////////////////////
// FUNCTION: bspline_cuda_initialize_j_zcpy()
// 
// Initialize the GPU to execute bspline_cuda_score_j_mse(),
// but use some zero-copy for stuff like image volumes.
//
// AUTHOR: James Shackleford
// DATE  : October 26, 2010
////////////////////////////////////////////////////////////////////////////////
void
bspline_cuda_initialize_j (
    Dev_Pointers_Bspline* dev_ptrs,
    Volume* fixed,
    Volume* moving,
    Volume* moving_grad,
    Bspline_xform* bxf,
    Bspline_parms* parms
)
{
    // Keep track of how much memory we allocated
    // in the GPU global memory.
    long unsigned GPU_Memory_Bytes = 0;

    printf ("Allocating GPU Memory");

    // Get GPU properties (can we zero-copy?)
    parms->gpu_zcpy = gpu_zero_copy_check (parms) && parms->gpu_zcpy;
    if (parms->gpu_zcpy) {
        // Enable GPU to use pinned CPU memory
        hipSetDeviceFlags (hipDeviceMapHost);
        printf ("GPU is using zero copy\n");
    }

    // Fixed Image (zero copy if possible)
    // ----------------------------------------------------------
    dev_ptrs->fixed_image_size = fixed->npix * fixed->pix_size;
    if (parms->gpu_zcpy) {
        gpu_alloc_copy ((void **)&dev_ptrs->fixed_image,
                        (void **)&fixed->img,
                        dev_ptrs->fixed_image_size,
                        cudaZeroCopy);
    } else {
        gpu_alloc_copy ((void **)&dev_ptrs->fixed_image,
                        (void **)&fixed->img,
                        dev_ptrs->fixed_image_size,
                        cudaGlobalMem);
        GPU_Memory_Bytes += dev_ptrs->fixed_image_size;
        printf(".");
    }
    // ----------------------------------------------------------


    // Moving Image (must be global)
    // ----------------------------------------------------------
    dev_ptrs->moving_image_size = moving->npix * moving->pix_size;
    gpu_alloc_copy ((void **)&dev_ptrs->moving_image,
                    (void **)&moving->img,
                    dev_ptrs->moving_image_size,
                    cudaGlobalMem);

    hipBindTexture(0, tex_moving_image,
                    dev_ptrs->moving_image,
                    dev_ptrs->moving_image_size);

    cuda_utils_check_error("Failed to bind dev_ptrs->moving_image to texture reference!");
    GPU_Memory_Bytes += dev_ptrs->moving_image_size;
    printf(".");
    // ----------------------------------------------------------


    // Moving Image Gradient
    // ----------------------------------------------------------
    dev_ptrs->moving_grad_size = moving_grad->npix * moving_grad->pix_size;
    if (parms->gpu_zcpy) {
        gpu_alloc_copy ((void **)&dev_ptrs->moving_grad,
                        (void **)&moving_grad->img,
                        dev_ptrs->moving_grad_size,
                        cudaZeroCopy);
    } else {
        gpu_alloc_copy ((void **)&dev_ptrs->moving_grad,
                        (void **)&moving_grad->img,
                        dev_ptrs->moving_grad_size,
                        cudaGlobalMem);
        GPU_Memory_Bytes += dev_ptrs->moving_grad_size;
        printf(".");
    }
    // ----------------------------------------------------------


    // Coefficient LUT
    // ----------------------------------------------------------
    dev_ptrs->coeff_size = sizeof(float) * bxf->num_coeff;
    gpu_alloc_zero ((void **)&dev_ptrs->coeff,
                    dev_ptrs->coeff_size,
                    cudaAllocStern);

    hipBindTexture(0, tex_coeff,
                    dev_ptrs->coeff,
                    dev_ptrs->coeff_size);

    cuda_utils_check_error("Failed to bind dev_ptrs->coeff to texture reference!");
    GPU_Memory_Bytes += dev_ptrs->coeff_size;
    printf(".");
    // ----------------------------------------------------------



    // Score
    // ----------------------------------------------------------
    dev_ptrs->score_size = sizeof(float) * fixed->npix;
    gpu_alloc_zero ((void **)&dev_ptrs->score,
                    dev_ptrs->score_size,
                    cudaAllocStern);

    GPU_Memory_Bytes += dev_ptrs->score_size;
    printf(".");
    // ----------------------------------------------------------



    // Skipped Voxels
    // ----------------------------------------------------------
    dev_ptrs->skipped_size = sizeof(float) * fixed->npix;
    gpu_alloc_zero ((void **)&dev_ptrs->skipped,
                    dev_ptrs->skipped_size,
                    cudaAllocStern);

    GPU_Memory_Bytes += dev_ptrs->skipped_size;
    printf(".");
    // ----------------------------------------------------------



    // Gradient (dC_cP)
    // ----------------------------------------------------------
    dev_ptrs->grad_size = sizeof(float) * bxf->num_coeff;
    gpu_alloc_zero ((void **)&dev_ptrs->grad,
                    dev_ptrs->grad_size,
                    cudaAllocStern);

    cuda_utils_check_error("Failed to bind dev_ptrs->grad to texture reference!");
    GPU_Memory_Bytes += dev_ptrs->grad_size;
    printf(".");
    // ----------------------------------------------------------


    // dc_dv_x,  dc_dv_y,  and  dc_dv_z
    // ----------------------------------------------------------
    int3 vol_dim;
    vol_dim.x = fixed->dim[0];
    vol_dim.y = fixed->dim[1];
    vol_dim.z = fixed->dim[2];

    int3 tile_dim;
    tile_dim.x = bxf->vox_per_rgn[0];
    tile_dim.y = bxf->vox_per_rgn[1];
    tile_dim.z = bxf->vox_per_rgn[2];

    int4 num_tile;
    num_tile.x = (vol_dim.x+tile_dim.x-1) / tile_dim.x;
    num_tile.y = (vol_dim.y+tile_dim.y-1) / tile_dim.y;
    num_tile.z = (vol_dim.z+tile_dim.z-1) / tile_dim.z;
    num_tile.w = num_tile.x * num_tile.y * num_tile.z;

    int tile_padding = 64 - ((tile_dim.x * tile_dim.y * tile_dim.z) % 64);
    int tile_bytes = (tile_dim.x * tile_dim.y * tile_dim.z);

    dev_ptrs->dc_dv_x_size = ((tile_bytes + tile_padding) * num_tile.w) * sizeof(float);
    dev_ptrs->dc_dv_y_size = dev_ptrs->dc_dv_x_size;
    dev_ptrs->dc_dv_z_size = dev_ptrs->dc_dv_x_size;


    gpu_alloc_zero ((void **)&dev_ptrs->dc_dv_x,
                    dev_ptrs->dc_dv_x_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->dc_dv_x_size;
    printf(".");

    gpu_alloc_zero ((void **)&dev_ptrs->dc_dv_y,
                    dev_ptrs->dc_dv_y_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->dc_dv_y_size;
    printf(".");

    gpu_alloc_zero ((void **)&dev_ptrs->dc_dv_z,
                    dev_ptrs->dc_dv_z_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->dc_dv_z_size;
    printf(".");
    // ----------------------------------------------------------


    // Tile Offset LUT
    // ----------------------------------------------------------
    int* offsets = calc_offsets(bxf->vox_per_rgn, bxf->cdims);
    int num_tiles = (bxf->cdims[0]-3) * (bxf->cdims[1]-3) * (bxf->cdims[2]-3);

    dev_ptrs->LUT_Offsets_size = num_tiles*sizeof(int);

    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Offsets,
                    (void **)&offsets,
                    dev_ptrs->LUT_Offsets_size,
                    cudaGlobalMem);

    GPU_Memory_Bytes += dev_ptrs->LUT_Offsets_size;
    printf(".");

    free (offsets);
    // ----------------------------------------------------------


    // Control Point (Knot) LUT
    // ----------------------------------------------------------
    dev_ptrs->LUT_Knot_size = 64*num_tiles*sizeof(int);

    int* local_set_of_64 = (int*)malloc(64*sizeof(int));
    int* LUT_Knot = (int*)malloc(dev_ptrs->LUT_Knot_size);

    int i,j;
    for (i = 0; i < num_tiles; i++)
    {
        find_knots(local_set_of_64, i, bxf->rdims, bxf->cdims);
        for (j = 0; j < 64; j++) {
            LUT_Knot[64*i + j] = local_set_of_64[j];
        }
    }

    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Knot,
                    (void **)&LUT_Knot,
                    dev_ptrs->LUT_Knot_size,
                    cudaGlobalMem);

    free (local_set_of_64);
    free (LUT_Knot);

    GPU_Memory_Bytes += dev_ptrs->LUT_Knot_size;
    printf (".");
    // ----------------------------------------------------------



    // Condensed dc_dv vectors
    // ----------------------------------------------------------
    dev_ptrs->cond_x_size = 64*bxf->num_knots*sizeof(float);
    gpu_alloc_zero ((void **)&dev_ptrs->cond_x,
                    dev_ptrs->cond_x_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->cond_x_size;
    printf(".");

    dev_ptrs->cond_y_size = 64*bxf->num_knots*sizeof(float);
    gpu_alloc_zero ((void **)&dev_ptrs->cond_y,
                    dev_ptrs->cond_y_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->cond_y_size;
    printf(".");

    dev_ptrs->cond_z_size = 64*bxf->num_knots*sizeof(float);
    gpu_alloc_zero ((void **)&dev_ptrs->cond_z,
                    dev_ptrs->cond_z_size,
                    cudaAllocStern);
    GPU_Memory_Bytes += dev_ptrs->cond_z_size;
    printf(".");
    // ----------------------------------------------------------


    // B-spline LUT
    // ----------------------------------------------------------
    dev_ptrs->LUT_Bspline_x_size = 4*bxf->vox_per_rgn[0]* sizeof(float);
    dev_ptrs->LUT_Bspline_y_size = 4*bxf->vox_per_rgn[1]* sizeof(float);
    dev_ptrs->LUT_Bspline_z_size = 4*bxf->vox_per_rgn[2]* sizeof(float);
    float* LUT_Bspline_x = (float*)malloc(dev_ptrs->LUT_Bspline_x_size);
    float* LUT_Bspline_y = (float*)malloc(dev_ptrs->LUT_Bspline_y_size);
    float* LUT_Bspline_z = (float*)malloc(dev_ptrs->LUT_Bspline_z_size);

    for (j = 0; j < 4; j++)
    {
        for (i = 0; i < bxf->vox_per_rgn[0]; i++) {
            LUT_Bspline_x[j*bxf->vox_per_rgn[0] + i] = CPU_obtain_spline_basis_function (j, i, bxf->vox_per_rgn[0]);
        }

        for (i = 0; i < bxf->vox_per_rgn[1]; i++) {
            LUT_Bspline_y[j*bxf->vox_per_rgn[1] + i] = CPU_obtain_spline_basis_function (j, i, bxf->vox_per_rgn[1]);
        }

        for (i = 0; i < bxf->vox_per_rgn[2]; i++) {
            LUT_Bspline_z[j*bxf->vox_per_rgn[2] + i] = CPU_obtain_spline_basis_function (j, i, bxf->vox_per_rgn[2]);
        }
    }

    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Bspline_x,
                    (void **)&LUT_Bspline_x,
                    dev_ptrs->LUT_Bspline_x_size,
                    cudaGlobalMem);

    hipBindTexture(0, tex_LUT_Bspline_x, dev_ptrs->LUT_Bspline_x, dev_ptrs->LUT_Bspline_x_size);
    GPU_Memory_Bytes += dev_ptrs->LUT_Bspline_x_size;
    printf(".");


    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Bspline_y,
                    (void **)&LUT_Bspline_y,
                    dev_ptrs->LUT_Bspline_y_size,
                    cudaGlobalMem);

    hipBindTexture(0, tex_LUT_Bspline_y, dev_ptrs->LUT_Bspline_y, dev_ptrs->LUT_Bspline_y_size);
    GPU_Memory_Bytes += dev_ptrs->LUT_Bspline_y_size;
    printf(".");

    gpu_alloc_copy ((void **)&dev_ptrs->LUT_Bspline_z,
                    (void **)&LUT_Bspline_z,
                    dev_ptrs->LUT_Bspline_z_size,
                    cudaGlobalMem);

    hipBindTexture(0, tex_LUT_Bspline_z, dev_ptrs->LUT_Bspline_z, dev_ptrs->LUT_Bspline_z_size);
    GPU_Memory_Bytes += dev_ptrs->LUT_Bspline_z_size;
    printf(".");


    free (LUT_Bspline_x);
    free (LUT_Bspline_y);
    free (LUT_Bspline_z);
    // ----------------------------------------------------------

    // Inform user we are finished.
    printf("done.\n");

    // Report global memory allocation.
    printf("  Allocated: %ld MB\n", GPU_Memory_Bytes / 1048576);

}
////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
// FUNCTION: bspline_cuda_clean_up_j()
//
// AUTHOR: James Shackleford
// DATE  : September 11th, 2009
////////////////////////////////////////////////////////////////////////////////
void
bspline_cuda_clean_up_mse_j (
    dev_pointers_bspline* dev_ptrs,
    volume* fixed,
    volume* moving,
    volume* moving_grad
)
{
    // Textures
    hipUnbindTexture(tex_moving_image);
    hipUnbindTexture(tex_coeff);
    hipUnbindTexture(tex_LUT_Bspline_x);
    hipUnbindTexture(tex_LUT_Bspline_y);
    hipUnbindTexture(tex_LUT_Bspline_z);

    // Zero paged memory
    hipHostFree (fixed->img);
    if (hipGetLastError() == hipSuccess) { fixed->img = 0; }
    hipHostFree (moving->img);
    if (hipGetLastError() == hipSuccess) { moving->img = 0; }
    hipHostFree (moving_grad->img);
    if (hipGetLastError() == hipSuccess) { moving_grad->img = 0; }
    
    // Global Memory
    hipFree(dev_ptrs->fixed_image);
    hipFree(dev_ptrs->moving_image);
    hipFree(dev_ptrs->moving_grad);
    hipFree(dev_ptrs->coeff);
    hipFree(dev_ptrs->score);
    hipFree(dev_ptrs->grad);
    hipFree(dev_ptrs->dc_dv_x);
    hipFree(dev_ptrs->dc_dv_y);
    hipFree(dev_ptrs->dc_dv_z);
    hipFree(dev_ptrs->LUT_Offsets);
    hipFree(dev_ptrs->LUT_Knot);
    hipFree(dev_ptrs->cond_x);
    hipFree(dev_ptrs->cond_y);
    hipFree(dev_ptrs->cond_z);
    hipFree(dev_ptrs->LUT_Bspline_x);
    hipFree(dev_ptrs->LUT_Bspline_y);
    hipFree(dev_ptrs->LUT_Bspline_z);
    hipFree(dev_ptrs->skipped);

}


////////////////////////////////////////////////////////////////////////////////
// FUNCTION: bspline_cuda_clean_up_mi_a()
//
// AUTHOR: James Shackleford
// DATE  : October 29th, 2010
////////////////////////////////////////////////////////////////////////////////
void
bspline_cuda_clean_up_mi_a (
    dev_pointers_bspline* dev_ptrs,
    volume* fixed,
    volume* moving,
    volume* moving_grad
)
{
    // Textures
    hipUnbindTexture(tex_coeff);
    hipUnbindTexture(tex_LUT_Bspline_x);
    hipUnbindTexture(tex_LUT_Bspline_y);
    hipUnbindTexture(tex_LUT_Bspline_z);

    // Zero paged memory
    hipHostFree (fixed->img);
    if (hipGetLastError() == hipSuccess) { fixed->img = 0; }
    hipHostFree (moving->img);
    if (hipGetLastError() == hipSuccess) { moving->img = 0; }

    // Things that *could* have been allocated into gpu vmem
    if (gpu_free_vmem (dev_ptrs->dc_dv_x, dev_ptrs)) {
        hipFree(dev_ptrs->dc_dv_x);
    }
    if (gpu_free_vmem (dev_ptrs->dc_dv_y, dev_ptrs)) {
        hipFree(dev_ptrs->dc_dv_y);
    }
    if (gpu_free_vmem (dev_ptrs->dc_dv_z, dev_ptrs)) {
        hipFree(dev_ptrs->dc_dv_z);
    }
    
    // Global Memory
    hipFree(dev_ptrs->fixed_image);
    hipFree(dev_ptrs->moving_image);
    hipFree(dev_ptrs->skipped);
    hipFree(dev_ptrs->f_hist);
    hipFree(dev_ptrs->m_hist);
    hipFree(dev_ptrs->j_hist);
    hipFree(dev_ptrs->q_lut);
    hipFree(dev_ptrs->c_lut);
    hipFree(dev_ptrs->coeff);
    hipFree(dev_ptrs->score);
    hipFree(dev_ptrs->grad);
    hipFree(dev_ptrs->cond_x);
    hipFree(dev_ptrs->cond_y);
    hipFree(dev_ptrs->cond_z);
    hipFree(dev_ptrs->LUT_Offsets);
    hipFree(dev_ptrs->LUT_Knot);
    hipFree(dev_ptrs->LUT_Bspline_x);
    hipFree(dev_ptrs->LUT_Bspline_y);
    hipFree(dev_ptrs->LUT_Bspline_z);

    // Just to be safe (in case we missed something)
    gpu_freeall_vmem (dev_ptrs);
}


extern "C" int
CUDA_bspline_MI_a_hist (
    Dev_Pointers_Bspline *dev_ptrs,
    BSPLINE_MI_Hist* mi_hist,
    Volume* fixed,
    Volume* moving,
    Bspline_xform* bxf)
{
    hipMemset(dev_ptrs->skipped_atomic, 0, dev_ptrs->skipped_size);

    // Generate the fixed histogram (48 ms)
    CUDA_bspline_MI_a_hist_fix (dev_ptrs, mi_hist, fixed, moving, bxf);

    // Generate the moving histogram (150 ms)
    CUDA_bspline_MI_a_hist_mov (dev_ptrs, mi_hist, fixed, moving, bxf);

    // Generate the joint histogram (~600 ms)
    return CUDA_bspline_MI_a_hist_jnt (dev_ptrs, mi_hist, fixed, moving, bxf);
}



extern "C" void
CUDA_bspline_MI_a_hist_fix (
    Dev_Pointers_Bspline *dev_ptrs,
    BSPLINE_MI_Hist* mi_hist,
    Volume* fixed,
    Volume* moving,
    Bspline_xform *bxf)
{
    dim3 dimGrid;
    dim3 dimBlock;
    int num_blocks;

    GPU_Bspline_Data gbd; 
    build_gbd (&gbd, bxf, fixed, moving);

    // Initialize histogram memory on GPU
    hipMemset(dev_ptrs->f_hist, 0, dev_ptrs->f_hist_size);
    cuda_utils_check_error ("Failed to initialize memory for f_hist");

    num_blocks = build_exec_conf_1tpe (
        &dimGrid,          // OUTPUT: Grid  dimensions
        &dimBlock,         // OUTPUT: Block dimensions
        fixed->npix,       // INPUT: Total # of threads
        32,                // INPUT: Threads per block
        false);            // INPUT: Is threads per block negotiable?

    int smemSize = dimBlock.x * mi_hist->fixed.bins * sizeof(float);

    dev_ptrs->f_hist_seg_size = mi_hist->fixed.bins * num_blocks * sizeof(float);
    hipMalloc ((void**)&dev_ptrs->f_hist_seg, dev_ptrs->f_hist_seg_size);
    cuda_utils_check_error ("Failed to allocate memory for f_hist_seg");
    hipMemset(dev_ptrs->f_hist_seg, 0, dev_ptrs->f_hist_seg_size);
    cuda_utils_check_error ("Failed to initialize memory for f_hist_seg");


    // Launch kernel with one thread per voxel
    kernel_bspline_MI_a_hist_fix <<<dimGrid, dimBlock, smemSize>>> (
        dev_ptrs->f_hist_seg,       // partial histogram (moving image)
        dev_ptrs->fixed_image,      // moving image voxels
        mi_hist->fixed.offset,      // histogram offset
        1.0f/mi_hist->fixed.delta,  // histogram delta
        mi_hist->fixed.bins,        // # histogram bins
        gbd.vox_per_rgn,            // voxels per region
        gbd.fix_dim,                // fixed  image dimensions
        gbd.mov_dim,                // moving image dimensions
        gbd.rdims,                  //       region dimensions
        gbd.cdims,                  // # control points in x,y,z
        gbd.img_origin,             // image origin
        gbd.img_spacing,            // image spacing
        gbd.mov_offset,             // moving image offset
        gbd.mov_spacing,            // moving image pixel spacing
        dev_ptrs->c_lut,            // DEBUG
        dev_ptrs->q_lut,            // DEBUG
        dev_ptrs->coeff);           // DEBUG

    cuda_utils_check_error ("kernel hist_mov");

    int num_sub_hists = num_blocks;

    // Merge sub-histograms
    dim3 dimGrid2 (mi_hist->fixed.bins, 1, 1);
    dim3 dimBlock2 (512, 1, 1);
    smemSize = 512 * sizeof(float);
    
    // this kernel can be ran with any thread-block size
    kernel_bspline_MI_a_hist_fix_merge <<<dimGrid2 , dimBlock2, smemSize>>> (
                	dev_ptrs->f_hist,
                	dev_ptrs->f_hist_seg,
                	num_sub_hists);

    cuda_utils_check_error ("kernel hist_fix_merge");

    /* copy result back to host
     *   -- Note CPU uses doubles whereas the GPU uses floats
     *      due to lack of double precision floats.  This is okay
     *      since the GPU's ability to add small numbers to large
     *      using single precision is more accurate than the CPU.
     *   
     *   -- However, this does result in the little bit of nastiness
     *      found below.  We copy these back to the CPU for the score
     *      computation, which the CPU completes very quickly.
     */
    float* f_hist_f = (float*)malloc(dev_ptrs->f_hist_size);

    hipMemcpy (f_hist_f, dev_ptrs->f_hist, dev_ptrs->f_hist_size, hipMemcpyDeviceToHost);
    cuda_utils_check_error ("Unable to copy fixed histograms from GPU to CPU!\n");

    /* type cast to CPU friendly double */
    for (int i=0; i< mi_hist->fixed.bins; i++) {
        mi_hist->f_hist[i] = (double)f_hist_f[i];
    }

    free (f_hist_f);

    hipFree (dev_ptrs->f_hist_seg);
    cuda_utils_check_error ("Error freeing sub-histograms from GPU memory!\n");

}


extern "C" void
CUDA_bspline_MI_a_hist_mov (
    Dev_Pointers_Bspline *dev_ptrs,
    BSPLINE_MI_Hist* mi_hist,
    Volume* fixed,
    Volume* moving,
    Bspline_xform *bxf)
{
    dim3 dimGrid;
    dim3 dimBlock;
    int num_blocks;

    GPU_Bspline_Data gbd;
    build_gbd (&gbd, bxf, fixed, moving);

    // Initialize histogram memory on GPU
    hipMemset(dev_ptrs->m_hist, 0, dev_ptrs->m_hist_size);
    cuda_utils_check_error ("Failed to initialize memory for m_hist");
    
    num_blocks = 
	build_exec_conf_1tpe (
	    &dimGrid,          // OUTPUT: Grid  dimensions
	    &dimBlock,         // OUTPUT: Block dimensions
	    fixed->npix,       // INPUT: Total # of threads
	    32,                // INPUT: Threads per block
	    false);            // INPUT: Is threads per block negotiable?

    int smemSize = dimBlock.x * mi_hist->moving.bins * sizeof(float);


    dev_ptrs->m_hist_seg_size = mi_hist->moving.bins * num_blocks * sizeof(float);
    hipMalloc ((void**)&dev_ptrs->m_hist_seg, dev_ptrs->m_hist_seg_size);
    cuda_utils_check_error ("Failed to allocate memory for m_hist_seg");
    hipMemset(dev_ptrs->m_hist_seg, 0, dev_ptrs->m_hist_seg_size);
    cuda_utils_check_error ("Failed to initialize memory for m_hist_seg");


    // Launch kernel with one thread per voxel
    kernel_bspline_MI_a_hist_mov <<<dimGrid, dimBlock, smemSize>>> (
        dev_ptrs->m_hist_seg,       // partial histogram (moving image)
        dev_ptrs->moving_image,     // moving image voxels
        mi_hist->moving.offset,     // histogram offset
        1.0f/mi_hist->moving.delta, // histogram delta
        mi_hist->moving.bins,       // # histogram bins
        gbd.vox_per_rgn,            // voxels per region
        gbd.fix_dim,                // fixed  image dimensions
        gbd.mov_dim,                // moving image dimensions
        gbd.rdims,                  //       region dimensions
        gbd.cdims,                  // # control points in x,y,z
        gbd.img_origin,             // image origin
        gbd.img_spacing,            // image spacing
        gbd.mov_offset,             // moving image offset
        gbd.mov_spacing,            // moving image pixel spacing
        dev_ptrs->c_lut,            // DEBUG
        dev_ptrs->q_lut,            // DEBUG
        dev_ptrs->coeff);           // DEBUG

    cuda_utils_check_error ("kernel hist_mov");

    int num_sub_hists = num_blocks;


    // Merge sub-histograms
    dim3 dimGrid2 (mi_hist->moving.bins, 1, 1);
    dim3 dimBlock2 (512, 1, 1);
    smemSize = 512 * sizeof(float);
    
    // this kernel can be ran with any thread-block size
    kernel_bspline_MI_a_hist_fix_merge <<<dimGrid2 , dimBlock2, smemSize>>> (
        dev_ptrs->m_hist,
        dev_ptrs->m_hist_seg,
        num_sub_hists);

    cuda_utils_check_error ("kernel hist_mov_merge");

    /* copy result back to host
     *   -- Note CPU uses doubles whereas the GPU uses floats
     *      due to lack of double precision floats.  This is okay
     *      since the GPU's ability to add small numbers to large
     *      using single precision is more accurate than the CPU.
     *   
     *   -- However, this does result in the little bit of nastiness
     *      found below.  We copy these back to the CPU for the score
     *      computation, which the CPU completes very quickly.
     */
    float* m_hist_f = (float*)malloc(dev_ptrs->m_hist_size);

    hipMemcpy (m_hist_f, dev_ptrs->m_hist, dev_ptrs->m_hist_size, hipMemcpyDeviceToHost);
    cuda_utils_check_error ("Unable to copy moving histograms from GPU to CPU!\n");

    /* type cast to CPU friendly double */
    for (int i=0; i< mi_hist->moving.bins; i++) {
        mi_hist->m_hist[i] = (double)m_hist_f[i];
    }

    free (m_hist_f);

    hipFree (dev_ptrs->m_hist_seg);
    cuda_utils_check_error ("Error freeing sub-histograms from GPU memory!\n");

}


extern "C" int
CUDA_bspline_MI_a_hist_jnt (
    Dev_Pointers_Bspline *dev_ptrs,
    BSPLINE_MI_Hist* mi_hist,
    Volume* fixed,
    Volume* moving,
    Bspline_xform *bxf)
{
    GPU_Bspline_Data gbd;
    build_gbd (&gbd, bxf, fixed, moving);

    // Initialize histogram memory on GPU
    hipMemset(dev_ptrs->j_hist, 0, dev_ptrs->j_hist_size);


    int num_bins = (int)mi_hist->fixed.bins * (int)mi_hist->moving.bins;


    // ----------------------
    // --- INITIALIZE GRID ---
    // ----------------------
    int i;
    int Grid_x = 0;
    int Grid_y = 0;
    int threads_per_block = 128;
    int num_threads = fixed->npix;
    int sqrt_num_blocks;
    int num_blocks;
    int smemSize;
    int found_flag = 0;

    // Search for a valid execution configuration
    // for the required # of blocks.
    for (threads_per_block = 192; threads_per_block > 32; threads_per_block -= 32) {
        num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;
        sqrt_num_blocks = (int)sqrt((float)num_blocks);

        for (i = sqrt_num_blocks; i < 65535; i++) {
            if (num_blocks % i == 0) {
                Grid_x = i;
                Grid_y = num_blocks / Grid_x;
                found_flag = 1;
                break;
            }
        }

        if (found_flag == 1) {
            break;
        }
    }

    // Were we able to find a valid exec config?
    if (Grid_x == 0) {
        printf("\n[ERROR] Unable to find suitable bspline_cuda_score_j_mse_kernel1() configuration!\n");
        exit(0);
    } else {
//        printf ("Grid [%i,%i], %d threads_per_block.\n", Grid_x, Grid_y, threads_per_block);
    }

    dim3 dimGrid1(Grid_x, Grid_y, 1);
    dim3 dimBlock1(threads_per_block, 1, 1);
    // ----------------------
    // ----------------------
    // ----------------------

    dev_ptrs->j_hist_seg_size = dev_ptrs->j_hist_size * num_blocks;

#if defined (commentout)
    printf ("Attempting to allocate j_hist_set (%i MB)\n", dev_ptrs->j_hist_seg_size/1048576);
    int out_of_gmem = 
    gpu_alloc_zero ((void **)&dev_ptrs->j_hist_seg,
                    dev_ptrs->j_hist_seg_size,
                    cudaAllocCasual);

    if (out_of_gmem) {
//        if (parms->gpu_zcpy) {
            gpu_alloc_vmem ((void **)&dev_ptrs->j_hist_seg,
                            dev_ptrs->j_hist_seg_size,
                            dev_ptrs);
//        } else {
//            printf ("Failed to allocate memory for j_hist_seg\n");
//            exit (0);
//        }
    }
#endif

    hipMalloc ((void**)&dev_ptrs->j_hist_seg, dev_ptrs->j_hist_seg_size);
    hipMemset(dev_ptrs->j_hist_seg, 0, dev_ptrs->j_hist_seg_size);
    cuda_utils_check_error ("Failed to allocate memory for j_hist_seg");
    smemSize = (num_bins + 1) * sizeof(float);

    // Launch kernel with one thread per voxel
    kernel_bspline_MI_a_hist_jnt <<<dimGrid1, dimBlock1, smemSize>>> (
            dev_ptrs->skipped_atomic,   // # voxels that map outside moving
            dev_ptrs->j_hist_seg,       // partial histogram (moving image)
            dev_ptrs->fixed_image,      // fixed  image voxels
            dev_ptrs->moving_image,     // moving image voxels
            mi_hist->fixed.offset,      // fixed histogram offset
            mi_hist->moving.offset,     // moving histogram offset
            1.0f/mi_hist->fixed.delta,  // fixed histogram delta
            1.0f/mi_hist->moving.delta, // moving histogram delta
            mi_hist->fixed.bins,        // # fixed bins
            mi_hist->moving.bins,       // # moving bins
            gbd.vox_per_rgn,            // voxels per region
            gbd.fix_dim,                // fixed  image dimensions
            gbd.mov_dim,                // moving image dimensions
            gbd.rdims,                  //       region dimensions
            gbd.cdims,                  // # control points in x,y,z
            gbd.img_origin,             // image origin
            gbd.img_spacing,            // image spacing
            gbd.mov_offset,             // moving image offset
            gbd.mov_spacing,            // moving image pixel spacing
            gbd.roi_dim,                // region dims
            gbd.roi_offset,             // region offset
            dev_ptrs->c_lut,            // DEBUG
            dev_ptrs->q_lut,            // DEBUG
            dev_ptrs->coeff);           // DEBUG

    hipDeviceSynchronize();
    cuda_utils_check_error ("kernel hist_jnt");



    // Merge sub-histograms
    threads_per_block = 512;
    dim3 dimGrid2 (num_bins, 1, 1);
    dim3 dimBlock2 (threads_per_block, 1, 1);
    smemSize = 512 * sizeof(float);

    // this kernel can be ran with any thread-block size
    int num_sub_hists = num_blocks;
    kernel_bspline_MI_a_hist_fix_merge <<<dimGrid2 , dimBlock2, smemSize>>> (
            dev_ptrs->j_hist,
            dev_ptrs->j_hist_seg,
            num_sub_hists);

    cuda_utils_check_error ("kernel hist_jnt_merge");

    /* copy result back to host
     *   -- Note CPU uses doubles whereas the GPU uses floats
     *      due to lack of double precision floats.  This is okay
     *      since the GPU's ability to add small numbers to large
     *      using single precision is more accurate than the CPU.
     *   
     *   -- However, this does result in the little bit of nastiness
     *      found below.  We copy these back to the CPU for the score
     *      computation, which the CPU completes very quickly.
     */
    float* j_hist_f = (float*)malloc(dev_ptrs->j_hist_size);

    hipMemcpy (j_hist_f, dev_ptrs->j_hist, dev_ptrs->j_hist_size, hipMemcpyDeviceToHost);
    cuda_utils_check_error ("Unable to copy joint histograms from GPU to CPU!\n");

    /* type cast to CPU friendly double */
    for (int i=0; i< mi_hist->moving.bins * mi_hist->fixed.bins; i++) {
        mi_hist->j_hist[i] = (double)j_hist_f[i];
    }

    free (j_hist_f);


    hipFree (dev_ptrs->j_hist_seg);
    cuda_utils_check_error ("Error freeing sub-histograms from GPU memory!");


    // Get # of skipped voxels and compute num_vox 
    unsigned int skipped;
    int num_vox;
    hipMemcpy(&skipped, dev_ptrs->skipped_atomic, sizeof(unsigned int), hipMemcpyDeviceToHost);
    num_vox = (gbd.fix_dim.x * gbd.fix_dim.y * gbd.fix_dim.z) - skipped;


    // Now, we back compute bin 0,0 for the joint histogram
    int j = 0;
    for (i = 1; i < mi_hist->fixed.bins * mi_hist->moving.bins; i++) {
        j += mi_hist->j_hist[i];
    }

    mi_hist->j_hist[0] = num_vox - j;

    return num_vox;

}


extern "C" void
CUDA_MI_Grad_a (
    BSPLINE_MI_Hist* mi_hist,
    Bspline_state *bst,
    Bspline_xform *bxf,
    Volume* fixed,
    Volume* moving,
    float num_vox_f,
    Dev_Pointers_Bspline *dev_ptrs)
{
    GPU_Bspline_Data gbd;
    build_gbd (&gbd, bxf, fixed, moving);


    BSPLINE_Score* ssd = &bst->ssd;
    float* host_grad = ssd->grad;
    float score = ssd->score;

    // Initialize histogram memory on GPU
    // (only necessary if histograms are CPU generated)
#if defined (MI_HIST_CPU)
    float* f_hist_f = (float*)malloc(dev_ptrs->f_hist_size);
    float* m_hist_f = (float*)malloc(dev_ptrs->m_hist_size);
    float* j_hist_f = (float*)malloc(dev_ptrs->j_hist_size);

    hipMemcpy (f_hist_f, dev_ptrs->f_hist, dev_ptrs->f_hist_size, hipMemcpyDeviceToHost);
    cuda_utils_check_error ("Unable to copy fixed histograms from GPU to CPU!\n");
    hipMemcpy (m_hist_f, dev_ptrs->m_hist, dev_ptrs->m_hist_size, hipMemcpyDeviceToHost);
    cuda_utils_check_error ("Unable to copy moving histograms from GPU to CPU!\n");
    hipMemcpy (j_hist_f, dev_ptrs->j_hist, dev_ptrs->j_hist_size, hipMemcpyDeviceToHost);
    cuda_utils_check_error ("Unable to copy joint histograms from GPU to CPU!\n");

    /* type cast to CPU friendly double */
    for (int i=0; i< mi_hist->fixed.bins; i++) {
        mi_hist->f_hist[i] = (double)f_hist_f[i];
    }

    for (int i=0; i< mi_hist->moving.bins; i++) {
        mi_hist->m_hist[i] = (double)m_hist_f[i];
    }

    for (int i=0; i< mi_hist->fixed.bin * mi_hist->moving.bins; i++) {
        mi_hist->j_hist[i] = (double)j_hist_f[i];
    }

    free (f_hist_f);
    free (m_hist_f);
    free (j_hist_f);
#endif

    // Initial dc_dv streams
    hipMemset(dev_ptrs->dc_dv_x, 0, dev_ptrs->dc_dv_x_size);
       cuda_utils_check_error("hipMemset(): dev_ptrs->dc_dv_x");
    hipMemset(dev_ptrs->dc_dv_y, 0, dev_ptrs->dc_dv_y_size);
       cuda_utils_check_error("hipMemset(): dev_ptrs->dc_dv_y");
    hipMemset(dev_ptrs->dc_dv_z, 0, dev_ptrs->dc_dv_z_size);
       cuda_utils_check_error("hipMemset(): dev_ptrs->dc_dv_z");
    

    // --- INITIALIZE GRID ---
    int i;
    int Grid_x = 0;
    int Grid_y = 0;
    int threads_per_block = 128;
    int num_threads = fixed->npix;
    int sqrt_num_blocks;
    int num_blocks;
    int found_flag = 0;

    // Search for a valid execution configuration
    // for the required # of blocks.
    for (threads_per_block = 192; threads_per_block > 32; threads_per_block -= 32) {
    num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;
    sqrt_num_blocks = (int)sqrt((float)num_blocks);

        for (i = sqrt_num_blocks; i < 65535; i++) {
            if (num_blocks % i == 0) {
                Grid_x = i;
                Grid_y = num_blocks / Grid_x;
                found_flag = 1;
                break;
            }
        }

        if (found_flag == 1) {
            break;
        }
    }

    // Were we able to find a valid exec config?
    if (Grid_x == 0) {
        // If this happens we should consider falling back to a
        // CPU implementation, using a different CUDA algorithm,
        // or padding the input dc_dv stream to work with this
        // CUDA algorithm.
        printf("\n[ERROR] Unable to find suitable bspline_cuda_score_j_mse_kernel1() configuration!\n");
        exit(0);
    } else {
#if defined (commentout)
        printf ("Grid [%i,%i], %d threads_per_block.\n", 
            Grid_x, Grid_y, threads_per_block);
#endif
    }

    dim3 dimGrid1(Grid_x, Grid_y, 1);
    dim3 dimBlock1(threads_per_block, 1, 1);


    int tile_padding = 64 - ((gbd.vox_per_rgn.x * gbd.vox_per_rgn.y * gbd.vox_per_rgn.z) % 64);

    // Launch kernel with one thread per voxel
    kernel_bspline_MI_dc_dv_a <<<dimGrid1, dimBlock1>>> (
        dev_ptrs->dc_dv_x,
        dev_ptrs->dc_dv_y,
        dev_ptrs->dc_dv_z,  
        dev_ptrs->f_hist,
        dev_ptrs->m_hist,
        dev_ptrs->j_hist,
        dev_ptrs->fixed_image,
        dev_ptrs->moving_image,
        mi_hist->fixed.offset,
        mi_hist->moving.offset,
        1.0f/mi_hist->fixed.delta,
        1.0f/mi_hist->moving.delta,
        mi_hist->fixed.bins,
        mi_hist->moving.bins,
        gbd.vox_per_rgn,
        gbd.fix_dim,
        gbd.mov_dim,
        gbd.rdims,
        gbd.cdims,
        gbd.img_origin,
        gbd.img_spacing,
        gbd.mov_offset,
        gbd.mov_spacing,
        gbd.roi_dim,
        gbd.roi_offset,
        dev_ptrs->c_lut,
        dev_ptrs->q_lut,
        dev_ptrs->coeff,
        num_vox_f,
        score,
        tile_padding);


    ////////////////////////////////
    // Prepare for the next kernel
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_bspline_MI_dc_dv_a()");

    // Clear out the condensed dc_dv streams
    hipMemset(dev_ptrs->cond_x, 0, dev_ptrs->cond_x_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->cond_x");
    hipMemset(dev_ptrs->cond_y, 0, dev_ptrs->cond_y_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->cond_y");
    hipMemset(dev_ptrs->cond_z, 0, dev_ptrs->cond_z_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->cond_z");
    
    // Invoke kernel condense
    int num_tiles = (bxf->cdims[0]-3) * (bxf->cdims[1]-3) * (bxf->cdims[2]-3);
    CUDA_bspline_mse_condense_64_texfetch (
            dev_ptrs,
            bxf->vox_per_rgn, 
            num_tiles);
    
    // Prepare for the next kernel
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_bspline_mse_condense_64_texfetch()");

    // Clear out the gradient
    hipMemset(dev_ptrs->grad, 0, dev_ptrs->grad_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->grad");

    // Invoke kernel reduce
    CUDA_bspline_mse_reduce (dev_ptrs, bxf->num_knots);

    // Prepare for the next kernel
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_bspline_mse_condense()");

    // --- RETREIVE THE GRAD FROM GPU ---------------------------
    hipMemcpy(host_grad, dev_ptrs->grad, sizeof(float) * bxf->num_coeff, hipMemcpyDeviceToHost);
    cuda_utils_check_error("Failed to copy dev_ptrs->grad to CPU");
    // ----------------------------------------------------------
}




//////////////////////////////////////////////////////////////////////////////
// STUB: CUDA_bspline_mse_score_dc_dv()
//
// KERNELS INVOKED:
//   kernel_bspline_mse_reduce()
//
// AUTHOR: James Shackleford
//   DATE: 19 August, 2009
//////////////////////////////////////////////////////////////////////////////
extern "C" void
CUDA_bspline_mse_score_dc_dv (
    Dev_Pointers_Bspline* dev_ptrs,
    Bspline_xform* bxf,
    Volume* fixed,
    Volume* moving)
{
    dim3 dimGrid1;
    dim3 dimBlock1;
    GPU_Bspline_Data gbd;   

    build_gbd (&gbd, bxf, fixed, moving);

    build_exec_conf_1tpe (
        &dimGrid1,          // OUTPUT: Grid  dimensions
        &dimBlock1,         // OUTPUT: Block dimensions
        fixed->npix,        // INPUT: Total # of threads
        192,                // INPUT: Threads per block
        true);              // INPUT: Is threads per block negotiable?

#if defined (commentout)
    int smemSize = 12 * sizeof(float) * dimBlock1.x;
#endif

    // --- BEGIN KERNEL EXECUTION ---
    hipMemset(dev_ptrs->dc_dv_x, 0, dev_ptrs->dc_dv_x_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->dc_dv_x");

    hipMemset(dev_ptrs->dc_dv_y, 0, dev_ptrs->dc_dv_y_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->dc_dv_y");

    hipMemset(dev_ptrs->dc_dv_z, 0, dev_ptrs->dc_dv_z_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->dc_dv_z");

    int tile_padding = 64 - 
    ((gbd.vox_per_rgn.x * gbd.vox_per_rgn.y * gbd.vox_per_rgn.z) % 64);

    kernel_bspline_mse_score_dc_dv <<<dimGrid1, dimBlock1>>> (
            dev_ptrs->score,
            dev_ptrs->skipped,
            dev_ptrs->dc_dv_x,
            dev_ptrs->dc_dv_y,
            dev_ptrs->dc_dv_z,
            dev_ptrs->fixed_image,
            dev_ptrs->moving_image,
            dev_ptrs->moving_grad,
            gbd.fix_dim,
            gbd.mov_dim,
            gbd.rdims,
            gbd.cdims,
            gbd.vox_per_rgn,
            gbd.img_origin,
            gbd.img_spacing,
            gbd.mov_offset,
            gbd.mov_spacing,
            tile_padding);
}


//////////////////////////////////////////////////////////////////////////////
// STUB: CUDA_bspline_mse_condense_64_texfetch()
//
// KERNELS INVOKED:
//   kernel_bspline_mse_condense_64()
//
// AUTHOR: James Shackleford
//   DATE: September 16th, 2009
//////////////////////////////////////////////////////////////////////////////
void
CUDA_bspline_mse_condense_64_texfetch (
    Dev_Pointers_Bspline* dev_ptrs,
    int* vox_per_rgn,
    int num_tiles)
{
    dim3 dimGrid;
    dim3 dimBlock;

    int4 vox_per_region;
    vox_per_region.x = vox_per_rgn[0];
    vox_per_region.y = vox_per_rgn[1];
    vox_per_region.z = vox_per_rgn[2];
    vox_per_region.w = vox_per_region.x * vox_per_region.y * vox_per_region.z;

    int pad = 64 - (vox_per_region.w % 64);

    vox_per_region.w += pad;

    build_exec_conf_1bpe (
        &dimGrid,         // OUTPUT: Grid  dimensions
        &dimBlock,        // OUTPUT: Block dimensions
        num_tiles,        // INPUT: Number of blocks
        64);              // INPUT: Threads per block

    int smemSize = 576*sizeof(float);

    kernel_bspline_mse_condense_64_texfetch<<<dimGrid, dimBlock, smemSize>>>(
        dev_ptrs->cond_x,       // Return: condensed dc_dv_x values
        dev_ptrs->cond_y,       // Return: condensed dc_dv_y values
        dev_ptrs->cond_z,       // Return: condensed dc_dv_z values
        dev_ptrs->dc_dv_x,      // Input : dc_dv_x values
        dev_ptrs->dc_dv_y,      // Input : dc_dv_y values
        dev_ptrs->dc_dv_z,      // Input : dc_dv_z values
        dev_ptrs->LUT_Offsets,  // Input : tile offsets
        dev_ptrs->LUT_Knot,     // Input : linear knot indicies
        pad,                    // Input : amount of tile padding
        vox_per_region,         // Input : dims of tiles
        (float)1/6);            // Input : GPU Division is slow
}
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
// STUB: CUDA_bspline_mse_reduce()
//
// KERNELS INVOKED:
//   kernel_bspline_mse_reduce()
//
// AUTHOR: James Shackleford
//   DATE: 19 August, 2009
////////////////////////////////////////////////////////////////////////////////
extern "C" void
CUDA_bspline_mse_reduce (
    Dev_Pointers_Bspline* dev_ptrs,
    int num_knots)
{
    dim3 dimGrid;
    dim3 dimBlock;

    build_exec_conf_1bpe (
        &dimGrid,         // OUTPUT: Grid  dimensions
        &dimBlock,        // OUTPUT: Block dimensions
        num_knots,        // INPUT: Number of blocks
        64);              // INPUT: Threads per block

    int smemSize = 195*sizeof(float);

    kernel_bspline_mse_reduce<<<dimGrid, dimBlock, smemSize>>>(
        dev_ptrs->grad,     // Return: interleaved dc_dp values
        dev_ptrs->cond_x,   // Input : condensed dc_dv_x values
        dev_ptrs->cond_y,   // Input : condensed dc_dv_y values
        dev_ptrs->cond_z);  // Input : condensed dc_dv_z values
}
////////////////////////////////////////////////////////////////////////////////


/**
 * Calculates the B-spline score and gradient using CUDA implementation J.
 *
 * @param fixed The fixed volume
 * @param moving The moving volume
 * @param moving_grad The spatial gradient of the moving volume
 * @param bxf Pointer to the B-spline Xform
 * @param parms Pointer to the B-spline parameters
 * @param dev_ptrs Pointer the GPU device pointers
 *
 * @see bspline_cuda_score_j_mse_kernel1()
 * @see CUDA_bspline_mse_condense_64_texfetch()
 * @see CUDA_bspline_mse_reduce()
 *
 * @author James A. Shackleford
 */
extern "C" void
bspline_cuda_j_stage_1 (
    Volume* fixed,
    Volume* moving,
    Volume* moving_grad,
    Bspline_xform* bxf,
    Bspline_parms* parms,
    Dev_Pointers_Bspline* dev_ptrs)
{
#if defined (PROFILE_J)
    hipEvent_t start, stop;
    float time;
#endif

    // Reset our "voxels fallen outside" counter
    hipMemset (dev_ptrs->skipped, 0, dev_ptrs->skipped_size);
    cuda_utils_check_error ("hipMemset(): dev_ptrs->skipped");
    hipMemset (dev_ptrs->score, 0, dev_ptrs->score_size);
    cuda_utils_check_error ("hipMemset(): dev_ptrs->score");


#if defined (PROFILE_J)
    // Start timing the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
#endif


    // Calculate the score and dc_dv
    CUDA_bspline_mse_score_dc_dv (dev_ptrs, bxf, fixed, moving);


#if defined (PROFILE_J)
    // Stop timing the kernel
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);
    hipEventDestroy (start);
    hipEventDestroy (stop);
    printf("[%f ms] score & dc_dv\n", time);
#endif

    // Prepare for the next kernel
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_bspline_g_mse_1");

    // Clear out the condensed dc_dv streams
    hipMemset(dev_ptrs->cond_x, 0, dev_ptrs->cond_x_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->cond_x");
    hipMemset(dev_ptrs->cond_y, 0, dev_ptrs->cond_y_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->cond_y");
    hipMemset(dev_ptrs->cond_z, 0, dev_ptrs->cond_z_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->cond_z");


#if defined (PROFILE_J)
    // Start timing the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
#endif

    // Invoke kernel condense
    int num_tiles = (bxf->cdims[0]-3) * (bxf->cdims[1]-3) * (bxf->cdims[2]-3);
    CUDA_bspline_mse_condense_64_texfetch (dev_ptrs,
                                             bxf->vox_per_rgn, 
                                             num_tiles);

#if defined (PROFILE_J)
    // Stop timing the kernel
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);
    hipEventDestroy (start);
    hipEventDestroy (stop);
    printf("[%f ms] Condense\n", time);
#endif

    // Prepare for the next kernel
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_bspline_mse_condense()");

#if defined (PROFILE_J)
    // Start timing the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
#endif

    // Clear out the gradient
    hipMemset(dev_ptrs->grad, 0, dev_ptrs->grad_size);
    cuda_utils_check_error("hipMemset(): dev_ptrs->grad");

    // Invoke kernel reduce
    CUDA_bspline_mse_reduce (dev_ptrs, bxf->num_knots);

#if defined (PROFILE_J)
    // Stop timing the kernel
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);
    hipEventDestroy (start);
    hipEventDestroy (stop);
    printf("[%f ms] Reduce\n\n", time);
#endif

    // Prepare for the next kernel
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_bspline_mse_condense()");
}



////////////////////////////////////////////////////////////////////////////////
// STUB: bspline_cuda_j_stage_2()
//
// KERNELS INVOKED:
//   sum_reduction_kernel()
//   sum_reduction_last_step_kernel()
//   bspline_cuda_update_grad_kernel()
//   bspline_cuda_compute_grad_mean_kernel()
//   sum_reduction_last_step_kernel()
//   bspline_cuda_compute_grad_norm_kernel
//   sum_reduction_last_step_kernel()
//
// bspline_cuda_final_steps_f()
////////////////////////////////////////////////////////////////////////////////
extern "C" void
bspline_cuda_j_stage_2 (
    Bspline_parms* parms, 
    Bspline_xform* bxf,
    Volume* fixed,
    int*   vox_per_rgn,
    int*   volume_dim,
    float* host_score,
    float* host_grad,
    float* host_grad_mean,
    float* host_grad_norm,
    Dev_Pointers_Bspline* dev_ptrs,
    int *num_vox)
{

#if defined (PROFILE_J)
    hipEvent_t start, stop;
    float time;
#endif


    dim3 dimGrid;
    dim3 dimBlock;

    int num_elems = volume_dim[0] * volume_dim[1] * volume_dim[2];
    int num_blocks = (num_elems + 511) / 512;

    build_exec_conf_1bpe (
        &dimGrid,         // OUTPUT: Grid  dimensions
        &dimBlock,        // OUTPUT: Block dimensions
        num_blocks,       // INPUT: Number of blocks
        512);             // INPUT: Threads per block

    int smemSize = 512*sizeof(float);


#if defined (commentout)
    /* Compute score on cpu for debugging */
    {
    int i;
    float *cpu_score = (float*) malloc (dev_ptrs->score_size);
    int num_ele = dev_ptrs->score_size / sizeof (float);
    double sse = 0.0;
    FILE *fp;

    hipMemcpy (cpu_score, dev_ptrs->score, dev_ptrs->score_size, 
        hipMemcpyDeviceToHost);
    for (i = 0; i < num_ele; i++) {
        sse += (double) cpu_score[i];
    }
    sse /= 128480.;
    printf ("CPU computed score as %f\n", sse);
    
    fp = fopen ("gpu_score.txt", "wb");
    for (i = 0; i < num_ele; i++) {
        fprintf (fp, "%f\n", cpu_score[i]);
    }
    fclose (fp);
    }
#endif


#if defined (PROFILE_J)
    // Start timing the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
#endif

    // --- BEGIN KERNEL EXECUTION -------------------------------
    sum_reduction_kernel<<<dimGrid, dimBlock, smemSize>>> (
        dev_ptrs->score,
        dev_ptrs->score,
        num_elems);
    // ----------------------------------------------------------


    // --- PREPARE FOR NEXT KERNEL ------------------------------
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_sum_reduction()");
    // ----------------------------------------------------------

#if defined (PROFILE_J)
    // Stop timing the kernel
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);
    hipEventDestroy (start);
    hipEventDestroy (stop);
    printf("[%f ms] score reduction\n", time);
#endif

    // --- BEGIN KERNEL EXECUTION -------------------------------
    sum_reduction_last_step_kernel<<<dimGrid, dimBlock>>> (
        dev_ptrs->score,
        dev_ptrs->score,
        num_elems);
    // ----------------------------------------------------------


    // --- PREPARE FOR NEXT KERNEL ------------------------------
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_sum_reduction_last_step()");
    // ----------------------------------------------------------


#if defined (PROFILE_J)
    // Start timing the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
#endif

    // --- RETREIVE THE SCORE FROM GPU --------------------------
    hipMemcpy(host_score, dev_ptrs->score,  sizeof(float), hipMemcpyDeviceToHost);
    cuda_utils_check_error("Failed to copy score from GPU to host");
    // ----------------------------------------------------------


#if defined (PROFILE_J)
    // Stop timing the kernel
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);
    hipEventDestroy (start);
    hipEventDestroy (stop);
    printf("[%f ms] score memcpy\n", time);
#endif



#if defined (PROFILE_J)
    // Start timing the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
#endif



    //  for (i = 1; i < (dev_ptrs->skipped_size / sizeof(int)); i++)
    //      skipped[0] += skipped[i];

    // --- BEGIN KERNEL EXECUTION -------------------------------
    sum_reduction_kernel<<<dimGrid, dimBlock, smemSize>>> (
        dev_ptrs->skipped,
        dev_ptrs->skipped,
        num_elems);
    // ----------------------------------------------------------


    // --- PREPARE FOR NEXT KERNEL ------------------------------
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_sum_reduction()");
    // ----------------------------------------------------------


    // --- BEGIN KERNEL EXECUTION -------------------------------
    sum_reduction_last_step_kernel<<<dimGrid, dimBlock>>> (
        dev_ptrs->skipped,
        dev_ptrs->skipped,
        num_elems);
    // ----------------------------------------------------------

    float skipped;
    hipMemcpy(&skipped, dev_ptrs->skipped, sizeof(float), hipMemcpyDeviceToHost);

    *num_vox = (volume_dim[0] * volume_dim[1] * volume_dim[2]) - skipped;

    *host_score = *host_score / *num_vox;

#if defined (PROFILE_J)
    // Stop timing the kernel
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);
    hipEventDestroy (start);
    hipEventDestroy (stop);
    printf("[%f ms] skipped reduction\n", time);
#endif



    /////////////////////////////////////////////////////////////
    /////////////////////// CALCULATE ///////////////////////////
    ////////////// GRAD, GRAD NORM *AND* GRAD MEAN //////////////
    /////////////////////////////////////////////////////////////


    num_elems = bxf->num_coeff;
    num_blocks = (num_elems + 511) / 512;

    build_exec_conf_1bpe (
        &dimGrid,         // OUTPUT: Grid  dimensions
        &dimBlock,        // OUTPUT: Block dimensions
        num_blocks,       // INPUT: Number of blocks
        512);             // INPUT: Threads per block


#if defined (PROFILE_J)
    // Start timing the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
#endif
    
    // --- BEGIN KERNEL EXECUTION -------------------------------
    bspline_cuda_update_grad_kernel<<<dimGrid, dimBlock>>> (
        dev_ptrs->grad,
        *num_vox,
        num_elems);
    // ----------------------------------------------------------


#if defined (PROFILE_J)
    // Stop timing the kernel
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);
    hipEventDestroy (start);
    hipEventDestroy (stop);
    printf("[%f ms] gradient update\n", time);
#endif

    // --- PREPARE FOR NEXT KERNEL ------------------------------
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] bspline_cuda_update_grad_kernel");
    // ----------------------------------------------------------


#if defined (PROFILE_J)
    // Start timing the kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
#endif

    // --- RETREIVE THE GRAD FROM GPU ---------------------------
    hipMemcpy(host_grad, dev_ptrs->grad, sizeof(float) * bxf->num_coeff, hipMemcpyDeviceToHost);
    cuda_utils_check_error("Failed to copy dev_ptrs->grad to CPU");
    // ----------------------------------------------------------


#if defined (PROFILE_J)
    // Stop timing the kernel
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);
    hipEventDestroy (start);
    hipEventDestroy (stop);
    printf("[%f ms] gradient memcpy\n", time);
#endif


    // The following is unnecessary since report_score()
    // calculates the grad mean & norm from grad[] anyway.
    /*
    // --- BEGIN KERNEL EXECUTION -------------------------------
    bspline_cuda_compute_grad_mean_kernel<<<dimGrid2, dimBlock2, smemSize>>>(
    dev_ptrs->grad,
    dev_ptrs->grad_temp,
    num_elems);
    // ----------------------------------------------------------


    // --- PREPARE FOR NEXT KERNEL ------------------------------
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] bspline_cuda_grad_mean_kernel()");
    // ----------------------------------------------------------


    // --- BEGIN KERNEL EXECUTION -------------------------------
    sum_reduction_last_step_kernel<<<dimGrid2, dimBlock2>>>(
    dev_ptrs->grad_temp,
    dev_ptrs->grad_temp,
    num_elems);
    // ----------------------------------------------------------


    // --- PREPARE FOR NEXT KERNEL ------------------------------
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_sum_reduction_last_step()");
    // ----------------------------------------------------------


    // --- RETREIVE THE GRAD MEAN FROM GPU ----------------------
    hipMemcpy(host_grad_mean, dev_ptrs->grad_temp, sizeof(float), hipMemcpyDeviceToHost);
    cuda_utils_check_error("Failed to copy grad_mean from GPU to host");
    // ----------------------------------------------------------


    // --- BEGIN KERNEL EXECUTION -------------------------------
    bspline_cuda_compute_grad_norm_kernel<<<dimGrid2, dimBlock2, smemSize>>>(
    dev_ptrs->grad,
    dev_ptrs->grad_temp,
    num_elems);
    // ----------------------------------------------------------


    // --- PREPARE FOR NEXT KERNEL ------------------------------
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] bspline_cuda_compute_grad_norm_kernel()");
    // ----------------------------------------------------------


    // --- BEGIN KERNEL EXECUTION -------------------------------
    sum_reduction_last_step_kernel<<<dimGrid2, dimBlock2>>>(
    dev_ptrs->grad_temp,
    dev_ptrs->grad_temp,
    num_elems);
    // ----------------------------------------------------------


    // --- PREPARE FOR NEXT KERNEL ------------------------------
    hipDeviceSynchronize();
    cuda_utils_check_error("[Kernel Panic!] kernel_sum_reduction_last_step()");
    // ----------------------------------------------------------


    // --- RETREIVE THE GRAD NORM FROM GPU ----------------------
    hipMemcpy(host_grad_norm, dev_ptrs->grad_temp, sizeof(float), hipMemcpyDeviceToHost);
    cuda_utils_check_error("Failed to copy grad_norm from GPU to host");
    // ----------------------------------------------------------
    */
}


////////////////////////////////////////////////////////////////////////////////
// Generates many sub-histograms of the moving image
//
//                 --- Neightborhood of 8 ---
//
// NOTE: The main focus of this kernel is to avoid shared memory
//       bank conflicts.
////////////////////////////////////////////////////////////////////////////////
__global__ void
kernel_bspline_MI_a_hist_fix (
    float* f_hist_seg,  // partial histogram (moving image)
    float* f_img,       // moving image voxels
    float offset,       // histogram offset
    float delta,        // histogram delta
    long bins,          // # histogram bins
    int3 vpr,           // voxels per region
    int3 fdim,          // fixed  image dimensions
    int3 mdim,          // moving image dimensions
    int3 rdim,          //       region dimensions
    int3 cdim,          // # control points in x,y,z
    float3 img_origin,  // image origin
    float3 img_spacing, // image spacing
    float3 mov_offset,  // moving image offset
    float3 mov_ps,      // moving image pixel spacing
    int* c_lut,         // DEBUG
    float* q_lut,       // DEBUG
    float* coeff)       // DEBUG
{
    // -- Setup Thread Attributes -----------------------------
    int threadsPerBlock = (blockDim.x * blockDim.y * blockDim.z);

    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
    int thread_idxl     = (((blockDim.y * threadIdx.z) + threadIdx.y) * blockDim.x) + threadIdx.x;
    int thread_idxg     = (blockIdxInGrid * threadsPerBlock) + thread_idxl;
    // --------------------------------------------------------

    // -- Initialize Shared Memory ----------------------------
    // Amount: 32 * # bins
    extern __shared__ float s_Fixed[];

    for (long i=0; i < bins; i++) {
        s_Fixed[threadIdx.x + i*threadsPerBlock] = 0.0f;
    }
    // --------------------------------------------------------


    __syncthreads();


    // -- Only process threads that map to voxels -------------
    if (thread_idxg > fdim.x * fdim.y * fdim.z) {
        return;
    }
    // --------------------------------------------------------

    // --------------------------------------------------------
    int4 q;     // Voxel index (local)
    int4 p;     // Tile index


    float3 f;       // Distance from origin (in mm )
    float3 m;       // Voxel Displacement   (in mm )
    float3 n;       // Voxel Displacement   (in vox)
    float3 d;       // Deformation vector

    int fv;     // fixed voxel
    // --------------------------------------------------------
    
    fv = thread_idxg;

    setup_indices (&p, &q, &f,
            fv, fdim, vpr, rdim, img_origin, img_spacing);


    int fell_out = find_correspondence (&d, &m, &n,
            f, mov_offset, mov_ps, mdim, cdim, vpr, p, q);

    __syncthreads();

    // -- Accumulate Into Segmented Histograms ----------------
    int idx_fbin;
    int f_mem;

    idx_fbin = (int) floorf ((f_img[fv] - offset) * delta);
    f_mem = threadIdx.x + idx_fbin*threadsPerBlock;
    s_Fixed[f_mem] += !fell_out;
    // --------------------------------------------------------

    __syncthreads();

    // -- Merge Segmented Histograms --------------------------
    if (threadIdx.x < bins)
    {
        float sum = 0.0f;

        // Stagger the starting shared memory bank
        // access for each thread so as to prevent
        // bank conflicts, which reasult in half
        // warp difergence / serialization.
        const int startPos = (threadIdx.x & 0x0F);
        const int offset   = threadIdx.x * threadsPerBlock;

        for (int i=0, accumPos = startPos; i < threadsPerBlock; i++) {
            sum += s_Fixed[offset + accumPos];
            if (++accumPos == threadsPerBlock) {
                accumPos = 0;
            }
        }

        f_hist_seg[blockIdxInGrid*bins + threadIdx.x] = sum;

    }
    // --------------------------------------------------------

    // Done.
    // We now have (num_thread_blocks) partial histograms that
    // need to be merged.  This will be done with another
    // kernel to be ran immediately following the completion
    // of this kernel.

    //NOTE:
    // fv = thread_idxg
    // fi = r.x
    // fj = r.y
    // fk = r.z
}



////////////////////////////////////////////////////////////////////////////////
// Generates many sub-histograms of the moving image
//
//                 --- Neightborhood of 8 ---
//
// NOTE: The main focus of this kernel is to avoid shared memory
//       bank conflicts.
////////////////////////////////////////////////////////////////////////////////
__global__ void
kernel_bspline_MI_a_hist_mov (
    float* m_hist_seg,  // partial histogram (moving image)
    float* m_img,       // moving image voxels
    float offset,       // histogram offset
    float delta,        // histogram delta
    long bins,          // # histogram bins
    int3 vpr,           // voxels per region
    int3 fdim,          // fixed  image dimensions
    int3 mdim,          // moving image dimensions
    int3 rdim,          //       region dimensions
    int3 cdim,          // # control points in x,y,z
    float3 img_origin,  // image origin
    float3 img_spacing, // image spacing
    float3 mov_offset,  // moving image offset
    float3 mov_ps,      // moving image pixel spacing
    int* c_lut,         // DEBUG
    float* q_lut,       // DEBUG
    float* coeff)       // DEBUG
{
    // -- Setup Thread Attributes -----------------------------
    int threadsPerBlock = (blockDim.x * blockDim.y * blockDim.z);

    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
    int thread_idxl     = (((blockDim.y * threadIdx.z) + threadIdx.y) * blockDim.x) + threadIdx.x;
    int thread_idxg     = (blockIdxInGrid * threadsPerBlock) + thread_idxl;
    // --------------------------------------------------------

    // -- Initialize Shared Memory ----------------------------
    // Amount: 32 * # bins
    extern __shared__ float s_Moving[];

    for (long i=0; i < bins; i++) {
        s_Moving[threadIdx.x + i*threadsPerBlock] = 0.0f;
    }
    // --------------------------------------------------------


    __syncthreads();


    // -- Only process threads that map to voxels -------------
    if (thread_idxg > fdim.x * fdim.y * fdim.z) {
        return;
    }
    // --------------------------------------------------------


    // --------------------------------------------------------
    int4 q;     // Voxel index (local)
    int4 p;     // Tile index


    float3 f;       // Distance from origin (in mm )
    float3 m;       // Voxel Displacement   (in mm )
    float3 n;       // Voxel Displacement   (in vox)
    int3 n_f;   // Voxel Displacement floor
    int3 n_r;   // Voxel Displacement round
    float3 d;       // Deformation vector

    int fv;     // fixed voxel
    int mvf;        // moving voxel (floor)
    // --------------------------------------------------------
    
    fv = thread_idxg;

    setup_indices (&p, &q, &f,
            fv, fdim, vpr, rdim, img_origin, img_spacing);


    int fell_out = find_correspondence (&d, &m, &n,
            f, mov_offset, mov_ps, mdim, cdim, vpr, p, q);


    if (!fell_out) {
        float3 li_1, li_2;
        clamp_linear_interpolate_3d (&n, &n_f, &n_r, &li_1, &li_2, mdim);

        // -- Compute coordinates of 8 nearest neighbors ----------
        int n1, n2, n3, n4;
        int n5, n6, n7, n8;
    
        mvf = (n_f.z * mdim.y + n_f.y) * mdim.x + n_f.x;

        n1 = mvf;
        n2 = n1 + 1;
        n3 = n1 + mdim.x;
        n4 = n1 + mdim.x + 1;
        n5 = n1 + mdim.x * mdim.y;
        n6 = n1 + mdim.x * mdim.y + 1;
        n7 = n1 + mdim.x * mdim.y + mdim.x;
        n8 = n1 + mdim.x * mdim.y + mdim.x + 1;
        // --------------------------------------------------------


        // -- Compute differential PV slices ----------------------
        float w1, w2, w3, w4;
        float w5, w6, w7, w8;

        w1 = li_1.x * li_1.y * li_1.z;
        w2 = li_2.x * li_1.y * li_1.z;
        w3 = li_1.x * li_2.y * li_1.z;
        w4 = li_2.x * li_2.y * li_1.z;
        w5 = li_1.x * li_1.y * li_2.z;
        w6 = li_2.x * li_1.y * li_2.z;
        w7 = li_1.x * li_2.y * li_2.z;
        w8 = li_2.x * li_2.y * li_2.z;
        // --------------------------------------------------------


        __syncthreads();

        // -- Accumulate Into Segmented Histograms ----------------
        int idx_mbin;
        int m_mem;

        // PV 1
        idx_mbin = (int) floorf ((m_img[n1] - offset) * delta);
        m_mem = threadIdx.x + idx_mbin*threadsPerBlock;
        s_Moving[m_mem] += w1;

        // PV 2
        idx_mbin = (int) floorf ((m_img[n2] - offset) * delta);
        m_mem = threadIdx.x + idx_mbin*threadsPerBlock;
        s_Moving[m_mem] += w2;

        // PV 3
        idx_mbin = (int) floorf ((m_img[n3] - offset) * delta);
        m_mem = threadIdx.x + idx_mbin*threadsPerBlock;
        s_Moving[m_mem] += w3;

        // PV 4
        idx_mbin = (int) floorf ((m_img[n4] - offset) * delta);
        m_mem = threadIdx.x + idx_mbin*threadsPerBlock;
        s_Moving[m_mem] += w4;

        // PV 5
        idx_mbin = (int) floorf ((m_img[n5] - offset) * delta);
        m_mem = threadIdx.x + idx_mbin*threadsPerBlock;
        s_Moving[m_mem] += w5;

        // PV 6
        idx_mbin = (int) floorf ((m_img[n6] - offset) * delta);
        m_mem = threadIdx.x + idx_mbin*threadsPerBlock;
        s_Moving[m_mem] += w6;

        // PV 7
        idx_mbin = (int) floorf ((m_img[n7] - offset) * delta);
        m_mem = threadIdx.x + idx_mbin*threadsPerBlock;
        s_Moving[m_mem] += w7;

        // PV 8
        idx_mbin = (int) floorf ((m_img[n8] - offset) * delta);
        m_mem = threadIdx.x + idx_mbin*threadsPerBlock;
        s_Moving[m_mem] += w8;
        // --------------------------------------------------------
    }

    __syncthreads();

    // -- Merge Segmented Histograms --------------------------
    if (threadIdx.x < bins)
    {
        float sum = 0.0f;

        // Stagger the starting shared memory bank
        // access for each thread so as to prevent
        // bank conflicts, which reasult in half
        // warp difergence / serialization.
        const int startPos = (threadIdx.x & 0x0F);
        const int offset   = threadIdx.x * threadsPerBlock;

        for (int i=0, accumPos = startPos; i < threadsPerBlock; i++) {
            sum += s_Moving[offset + accumPos];
            if (++accumPos == threadsPerBlock) {
                accumPos = 0;
            }
        }

        m_hist_seg[blockIdxInGrid*bins + threadIdx.x] = sum;

    }
    // --------------------------------------------------------

    // Done.
    // We now have (num_thread_blocks) partial histograms that
    // need to be merged.  This will be done with another
    // kernel to be ran immediately following the completion
    // of this kernel.

    //NOTE:
    // fv = thread_idxg
    // fi = r.x
    // fj = r.y
    // fk = r.z
}




////////////////////////////////////////////////////////////////////////////////
// Generates the joint histogram
//
//                 --- Neightborhood of 8 ---
//
////////////////////////////////////////////////////////////////////////////////
__global__ void
kernel_bspline_MI_a_hist_jnt (
    unsigned int* skipped,   // OUTPUT:   # of skipped voxels
    float* j_hist,      // OUTPUT:  joint histogram
    float* f_img,   // INPUT:  fixed image voxels
    float* m_img,   // INPUT: moving image voxels
    float f_offset, // INPUT:  fixed histogram offset 
    float m_offset, // INPUT: moving histogram offset
    float f_delta,  // INPUT:  fixed histogram delta
    float m_delta,  // INPUT: moving histogram delta
    long f_bins,        // INPUT: #  fixed histogram bins
    long m_bins,    // INPUT: # moving histogram bins
    int3 vpr,       // INPUT: voxels per region
    int3 fdim,      // INPUT:  fixed image dimensions
    int3 mdim,      // INPUT: moving image dimensions
    int3 rdim,      // INPUT: region dimensions
    int3 cdim,          // # control points in x,y,z
    float3 img_origin,  // INPUT: image origin
    float3 img_spacing, // INPUT: image spacing
    float3 mov_offset,  // INPUT: moving image offset
    float3 mov_ps,  // INPUT: moving image pixel spacing
    int3 roi_dim,   // INPUT: ROI dimensions
    int3 roi_offset,    // INPUT: ROI Offset
    int* c_lut,     // INPUT: coefficient lut
    float* q_lut,   // INPUT: bspline product lut
    float* coeff)   // INPUT: coefficient array
{
/* This code requires compute capability 1.2 or greater.
 * DO NOT compile it for lesser target architectures or
 * nvcc will complain and stop the build; thus the #if
 */
#if defined (__CUDA_ARCH__) && __CUDA_ARCH__ >= 120

    // -- Setup Thread Attributes -----------------------------
    int threadsPerBlock = (blockDim.x * blockDim.y * blockDim.z);

    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
    int thread_idxl     = (((blockDim.y * threadIdx.z) + threadIdx.y) * blockDim.x) + threadIdx.x;
    int thread_idxg     = (blockIdxInGrid * threadsPerBlock) + thread_idxl;
    // --------------------------------------------------------

    // -- Initial shared memory for locks ---------------------
    extern __shared__ float shared_mem[]; 

    float* j_locks = (float*)shared_mem;
    int total_smem = f_bins * m_bins;

    int b = (total_smem + threadsPerBlock - 1) / threadsPerBlock;

    int i;
    for (i = 0; i < b; i++) {
        if ( (thread_idxl + i*threadsPerBlock) < total_smem ) {
            shared_mem[thread_idxl + i*threadsPerBlock] = 0.0f;
        }
    }
    // --------------------------------------------------------


    // -- Only process threads that map to voxels -------------
    if (thread_idxg > fdim.x * fdim.y * fdim.z) {
        return;
    }
    // --------------------------------------------------------


    // --------------------------------------------------------
    int4 q;     // Voxel index (local)
    int4 p;     // Tile index


    float3 f;       // Distance from origin (in mm )
    float3 m;       // Voxel Displacement   (in mm )
    float3 n;       // Voxel Displacement   (in vox)
    float3 d;       // Deformation vector

    int3 n_f;       // Voxel Displacement floor
    int3 n_r;       // Voxel Displacement round

    int fv;     // fixed voxel
    int mvf;        // moving voxel (floor)
    // --------------------------------------------------------
    
    fv = thread_idxg;

    setup_indices (&p, &q, &f,
            fv, fdim, vpr, rdim, img_origin, img_spacing);


    int fell_out = find_correspondence (&d, &m, &n,
            f, mov_offset, mov_ps, mdim, cdim, vpr, p, q);


    if (fell_out) {
        // Voxel doesn't map into the moving image.
        // Don't bin anything and count the miss.
        atomicAdd (skipped, 1);
        return;
    }

    float3 li_1, li_2;
    clamp_linear_interpolate_3d (&n, &n_f, &n_r, &li_1, &li_2, mdim);

    // -- Compute coordinates of 8 nearest neighbors ----------
    int n1, n2, n3, n4;
    int n5, n6, n7, n8;
    
    mvf = (n_f.z * mdim.y + n_f.y) * mdim.x + n_f.x;

    n1 = mvf;
    n2 = n1 + 1;
    n3 = n1 + mdim.x;
    n4 = n1 + mdim.x + 1;
    n5 = n1 + mdim.x * mdim.y;
    n6 = n1 + mdim.x * mdim.y + 1;
    n7 = n1 + mdim.x * mdim.y + mdim.x;
    n8 = n1 + mdim.x * mdim.y + mdim.x + 1;
    // --------------------------------------------------------


    // -- Compute differential PV slices ----------------------
    float w1, w2, w3, w4;
    float w5, w6, w7, w8;

    w1 = li_1.x * li_1.y * li_1.z;
    w2 = li_2.x * li_1.y * li_1.z;
    w3 = li_1.x * li_2.y * li_1.z;
    w4 = li_2.x * li_2.y * li_1.z;
    w5 = li_1.x * li_1.y * li_2.z;
    w6 = li_2.x * li_1.y * li_2.z;
    w7 = li_1.x * li_2.y * li_2.z;
    w8 = li_2.x * li_2.y * li_2.z;
    // --------------------------------------------------------

    __syncthreads();

    // -- Read from histograms and compute dC/dp_j * dp_j/dv --
    bool success;
    int idx_fbin, offset_fbin;
    int idx_mbin;
    int idx_jbin;
    int j_mem;
    long j_bins = f_bins * m_bins;

    long j_stride = blockIdxInGrid * j_bins;

    // Calculate fixed bin offset into joint
    idx_fbin = (int) floorf ((f_img[fv] - f_offset) * f_delta);
    offset_fbin = idx_fbin * m_bins;

    // Add PV w1 to moving & joint histograms
    idx_mbin = (int) floorf ((m_img[n1] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (idx_jbin != 0) {
        success = false;
        j_mem = j_stride + idx_jbin;
        while (!success) {
            if (atomicExch(&j_locks[idx_jbin], 1.0f) == 0.0f) {
               success = true;
               j_hist[j_mem] += w1;
               atomicExch(&j_locks[idx_jbin], 0.0f);
            }
            __threadfence();
        }
    }

    // Add PV w2 to moving & joint histograms
    idx_mbin = (int) floorf ((m_img[n2] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (idx_jbin != 0) {
        success = false;
        j_mem = j_stride + idx_jbin;
        while (!success) {
            if (atomicExch(&j_locks[idx_jbin], 1.0f) == 0.0f) {
               success = true;
               j_hist[j_mem] += w2;
               atomicExch(&j_locks[idx_jbin], 0.0f);
            }
            __threadfence();
        }
    }

    // Add PV w3 to moving & joint histograms
    idx_mbin = (int) floorf ((m_img[n3] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (idx_jbin != 0) {
        success = false;
        j_mem = j_stride + idx_jbin;
        while (!success) {
            if (atomicExch(&j_locks[idx_jbin], 1.0f) == 0.0f) {
               success = true;
               j_hist[j_mem] += w3;
               atomicExch(&j_locks[idx_jbin], 0.0f);
            }
            __threadfence();
        }
    }

    // Add PV w4 to moving & joint histograms
    idx_mbin = (int) floorf ((m_img[n4] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    success = false;
    j_mem = j_stride + idx_jbin;
    if (idx_jbin != 0) {
        while (!success) {
            if (atomicExch(&j_locks[idx_jbin], 1.0f) == 0.0f) {
               success = true;
               j_hist[j_mem] += w4;
               atomicExch(&j_locks[idx_jbin], 0.0f);
            }
            __threadfence();
        }
    }

    // Add PV w5 to moving & joint histograms
    idx_mbin = (int) floorf ((m_img[n5] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    success = false;
    j_mem = j_stride + idx_jbin;
    if (idx_jbin != 0) {
        while (!success) {
            if (atomicExch(&j_locks[idx_jbin], 1.0f) == 0.0f) {
               success = true;
               j_hist[j_mem] += w5;
               atomicExch(&j_locks[idx_jbin], 0.0f);
            }
            __threadfence();
        }
    }

    // Add PV w6 to moving & joint histograms
    idx_mbin = (int) floorf ((m_img[n6] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    success = false;
    j_mem = j_stride + idx_jbin;
    if (idx_jbin != 0) {
        while (!success) {
            if (atomicExch(&j_locks[idx_jbin], 1.0f) == 0.0f) {
               success = true;
               j_hist[j_mem] += w6;
               atomicExch(&j_locks[idx_jbin], 0.0f);
            }
            __threadfence();
        }
    }

    // Add PV w7 to moving & joint histograms
    idx_mbin = (int) floorf ((m_img[n7] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    success = false;
    j_mem = j_stride + idx_jbin;
    if (idx_jbin != 0) {
        while (!success) {
            if (atomicExch(&j_locks[idx_jbin], 1.0f) == 0.0f) {
               success = true;
               j_hist[j_mem] += w7;
               atomicExch(&j_locks[idx_jbin], 0.0f);
            }
            __threadfence();
        }
    }

    // Add PV w8 to moving & joint histograms
    idx_mbin = (int) floorf ((m_img[n8] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    success = false;
    j_mem = j_stride + idx_jbin;
    if (idx_jbin != 0) {
        while (!success) {
            if (atomicExch(&j_locks[idx_jbin], 1.0f) == 0.0f) {
               success = true;
               j_hist[j_mem] += w8;
               atomicExch(&j_locks[idx_jbin], 0.0f);
            }
            __threadfence();
        }
    }
    // --------------------------------------------------------

#endif // __CUDA_ARCH__
}


////////////////////////////////////////////////////////////////////////////////
// Merge Partial/Segmented Histograms
//
//   This kernel is designed to be executed after k_bspline_cuda_MI_a_hist_fix 
//   has genereated many partial histograms (equal to the number of thread-
//   blocks k_bspline_cuda_MI_a_hist_fix() was executed with).  Depending on
//   the image size, this could be as high as hundredes of thousands of
//   partial histograms needing to be merged.
//
//   >> Each thread-block is responsible for a bin number.
//
//   >> A thread-block will use multiple threads to pull down
//      multiple partial histogram bin values in parallel.
//
//   >> Because there are many more partial histograms than threads,
//      the threads in a thread-block will have to iterate through
//      all of the partial histograms using a for-loop.
//
//   >> The # of for-loop iterations is equal to the number of
//      partial histograms divided by the number of threads in a block.
//
//   >> Therefore, this kernel should be launched with:
//
//      -- num_seg_hist % num_threads = 0     (num_seg_hist % blockDim.x = 0)
//      -- num_blocks = num_bins
//
//   >> This means that a search must be executed to find the largest #
//      of threads that can fit within the number of partial histograms
//      we have.  This will exhibit the largest amount of parallelism.
//
////////////////////////////////////////////////////////////////////////////////
__global__ void
kernel_bspline_MI_a_hist_fix_merge (
    float *f_hist,
    float *f_hist_seg,
    long num_seg_hist)

{
    extern __shared__ float data[];

    float sum = 0.0f;

    // -- Work through all the sub-histograms ------------------------
    for (long i = threadIdx.x; i < num_seg_hist; i += blockDim.x) {
        sum += f_hist_seg[blockIdx.x + i * gridDim.x];
    }

    data[threadIdx.x] = sum;
    // ---------------------------------------------------------------

    __syncthreads();

    // -- Sum all of the thread sums for this bin --------------------
    for (long s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            data[threadIdx.x] += data[threadIdx.x + s];
        }

        __syncthreads();
    }
    // ---------------------------------------------------------------


    // -- Write the final bin value to Global ------------------------
    if (threadIdx.x == 0) {
        f_hist[blockIdx.x] = data[0];
    }
    // ---------------------------------------------------------------

    // Done.
}



////////////////////////////////////////////////////////////////////////////////
// Computes dC/dv for MI using PVI-8 interpolation
//
//                 --- Neightborhood of 8 ---
//
////////////////////////////////////////////////////////////////////////////////
__global__ void
kernel_bspline_MI_dc_dv_a (
    float* dc_dv_x,     // OUTPUT: dC / dv (x-component)
    float* dc_dv_y,     // OUTPUT: dC / dv (y-component)
    float* dc_dv_z,     // OUTPUT: dC / dv (z-component)
    float* f_hist,      // INPUT:  fixed histogram
    float* m_hist,      // INPUT: moving histogram
    float* j_hist,      // INPUT:  joint histogram
    float* f_img,       // INPUT:  fixed image voxels
    float* m_img,       // INPUT: moving image voxels
    float f_offset,     // INPUT:  fixed histogram offset 
    float m_offset,     // INPUT: moving histogram offset
    float f_delta,      // INPUT:  fixed histogram delta
    float m_delta,      // INPUT: moving histogram delta
    long f_bins,        // INPUT: #  fixed histogram bins
    long m_bins,        // INPUT: # moving histogram bins
    int3 vpr,           // INPUT: voxels per region
    int3 fdim,          // INPUT:  fixed image dimensions
    int3 mdim,          // INPUT: moving image dimensions
    int3 rdim,          // INPUT: region dimensions
    int3 cdim,          // # control points in x,y,z
    float3 img_origin,  // INPUT: image origin
    float3 img_spacing, // INPUT: image spacing
    float3 mov_offset,  // INPUT: moving image offset
    float3 mov_ps,      // INPUT: moving image pixel spacing
    int3 roi_dim,       // INPUT: ROI dimensions
    int3 roi_offset,    // INPUT: ROI Offset
    int* c_lut,         // INPUT: coefficient lut
    float* q_lut,       // INPUT: bspline product lut
    float* coeff,       // INPUT: coefficient array
    float num_vox_f,    // INPUT: # of voxels
    float score,        // INPUT: evaluated MI cost function
    int pad)            // INPUT: Tile padding
{
    // -- Setup Thread Attributes -----------------------------
    int threadsPerBlock = (blockDim.x * blockDim.y * blockDim.z);

    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
    int thread_idxl     = (((blockDim.y * threadIdx.z) + threadIdx.y) * blockDim.x) + threadIdx.x;
    int thread_idxg     = (blockIdxInGrid * threadsPerBlock) + thread_idxl;
    // --------------------------------------------------------

    
    // -- Only process threads that map to voxels -------------
    if (thread_idxg > fdim.x * fdim.y * fdim.z) {
        return;
    }
    // --------------------------------------------------------


    // -- Variables used by correspondence --------------------
    // -- (Block verified) ------------------------------------
    int3 r;     // Voxel index (global)
    int4 q;     // Voxel index (local)
    int4 p;     // Tile index


    float3 f;       // Distance from origin (in mm )
    float3 m;       // Voxel Displacement   (in mm )
    float3 n;       // Voxel Displacement   (in vox)
    float3 d;       // Deformation vector

    int3 n_f;       // Voxel Displacement floor
    int3 n_r;       // Voxel Displacement round

    int fv;     // fixed voxel
    int mvf;        // moving voxel (floor)
    // --------------------------------------------------------
    
    fv = thread_idxg;

    r.z = fv / (fdim.x * fdim.y);
    r.y = (fv - (r.z * fdim.x * fdim.y)) / fdim.x;
    r.x = fv - r.z * fdim.x * fdim.y - (r.y * fdim.x);
    
    setup_indices (&p, &q, &f,
            fv, fdim, vpr, rdim, img_origin, img_spacing);

    if (r.x > (roi_offset.x + roi_dim.x) ||
        r.y > (roi_offset.y + roi_dim.y) ||
        r.z > (roi_offset.z + roi_dim.z))
    {
        return;
    }

    int fell_out = find_correspondence (&d, &m, &n,
            f, mov_offset, mov_ps, mdim, cdim, vpr, p, q);

    if (fell_out) {
        return;
    }


    float3 li_1, li_2;
    clamp_linear_interpolate_3d (&n, &n_f, &n_r, &li_1, &li_2, mdim);

    // -- Compute coordinates of 8 nearest neighbors ----------
    int n1, n2, n3, n4;
    int n5, n6, n7, n8;

    mvf = (n_f.z * mdim.y + n_f.y) * mdim.x + n_f.x;

    n1 = mvf;
    n2 = n1 + 1;
    n3 = n1 + mdim.x;
    n4 = n1 + mdim.x + 1;
    n5 = n1 + mdim.x * mdim.y;
    n6 = n1 + mdim.x * mdim.y + 1;
    n7 = n1 + mdim.x * mdim.y + mdim.x;
    n8 = n1 + mdim.x * mdim.y + mdim.x + 1;
    // --------------------------------------------------------


    // -- Compute differential PV slices ----------------------
    float3 dw1, dw2, dw3, dw4;
    float3 dw5, dw6, dw7, dw8;

    dw1.x =  -1.0f * li_1.y * li_1.z;
    dw1.y = li_1.x *  -1.0f * li_1.z;
    dw1.z = li_1.x * li_1.y *  -1.0f;

    dw2.x =  +1.0f * li_1.y * li_1.z;
    dw2.y = li_2.x *  -1.0f * li_1.z;
    dw2.z = li_2.x * li_1.y *  -1.0f;

    dw3.x =  -1.0f * li_2.y * li_1.z;
    dw3.y = li_1.x *  +1.0f * li_1.z;
    dw3.z = li_1.x * li_2.y *  -1.0f;

    dw4.x =  +1.0f * li_2.y * li_1.z;
    dw4.y = li_2.x *  +1.0f * li_1.z;
    dw4.z = li_2.x * li_2.y *  -1.0f;

    dw5.x =  -1.0f * li_1.y * li_2.z;
    dw5.y = li_1.x *  -1.0f * li_2.z;
    dw5.z = li_1.x * li_1.y *  +1.0f;

    dw6.x =  +1.0f * li_1.y * li_2.z;
    dw6.y = li_2.x *  -1.0f * li_2.z;
    dw6.z = li_2.x * li_1.y *  +1.0f;

    dw7.x =  -1.0f * li_2.y * li_2.z;
    dw7.y = li_1.x *  +1.0f * li_2.z;
    dw7.z = li_1.x * li_2.y *  +1.0f;

    dw8.x =  +1.0f * li_2.y * li_2.z;
    dw8.y = li_2.x *  +1.0f * li_2.z;
    dw8.z = li_2.x * li_2.y *  +1.0f;
    // --------------------------------------------------------

    __syncthreads();

    // -- Read from histograms and compute dC/dp_j * dp_j/dv --
    float dS_dP;
    float3 dc_dv;
    int idx_fbin, offset_fbin;
    int idx_mbin;
    int idx_jbin;

    float ht = 0.000001f;

    dc_dv.x = 0.0f;
    dc_dv.y = 0.0f;
    dc_dv.z = 0.0f;


    idx_fbin = (int) floorf ((f_img[fv] - f_offset) * f_delta);
    offset_fbin = idx_fbin * m_bins;

    // PV w1
    idx_mbin = (int) floorf ((m_img[n1] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (j_hist[idx_jbin] > ht && f_hist[idx_fbin] > ht && m_hist[idx_mbin] > ht) {
        dS_dP = logf((num_vox_f * j_hist[idx_jbin]) / (f_hist[idx_fbin] * m_hist[idx_mbin])) - score;
        dc_dv.x -= dw1.x * dS_dP;
        dc_dv.y -= dw1.y * dS_dP;
        dc_dv.z -= dw1.z * dS_dP;
    }

    // PV w2
    idx_mbin = (int) floorf ((m_img[n2] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (j_hist[idx_jbin] > ht && f_hist[idx_fbin] > ht && m_hist[idx_mbin] > ht) {
        dS_dP = logf((num_vox_f * j_hist[idx_jbin]) / (f_hist[idx_fbin] * m_hist[idx_mbin])) - score;
        dc_dv.x -= dw2.x * dS_dP;
        dc_dv.y -= dw2.y * dS_dP;
        dc_dv.z -= dw2.z * dS_dP;
    }

    // PV w3
    idx_mbin = (int) floorf ((m_img[n3] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (j_hist[idx_jbin] > ht && f_hist[idx_fbin] > ht && m_hist[idx_mbin] > ht) {
        dS_dP = logf((num_vox_f * j_hist[idx_jbin]) / (f_hist[idx_fbin] * m_hist[idx_mbin])) - score;
        dc_dv.x -= dw3.x * dS_dP;
        dc_dv.y -= dw3.y * dS_dP;
        dc_dv.z -= dw3.z * dS_dP;
    }

    // PV w4
    idx_mbin = (int) floorf ((m_img[n4] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (j_hist[idx_jbin] > ht && f_hist[idx_fbin] > ht && m_hist[idx_mbin] > ht) {
        dS_dP = logf((num_vox_f * j_hist[idx_jbin]) / (f_hist[idx_fbin] * m_hist[idx_mbin])) - score;
        dc_dv.x -= dw4.x * dS_dP;
        dc_dv.y -= dw4.y * dS_dP;
        dc_dv.z -= dw4.z * dS_dP;
    }

    // PV w5
    idx_mbin = (int) floorf ((m_img[n5] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (j_hist[idx_jbin] > ht && f_hist[idx_fbin] > ht && m_hist[idx_mbin] > ht) {
        dS_dP = logf((num_vox_f * j_hist[idx_jbin]) / (f_hist[idx_fbin] * m_hist[idx_mbin])) - score;
        dc_dv.x -= dw5.x * dS_dP;
        dc_dv.y -= dw5.y * dS_dP;
        dc_dv.z -= dw5.z * dS_dP;
    }

    // PV w6
    idx_mbin = (int) floorf ((m_img[n6] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (j_hist[idx_jbin] > ht && f_hist[idx_fbin] > ht && m_hist[idx_mbin] > ht) {
        dS_dP = logf((num_vox_f * j_hist[idx_jbin]) / (f_hist[idx_fbin] * m_hist[idx_mbin])) - score;
        dc_dv.x -= dw6.x * dS_dP;
        dc_dv.y -= dw6.y * dS_dP;
        dc_dv.z -= dw6.z * dS_dP;
    }

    // PV w7
    idx_mbin = (int) floorf ((m_img[n7] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (j_hist[idx_jbin] > ht && f_hist[idx_fbin] > ht && m_hist[idx_mbin] > ht) {
        dS_dP = logf((num_vox_f * j_hist[idx_jbin]) / (f_hist[idx_fbin] * m_hist[idx_mbin])) - score;
        dc_dv.x -= dw7.x * dS_dP;
        dc_dv.y -= dw7.y * dS_dP;
        dc_dv.z -= dw7.z * dS_dP;
    }

    // PV w8
    idx_mbin = (int) floorf ((m_img[n8] - m_offset) * m_delta);
    idx_jbin = offset_fbin + idx_mbin;
    if (j_hist[idx_jbin] > ht && f_hist[idx_fbin] > ht && m_hist[idx_mbin] > ht) {
        dS_dP = logf((num_vox_f * j_hist[idx_jbin]) / (f_hist[idx_fbin] * m_hist[idx_mbin])) - score;
        dc_dv.x -= dw8.x * dS_dP;
        dc_dv.y -= dw8.y * dS_dP;
        dc_dv.z -= dw8.z * dS_dP;
    }
    // --------------------------------------------------------


    // -- Convert from voxels to mm ---------------------------
    dc_dv.x = dc_dv.x / mov_ps.x / num_vox_f;
    dc_dv.y = dc_dv.y / mov_ps.y / num_vox_f;
    dc_dv.z = dc_dv.z / mov_ps.z / num_vox_f;
    // --------------------------------------------------------

    __syncthreads();


    // -- Finally, write out dc_dv ----------------------------
    float* dc_dv_element_x;
    float* dc_dv_element_y;
    float* dc_dv_element_z;

    dc_dv_element_x = &dc_dv_x[((vpr.x * vpr.y * vpr.z) + pad) * p.w];
    dc_dv_element_y = &dc_dv_y[((vpr.x * vpr.y * vpr.z) + pad) * p.w];
    dc_dv_element_z = &dc_dv_z[((vpr.x * vpr.y * vpr.z) + pad) * p.w];

    dc_dv_element_x = &dc_dv_element_x[q.w];
    dc_dv_element_y = &dc_dv_element_y[q.w];
    dc_dv_element_z = &dc_dv_element_z[q.w];

    dc_dv_element_x[0] = dc_dv.x;
    dc_dv_element_y[0] = dc_dv.y;
    dc_dv_element_z[0] = dc_dv.z;
    // --------------------------------------------------------


    //NOTE:
    // fv = thread_idxg
    // fi = r.x
    // fj = r.y
    // fk = r.z
}


/**
 * This kernel partially computes the gradient by generating condensed dc_dv values.
 *
 * @warning It is required that input data tiles be aligned to 64 byte boundaries.
 *
 * @see CUDA_pad_64()
 * @see kernel_pad_64()
 *
 * @param cond_x Pointer to condensed dc_dv x-values
 * @param cond_y Pointer to condensed dc_dv y-values
 * @param cond_z Pointer to condensed dc_dv z-values
 * @param dc_dv_x Pointer to dc_dv x-values
 * @param dc_dv_y Pointer to dc_dv y-values
 * @param dc_dv_z Pointer to dc_dv z-values
 * @param LUT_Tile_Offsets Pointer to offset lookup table
 * @param LUT_Knot Pointer to linear knot indices
 * @param pad Amount of tile padding, in bytes
 * @param tile_dim Dimensions of input volume tiles
 * @param one_over_six The value 1/6
 *
 * @author: James A. Shackleford
 */
__global__ void
kernel_bspline_mse_condense_64_texfetch (
    float* cond_x,      // Return: condensed dc_dv_x values
    float* cond_y,      // Return: condensed dc_dv_y values
    float* cond_z,      // Return: condensed dc_dv_z values
    float* dc_dv_x,     // Input : dc_dv_x values
    float* dc_dv_y,     // Input : dc_dv_y values
    float* dc_dv_z,     // Input : dc_dv_z values
    int* LUT_Tile_Offsets,  // Input : tile offsets
    int* LUT_Knot,      // Input : linear knot indicies
    int pad,            // Input : amount of tile padding
    int4 tile_dim,      // Input : dims of tiles
    float one_over_six)     // Input : Precomputed since GPU division is slow
{
    int tileOffset;
    int voxel_cluster;
    int voxel_idx;
    float3 voxel_val;
    int3 voxel_loc;
    int4 tile_pos;
    float A,B,C;


    // -- Setup Thread Attributes -----------------------------
    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
    // --------------------------------------------------------


    // -- Setup Shared Memory ---------------------------------
    // -- SIZE: 9*64*sizeof(float)
    // --------------------------------------------------------
    extern __shared__ float sdata[]; 
    float* sBuffer_x = (float*)sdata;           // sBuffer_x[64]
    float* sBuffer_y = (float*)&sBuffer_x[64];      // sBuffer_y[64]
    float* sBuffer_z = (float*)&sBuffer_y[64];      // sBuffer_z[64]
    float* sBuffer_redux_x = (float*)&sBuffer_z[64];    // sBuffer_redux_x[64]
    float* sBuffer_redux_y = (float*)&sBuffer_redux_x[64];  // sBuffer_redux_y[64]
    float* sBuffer_redux_z = (float*)&sBuffer_redux_y[64];  // sBuffer_redux_z[64]
    float* sBuffer_redux_x2 = (float*)&sBuffer_redux_z[64]; // sBuffer_redux_x2[64]
    float* sBuffer_redux_y2 = (float*)&sBuffer_redux_x2[64];// sBuffer_redux_y2[64]
    float* sBuffer_redux_z2 = (float*)&sBuffer_redux_y2[64];// sBuffer_redux_z2[64]
    // --------------------------------------------------------


    // Clear Shared Memory!!
    sBuffer_x[threadIdx.x] = 0.0f;
    sBuffer_y[threadIdx.x] = 0.0f;
    sBuffer_z[threadIdx.x] = 0.0f;


    // First, get the offset of where our tile starts in memory.
    tileOffset = LUT_Tile_Offsets[blockIdxInGrid];

    // Main Loop for Warp Work
    // (Here we condense a tile into 64x3 floats)
    for (voxel_cluster=0; voxel_cluster < tile_dim.w; voxel_cluster+=64)
    {

    // ----------------------------------------------------------
    //                  STAGE 1 IN POWERPOINT
    // ----------------------------------------------------------
    // Second, we pulldown the current voxel cluster.
    // Each thread in the warp pulls down 1 voxel (3 values)
    // ----------------------------------------------------------
    voxel_val.x = dc_dv_x[tileOffset + voxel_cluster + threadIdx.x];
    voxel_val.y = dc_dv_y[tileOffset + voxel_cluster + threadIdx.x];
    voxel_val.z = dc_dv_z[tileOffset + voxel_cluster + threadIdx.x];
    // ----------------------------------------------------------

    // Third, find the [x,y,z] location within the current tile
    // for the voxel this thread is processing.
    voxel_idx = (voxel_cluster + threadIdx.x);
    voxel_loc.z = voxel_idx / (tile_dim.x * tile_dim.y);
    voxel_loc.y = (voxel_idx - (voxel_loc.z * tile_dim.x * tile_dim.y)) / tile_dim.x;
    voxel_loc.x = voxel_idx - voxel_loc.z * tile_dim.x * tile_dim.y - (voxel_loc.y * tile_dim.x);

    // Fourth, we will perform all 64x3 calculations on the current voxel cluster.
    // (Every thead in the warp will be doing this at the same time for its voxel)

    tile_pos.w = 0; // Current tile position within [0,63]

    for (tile_pos.z = 0; tile_pos.z < 4; tile_pos.z++)
    {
        C = tex1Dfetch(tex_LUT_Bspline_z, tile_pos.z * tile_dim.z + voxel_loc.z);
        for (tile_pos.y = 0; tile_pos.y < 4; tile_pos.y++)
        {
        B = C * tex1Dfetch(tex_LUT_Bspline_y, tile_pos.y * tile_dim.y + voxel_loc.y);
        tile_pos.x = 0;

        // #### FIRST HALF ####

        // ---------------------------------------------------------------------------------
        // Do the 1st two x-positions out of four using our two
        // blocks of shared memory for reduction

        // Calculate the b-spline multiplier for this voxel @ this tile
        // position relative to a given control knot.
        // ---------------------------------------------------------------------------------
        A = B * tex1Dfetch(tex_LUT_Bspline_x, tile_pos.x * tile_dim.x + voxel_loc.x);

        // Perform the multiplication and store to redux shared memory
        sBuffer_redux_x[threadIdx.x] = voxel_val.x * A;
        sBuffer_redux_y[threadIdx.x] = voxel_val.y * A;
        sBuffer_redux_z[threadIdx.x] = voxel_val.z * A;
        tile_pos.x++;

        // Calculate the b-spline multiplier for this voxel @ the next tile
        // position relative to a given control knot.
        A = B * tex1Dfetch(tex_LUT_Bspline_x, tile_pos.x * tile_dim.x + voxel_loc.x);

        // Perform the multiplication and store to redux shared memory
        // for the second position
        sBuffer_redux_x2[threadIdx.x] = voxel_val.x * A;
        sBuffer_redux_y2[threadIdx.x] = voxel_val.y * A;
        sBuffer_redux_z2[threadIdx.x] = voxel_val.z * A;
        __syncthreads();
        // ---------------------------------------------------------------------------------


        // ---------------------------------------------------------------------------------
        // All 64 dc_dv values in the current cluster have been processed
        // for the current 2 tile positions (out of 64 total tile positions).
                
        // We now perform a sum reduction on these 64 dc_dv values to
        // condense the data down to one value.
        // ---------------------------------------------------------------------------------
        if (threadIdx.x < 32)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 32];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 32];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 32];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 32];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 32];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 32];
        }
        __syncthreads();

        if (threadIdx.x < 16)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 16];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 16];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 16];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 16];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 16];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 16];
        }
        __syncthreads();

        if (threadIdx.x < 8)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 8];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 8];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 8];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 8];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 8];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 8];
        }
        __syncthreads();

        if (threadIdx.x < 4)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 4];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 4];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 4];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 4];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 4];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 4];
        }
        __syncthreads();

        if (threadIdx.x < 2)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 2];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 2];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 2];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 2];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 2];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 2];
        }
        __syncthreads();

        if (threadIdx.x < 1)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 1];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 1];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 1];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 1];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 1];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 1];
        }
        __syncthreads();
        // ---------------------------------------------------------------------------------



        // ---------------------------------------------------------------------------------
        // We then accumulate this single condensed value into the element of
        // shared memory that correlates to the current tile position.
        // ---------------------------------------------------------------------------------
        if (threadIdx.x == 0)
        {
            sBuffer_x[tile_pos.w] += sBuffer_redux_x[0];
            sBuffer_y[tile_pos.w] += sBuffer_redux_y[0];
            sBuffer_z[tile_pos.w] += sBuffer_redux_z[0];
            tile_pos.w++;

            sBuffer_x[tile_pos.w] += sBuffer_redux_x2[0];
            sBuffer_y[tile_pos.w] += sBuffer_redux_y2[0];
            sBuffer_z[tile_pos.w] += sBuffer_redux_z2[0];
            tile_pos.w++;
        }
        __syncthreads();
        // ---------------------------------------------------------------------------------


        // #### SECOND HALF ####

        // ---------------------------------------------------------------------------------
        // Do the 2nd two x-positions out of four using our two
        // blocks of shared memory for reduction
        // ---------------------------------------------------------------------------------
        tile_pos.x++;
        A = B * tex1Dfetch(tex_LUT_Bspline_x, tile_pos.x * tile_dim.x + voxel_loc.x);

        // Perform the multiplication and store to redux shared memory
        sBuffer_redux_x[threadIdx.x] = voxel_val.x * A;
        sBuffer_redux_y[threadIdx.x] = voxel_val.y * A;
        sBuffer_redux_z[threadIdx.x] = voxel_val.z * A;
        tile_pos.x++;

        // Calculate the b-spline multiplier for this voxel @ the next tile
        // position relative to a given control knot.
        A = B * tex1Dfetch(tex_LUT_Bspline_x, tile_pos.x * tile_dim.x + voxel_loc.x);

        // Perform the multiplication and store to redux shared memory
        // for the second position
        sBuffer_redux_x2[threadIdx.x] = voxel_val.x * A;
        sBuffer_redux_y2[threadIdx.x] = voxel_val.y * A;
        sBuffer_redux_z2[threadIdx.x] = voxel_val.z * A;
        __syncthreads();
        // ---------------------------------------------------------------------------------


                    
        // ---------------------------------------------------------------------------------
        // All 64 dc_dv values in the current cluster have been processed
        // for the current 2 tile positions (out of 64 total tile positions).
        //
        // We now perform a sum reduction on these 64 dc_dv values to
        // condense the data down to one value.
        // ---------------------------------------------------------------------------------
        if (threadIdx.x < 32)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 32];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 32];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 32];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 32];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 32];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 32];
        }
        __syncthreads();

        if (threadIdx.x < 16)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 16];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 16];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 16];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 16];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 16];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 16];
        }
        __syncthreads();

        if (threadIdx.x < 8)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 8];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 8];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 8];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 8];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 8];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 8];
        }
        __syncthreads();

        if (threadIdx.x < 4)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 4];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 4];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 4];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 4];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 4];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 4];
        }
        __syncthreads();

        if (threadIdx.x < 2)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 2];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 2];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 2];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 2];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 2];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 2];
        }
        __syncthreads();

        if (threadIdx.x < 1)
        {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + 1];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + 1];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + 1];
            sBuffer_redux_x2[threadIdx.x] += sBuffer_redux_x2[threadIdx.x + 1];
            sBuffer_redux_y2[threadIdx.x] += sBuffer_redux_y2[threadIdx.x + 1];
            sBuffer_redux_z2[threadIdx.x] += sBuffer_redux_z2[threadIdx.x + 1];
        }
        __syncthreads();
        // ---------------------------------------------------------------------------------



        // ---------------------------------------------------------------------------------
        // We then accumulate this single condensed value into the element of
        // shared memory that correlates to the current tile position.
        // ---------------------------------------------------------------------------------
        if (threadIdx.x == 0)
        {
            sBuffer_x[tile_pos.w] += sBuffer_redux_x[0];
            sBuffer_y[tile_pos.w] += sBuffer_redux_y[0];
            sBuffer_z[tile_pos.w] += sBuffer_redux_z[0];
            tile_pos.w++;

            sBuffer_x[tile_pos.w] += sBuffer_redux_x2[0];
            sBuffer_y[tile_pos.w] += sBuffer_redux_y2[0];
            sBuffer_z[tile_pos.w] += sBuffer_redux_z2[0];
            tile_pos.w++;
        }
        __syncthreads();
        // ---------------------------------------------------------------------------------

        }
    } // LOOP: 64 B-Spline Values for current voxel_cluster

    } // LOOP: voxel_clusters


    // ----------------------------------------------------------
    //                STAGE 3 IN POWERPOINT
    // ----------------------------------------------------------
    // By this point every voxel cluster within the tile has been
    // processed for every possible tile position (there are 64).
    // ----------------------------------------------------------
    // HERE, EACH WARP OPERATES ON A SINGLE TILE'S SET OF 64!!
    // ----------------------------------------------------------
    tileOffset = 64*blockIdxInGrid;

    tile_pos.x = 63 - threadIdx.x;

    int knot_num;

    knot_num = LUT_Knot[tileOffset + threadIdx.x];

    cond_x[ (64*knot_num) + tile_pos.x ] = sBuffer_x[threadIdx.x];
    cond_y[ (64*knot_num) + tile_pos.x ] = sBuffer_y[threadIdx.x];
    cond_z[ (64*knot_num) + tile_pos.x ] = sBuffer_z[threadIdx.x];
    // ----------------------------------------------------------

    // Done with tile.

    // END OF KERNEL
}



////////////////////////////////////////////////////////////////////////////////
// KERNEL: kernel_bspline_mse_reduce()
//
// * Each threadblock contains only 2 warps.
// * Each threadblock operates on 32 knots (1 at a time)
//
// * Each knot in a condense stream contains 64 single precision floats
// * Each knot is spread across the 3 condense streams [x,y,z]
// * The "high warp" will handle floats 32-63
// * The "low warp"  will handle floats  0-31
//
// * The 2 warps will work together to sum reduce the 64 floats to 1 float
// * The sum reduction result is stored in shared memory
//
// AUTHOR: James Shackleford
// DATE  : August 27th, 2009
////////////////////////////////////////////////////////////////////////////////
__global__ void
kernel_bspline_mse_reduce (
    float* grad,        // Return: interleaved dc_dp values
    float* cond_x,      // Input : condensed dc_dv_x values
    float* cond_y,      // Input : condensed dc_dv_y values
    float* cond_z)      // Input : condensed dc_dv_z values
{
    // -- Setup Thread Attributes -----------------------------
    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
    // --------------------------------------------------------

    // -- Setup Shared Memory ---------------------------------
    // -- SIZE: ((3*64)+3)*sizeof(float)
    // --------------------------------------------------------
    extern __shared__ float sdata[]; 
    float* sBuffer = (float*)sdata;             // sBuffer[3]
    float* sBuffer_redux_x = (float*)&sBuffer[3];       // sBuffer_redux_x[64]
    float* sBuffer_redux_y = (float*)&sBuffer_redux_x[64];  // sBuffer_redux_y[64]
    float* sBuffer_redux_z = (float*)&sBuffer_redux_y[64];  // sBuffer_redux_z[64]
    // --------------------------------------------------------

    // Pull down the 64 condensed dc_dv values for the knot this warp pair is working on
    sBuffer_redux_x[threadIdx.x] = cond_x[64*blockIdxInGrid + threadIdx.x];
    sBuffer_redux_y[threadIdx.x] = cond_y[64*blockIdxInGrid + threadIdx.x];
    sBuffer_redux_z[threadIdx.x] = cond_z[64*blockIdxInGrid + threadIdx.x];

    // This thread barrier is very important!
    __syncthreads();
    
    // Perform sum reduction on the 64 condensed dc_dv values
    for(unsigned int s = 32; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sBuffer_redux_x[threadIdx.x] += sBuffer_redux_x[threadIdx.x + s];
            sBuffer_redux_y[threadIdx.x] += sBuffer_redux_y[threadIdx.x + s];
            sBuffer_redux_z[threadIdx.x] += sBuffer_redux_z[threadIdx.x + s];
        }

        // Wait for all threads in to complete the current tier.
        __syncthreads();
    }



    // Store 3 resulting floats into the output buffer (shared memory)
    // These 3 floats are the dc_dp value [x,y,z] for the current knot
    // This shared memory store is interleaved so that the final global
    // memory store will be coalaced.
    if (threadIdx.x == 0) {
        sBuffer[0] = sBuffer_redux_x[0];
    }
    
    if (threadIdx.x == 1) {
        sBuffer[1] = sBuffer_redux_y[0];
    }

    if (threadIdx.x == 2) {
        sBuffer[2] = sBuffer_redux_z[0];
    }

    // Prevent read before write race condition
    __syncthreads();


    if (threadIdx.x < 3) {
        grad[3*blockIdxInGrid + threadIdx.x] = sBuffer[threadIdx.x];
    }

    // END OF KERNEL 
}


/* JAS 05.27.2010
 * 
 * This kernel was written as an intended replacement for
 * bspline_cuda_score_j_mse_kernel1().  The intended goal
 * was to produce a kernel with neater notation and code
 * structure that also shared the LUT_Bspline_x,y,z textured
 * lookup table that is utilized by the hyper-fast gradient
 * kernel kernel_bspline_mse_condense_64_texfetch().
 * 
 * It should be noted that the LUT_Bspline texture differs
 * from the CPU based q_lut in both structure and philosophy.
 * LUT_Bspline is three separate look-up-tables which contain
 * the pre-computed basis function values in each dimension,
 * whereas the q_lut has already pre-multiplied all of these
 * results.  For the GPU, the q-LUT requires in too many memory
 * load operations, even when employing the cacheing mechanisms
 * provided by textures.  The LUT_Bspline textures rely on the GPU
 * to perform these multiplications, thus achieving superior
 * run times.
 *
 * This code was authored with the intention of unifying the
 * design philosophy of the MSE B-spline GPU implementation,
 * which was spurred by my attempts to write the upcoming
 * GPU Gems 4 chapter.
 *
 * The code now also shares more similarities with
 * the CPU code.  So, now if you know one you know the other.
 *
 * This is about 6.5% faster (on my GTX 285) than
 *   bspline_cuda_score_j_mse_kernel1()
 */
__global__ void
kernel_bspline_mse_score_dc_dv (
    float* score,       // OUTPUT
    float* skipped,     // OUTPUT
    float* dc_dv_x,     // OUTPUT
    float* dc_dv_y,     // OUTPUT
    float* dc_dv_z,     // OUTPUT
    float* f_img,       // fixed image voxels
    float* m_img,       // moving image voxels
    float* m_grad,      // moving image gradient
    int3 fdim,          // fixed  image dimensions
    int3 mdim,          // moving image dimensions
    int3 rdim,          //       region dimensions
    int3 cdim,          // # control points in x,y,z
    int3 vpr,           // voxels per region
    float3 img_origin,  // image origin
    float3 img_spacing, // image spacing
    float3 mov_offset,  // moving image offset
    float3 mov_ps,      // moving image pixel spacing
    int pad             // tile padding
)
{
    /* Setup Thread Attributes */
    int threadsPerBlock = (blockDim.x * blockDim.y * blockDim.z);

    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
    int thread_idxl     = (((blockDim.y * threadIdx.z) + threadIdx.y) * blockDim.x) + threadIdx.x;
    int thread_idxg     = (blockIdxInGrid * threadsPerBlock) + thread_idxl;

    /* Only process threads that map to voxels */
    if (thread_idxg > fdim.x * fdim.y * fdim.z) {
        return;
    }

    int4 p;     // Tile index
    int4 q;     // Local Voxel index (within tile)
    float3 f;   // Distance from origin (in mm )

    float3 m;   // Voxel Displacement   (in mm )
    float3 n;   // Voxel Displacement   (in vox)
    int3 n_f;   // Voxel Displacement floor
    int3 n_r;   // Voxel Displacement round
    float3 d;   // Deformation vector
    int fv;     // fixed voxel
    
    fv = thread_idxg;

    setup_indices (&p, &q, &f,
            fv, fdim, vpr, rdim, img_origin, img_spacing);

    int fell_out = find_correspondence (&d, &m, &n,
            f, mov_offset, mov_ps, mdim, cdim, vpr, p, q);

    if (fell_out) {
        skipped[fv]++;
        return;
    }

    float3 li_1, li_2;
    clamp_linear_interpolate_3d (&n, &n_f, &n_r, &li_1, &li_2, mdim);

    float m_val = get_moving_value (n_f, mdim, li_1, li_2);

    float diff = m_val - f_img[fv];
    score[fv] = diff * diff;

    write_dc_dv (dc_dv_x, dc_dv_y, dc_dv_z,
            m_grad, diff, n_r, mdim, vpr, pad, p, q);
}


/***********************************************************************
 * bspline_cuda_update_grad_kernel
 *
 * This kernel updates each of the gradient values before the final
 * sum reduction of the gradient stream.
 ***********************************************************************/
__global__ void
bspline_cuda_update_grad_kernel(
    float *grad,
    int num_vox,
    int num_elems)
{
    // Calculate the index of the thread block in the grid.
    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

    // Calculate the total number of threads in each thread block.
    int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

    // Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
    int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

    // Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
    int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

    if(threadIdxInGrid < num_elems) {
        grad[threadIdxInGrid] = 2.0 * grad[threadIdxInGrid] / num_vox;
    }
}


/***********************************************************************
 * sum_reduction_kernel
 *
 * This kernel will reduce a stream to a single value.  It will work for
 * a stream with an arbitrary number of elements.  It is the same as 
 * bspline_cuda_compute_score_kernel, with the exception that it assumes
 * all values in the stream are valid and should be included in the final
 * reduced value.
 ***********************************************************************/
__global__ void
sum_reduction_kernel(
    float *idata, 
    float *odata, 
    int   num_elems)
{
    // Shared memory is allocated on a per block basis.  Therefore, only allocate 
    // (sizeof(data) * blocksize) memory when calling the kernel.
    extern __shared__ float sdata[];
  
    // Calculate the index of the thread block in the grid.
    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
  
    // Calculate the total number of threads in each thread block.
    int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);
  
    // Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
    int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;
  
    // Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
    int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

    // Load data into shared memory.
    if(threadIdxInGrid >= num_elems)
    sdata[threadIdxInBlock] = 0.0;
    else 
    sdata[threadIdxInBlock] = idata[threadIdxInGrid];

    // Wait for all threads in the block to reach this point.
    __syncthreads();
  
    // Perform the reduction in shared memory.  Stride over the block and reduce
    // parts until it is down to a single value (stored in sdata[0]).
    for(unsigned int s = threadsPerBlock / 2; s > 0; s >>= 1) {
        if (threadIdxInBlock < s) {
            sdata[threadIdxInBlock] += sdata[threadIdxInBlock + s];
        }

        // Wait for all threads to complete this stride.
        __syncthreads();
    }
  
    // Write the result for this block back to global memory.
    if(threadIdxInBlock == 0) {
        odata[threadIdxInGrid] = sdata[0];
    }
}


/***********************************************************************
 * sum_reduction_last_step_kernel
 *
 * This kernel sums together the remaining partial sums that are created
 * by the other sum reduction kernels.
 ***********************************************************************/
__global__ void
sum_reduction_last_step_kernel(
    float *idata,
    float *odata,
    int   num_elems)
{
    // Calculate the index of the thread block in the grid.
    int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

    // Calculate the total number of threads in each thread block.
    int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

    // Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
    int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

    // Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
    int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

    if(threadIdxInGrid == 0) {
    
        float sum = 0.0;
        
        for(int i = 0; i < num_elems; i += threadsPerBlock) {
            sum += idata[i];
        }

        odata[0] = sum;
    }
}



////////////////////////////////////////////////////////////////////////////////
// FUNCTION: bspline_cuda_h_push_coeff_lut()
//
// This function overwries the coefficient LUT to the GPU global
// memory with the new coefficient LUT in preparation for
// the next iteration of score calculation.
////////////////////////////////////////////////////////////////////////////////
void
bspline_cuda_h_push_coeff_lut(Dev_Pointers_Bspline* dev_ptrs, Bspline_xform* bxf)
{
    // Copy the coefficient LUT to the GPU.
    hipMemcpy(dev_ptrs->coeff, bxf->coeff, dev_ptrs->coeff_size, hipMemcpyHostToDevice);
    cuda_utils_check_error("[Kernel Panic!] Failed to copy coefficient LUT to GPU");
}
////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
// FUNCTION: bspline_cuda_h_clear_score()
//
// This function sets all elements in the score (located on the GPU) to zero
// in preparation for the next iteration of the kernel.
////////////////////////////////////////////////////////////////////////////////
extern "C" void
bspline_cuda_h_clear_score(Dev_Pointers_Bspline* dev_ptrs) 
{
    hipMemset(dev_ptrs->score, 0, dev_ptrs->score_size);
    cuda_utils_check_error("Failed to clear the score stream on GPU\n");
}
////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
// FUNCTION: bspline_cuda_h_clear_grad()
//
// This function sets all elemtns in the gradients (located on the GPU) to
// zero in preparation for the next iteration of the kernel.
////////////////////////////////////////////////////////////////////////////////
extern "C" void
bspline_cuda_h_clear_grad(Dev_Pointers_Bspline* dev_ptrs) 
{
    hipMemset(dev_ptrs->grad, 0, dev_ptrs->grad_size);
    cuda_utils_check_error("Failed to clear the grad stream on GPU\n");
}
////////////////////////////////////////////////////////////////////////////////





////////////////////////////////////////////////////////////////////////////////
// FUNCTION: CPU_obtain_spline_basis_function()
//
// AUTHOR: James Shackleford
// DATE  : 09.04.2009
////////////////////////////////////////////////////////////////////////////////
float
CPU_obtain_spline_basis_function (
    int t_idx, 
    int vox_idx, 
    int vox_per_rgn)
{
                                
    float i = (float)vox_idx / vox_per_rgn;
    float C;
                        
    switch(t_idx) {
    case 0:
        C = (1.0/6.0) * (- 1.0 * i*i*i + 3.0 * i*i - 3.0 * i + 1.0);
        break;
    case 1:
        C = (1.0/6.0) * (+ 3.0 * i*i*i - 6.0 * i*i           + 4.0);
        break;
    case 2:
        C = (1.0/6.0) * (- 3.0 * i*i*i + 3.0 * i*i + 3.0 * i + 1.0);
        break;
    case 3:
        C = (1.0/6.0) * (+ 1.0 * i*i*i);
        break;
    default:
        C = 0.0;
        break;
    }

    return C;
}
////////////////////////////////////////////////////////////////////////////////


/******************************************************
* This function computes the spline basis function at 
* index 0, 1, 2, or 3 for a voxel 
Author: Naga Kandasamy
Date: 07 July 2009
*******************************************************/

__device__ float
obtain_spline_basis_function (float one_over_six,
    int t_idx, 
    int vox_idx, 
    int vox_per_rgn)
{
    float i = (float)vox_idx / vox_per_rgn;
    float C;
                        
    switch(t_idx) {
    case 0:
        C = one_over_six * (- 1.0 * i*i*i + 3.0 * i*i - 3.0 * i + 1.0);
        break;
    case 1:
        C = one_over_six * (+ 3.0 * i*i*i - 6.0 * i*i           + 4.0);
        break;
    case 2:
        C = one_over_six * (- 3.0 * i*i*i + 3.0 * i*i + 3.0 * i + 1.0);
        break;
    case 3:
        C = one_over_six * (+ 1.0 * i*i*i);
        break;
    }

    return C;
}


__device__ inline void
clamp_linear_interpolate_3d (
    float3* n,
    int3* n_f,
    int3* n_r,
    float3* li_1,
    float3* li_2,
    int3 mdim
)
{
    /* x-dimension */
    n_f->x = (int) floorf (n->x);
    n_r->x = (int) rintf (n->x);

    li_2->x = n->x - n_f->x;
    if (n_f->x < 0) {
        n_f->x = 0;
        n_r->x = 0;
        li_2->x = 0.0f;
    }
    else if (n_f->x >= (mdim.x - 1)) {
        n_f->x = mdim.x - 2;
        n_r->x = mdim.x - 1;
        li_2->x = 1.0f;
    }
    li_1->x = 1.0f - li_2->x;


    /* y-dimension */
    n_f->y = (int) floorf (n->y);
    n_r->y = (int) rintf (n->y);

    li_2->y = n->y - n_f->y;
    if (n_f->y < 0) {
        n_f->y = 0;
        n_r->y = 0;
        li_2->y = 0.0f;
    }
    else if (n_f->y >= (mdim.y - 1)) {
        n_f->y = mdim.y - 2;
        n_r->y = mdim.y - 1;
        li_2->y = 1.0f;
    }
    li_1->y = 1.0f - li_2->y;


    /* z-dimension */
    n_f->z = (int) floorf (n->z);
    n_r->z = (int) rintf (n->z);

    li_2->z = n->z - n_f->z;
    if (n_f->z < 0) {
        n_f->z = 0;
        n_r->z = 0;
        li_2->z = 0.0f;
    }
    else if (n_f->z >= (mdim.z - 1)) {
        n_f->z = mdim.z - 2;
        n_r->z = mdim.z - 1;
        li_2->z = 1.0f;
    }
    li_1->z = 1.0f - li_2->z;
}


__device__ inline int
find_correspondence (
   float3 *d,
   float3 *m,
   float3 *n,
   float3 f,
   float3 mov_offset,
   float3 mov_ps,
   int3 mdim,
   int3 cdim,
   int3 vpr,
   int4 p,
   int4 q
)
{
    int i, j, k, z, cidx;
    double A,B,C,P;

    d->x = 0.0f;
    d->y = 0.0f;
    d->z = 0.0f;

    z = 0;
    for (k = 0; k < 4; k++) {
    C = tex1Dfetch (tex_LUT_Bspline_x, k * vpr.z + q.z);
        for (j = 0; j < 4; j++) {
        B = tex1Dfetch (tex_LUT_Bspline_x, j * vpr.y + q.y);
            for (i = 0; i < 4; i++) {
                A = tex1Dfetch (tex_LUT_Bspline_x, i * vpr.x + q.x);
                P = A * B * C;

                cidx = 3 * ((p.z + k) * cdim.x * cdim.y 
                            + (p.y + j) * cdim.x + (p.x + i));

                d->x += P * tex1Dfetch (tex_coeff, cidx + 0);
                d->y += P * tex1Dfetch (tex_coeff, cidx + 1);
                d->z += P * tex1Dfetch (tex_coeff, cidx + 2);

                z++;
            }
        }
    }
    // --------------------------------------------------------


    // -- Correspondence --------------------------------------
    m->x = f.x + d->x;  // Displacement in mm
    m->y = f.y + d->y;
    m->z = f.z + d->z;

    // Displacement in voxels
    n->x = (m->x - mov_offset.x) / mov_ps.x;
    n->y = (m->y - mov_offset.y) / mov_ps.y;
    n->z = (m->z - mov_offset.z) / mov_ps.z;

    if (n->x < -0.5 || n->x > mdim.x - 0.5 ||
        n->y < -0.5 || n->y > mdim.y - 0.5 ||
        n->z < -0.5 || n->z > mdim.z - 0.5)
    {
        return 1;
    }
    return 0;
    // --------------------------------------------------------
}

__device__ inline float
get_moving_value (
    int3 n_f,
    int3 mdim,
    float3 li_1,
    float3 li_2
)
{
    // -- Compute coordinates of 8 nearest neighbors ----------
    int mvf;               // moving voxel (floor)
    int n1, n2, n3, n4;    // neighbors
    int n5, n6, n7, n8;
    
    mvf = (n_f.z * mdim.y + n_f.y) * mdim.x + n_f.x;

    n1 = mvf;
    n2 = n1 + 1;
    n3 = n1 + mdim.x;
    n4 = n1 + mdim.x + 1;
    n5 = n1 + mdim.x * mdim.y;
    n6 = n1 + mdim.x * mdim.y + 1;
    n7 = n1 + mdim.x * mdim.y + mdim.x;
    n8 = n1 + mdim.x * mdim.y + mdim.x + 1;
    // --------------------------------------------------------


    // -- Compute Moving Image Intensity ----------------------
    float w1, w2, w3, w4;
    float w5, w6, w7, w8;

    w1 = li_1.x * li_1.y * li_1.z * tex1Dfetch(tex_moving_image, n1);
    w2 = li_2.x * li_1.y * li_1.z * tex1Dfetch(tex_moving_image, n2);
    w3 = li_1.x * li_2.y * li_1.z * tex1Dfetch(tex_moving_image, n3);
    w4 = li_2.x * li_2.y * li_1.z * tex1Dfetch(tex_moving_image, n4);
    w5 = li_1.x * li_1.y * li_2.z * tex1Dfetch(tex_moving_image, n5);
    w6 = li_2.x * li_1.y * li_2.z * tex1Dfetch(tex_moving_image, n6);
    w7 = li_1.x * li_2.y * li_2.z * tex1Dfetch(tex_moving_image, n7);
    w8 = li_2.x * li_2.y * li_2.z * tex1Dfetch(tex_moving_image, n8);

    return w1 + w2 + w3 + w4 + w5 + w6 + w7 + w8;
    // --------------------------------------------------------

}


__device__ inline void
setup_indices (
    int4 *p,
    int4 *q,
    float3 *f,
    int fv,
    int3 fdim,
    int3 vpr,
    int3 rdim,
    float3 img_origin,
    float3 img_spacing
)
{
    /* Setup Global Voxel Indices */
    int3 r;     // Voxel index (global)
    r.z = fv / (fdim.x * fdim.y);
    r.y = (fv - (r.z * fdim.x * fdim.y)) / fdim.x;
    r.x = fv - r.z * fdim.x * fdim.y - (r.y * fdim.x);
    
    /* Setup Tile Indicies */
    p->x = r.x / vpr.x;
    p->y = r.y / vpr.y;
    p->z = r.z / vpr.z;
    p->w = ((p->z * rdim.y + p->y) * rdim.x) + p->x;

    /* Setup Local Voxel Indices */
    q->x = r.x - p->x * vpr.x;
    q->y = r.y - p->y * vpr.y;
    q->z = r.z - p->z * vpr.z;
    q->w = ((q->z * vpr.y + q->y) * vpr.x) + q->x;

    /* Set up fixed image coordinates (mm) */
    f->x = img_origin.x + img_spacing.x * r.x;
    f->y = img_origin.y + img_spacing.y * r.y;
    f->z = img_origin.z + img_spacing.z * r.z;
}


__device__ inline void
write_dc_dv (
    float* dc_dv_x,
    float* dc_dv_y,
    float* dc_dv_z,
    float* m_grad,
    float diff,
    int3 n_r,
    int3 mdim,
    int3 vpr,
    int pad,
    int4 p,
    int4 q
)
{
    float* m_grad_element;
    float* dc_dv_element_x;
    float* dc_dv_element_y;
    float* dc_dv_element_z;

    m_grad_element = &m_grad[3 * n_r.z * mdim.y * mdim.x];
    m_grad_element = &m_grad_element[3 * n_r.y * mdim.x];
    m_grad_element = &m_grad_element[3 * n_r.x];

    dc_dv_element_x = &dc_dv_x[((vpr.x * vpr.y * vpr.z) + pad) * p.w];
    dc_dv_element_y = &dc_dv_y[((vpr.x * vpr.y * vpr.z) + pad) * p.w];
    dc_dv_element_z = &dc_dv_z[((vpr.x * vpr.y * vpr.z) + pad) * p.w];

    dc_dv_element_x = &dc_dv_element_x[q.w];
    dc_dv_element_y = &dc_dv_element_y[q.w];
    dc_dv_element_z = &dc_dv_element_z[q.w];

    dc_dv_element_x[0] = diff * m_grad_element[0];
    dc_dv_element_y[0] = diff * m_grad_element[1];
    dc_dv_element_z[0] = diff * m_grad_element[2];
}
