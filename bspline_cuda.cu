#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "volume.h"
#include "readmha.h"
#include "bspline_opts.h"
#include "bspline.h"
#include "bspline_cuda.h"

// Include the kernels.
#include "bspline_cuda_kernels.cu"

// Declare global variables.
float *gpu_fixed_image;  // The fixed image
float *gpu_moving_image; // The moving image
float *gpu_moving_grad;
int   *gpu_c_lut; // The c_lut indicating which control knots affect voxels within a region
float *gpu_q_lut; // The q_lut indicating the distance of a voxel to each of the 64 control knots
float *gpu_coeff; // The coefficient stream indicating the x, y, z coefficients of each control knot
float *gpu_dx; // Streams to store voxel displacement/gradient values in the X, Y, and Z directions 
float *gpu_dy; 
float *gpu_dz;
float *gpu_diff;
float *gpu_dc_dv_x;
float *gpu_dc_dv_y;
float *gpu_dc_dv_z;
int   *gpu_valid_voxels;
size_t coeff_mem_size;

void bspline_cuda_initialize(
	Volume *fixed,
	Volume *moving,
	Volume *moving_grad,
	BSPLINE_Xform *bxf,
	BSPLINE_Parms *parms)
{
	printf("Initializing CUDA... ");
	fflush(stdout);

	// Copy the fixed image to the GPU.
	if(hipMalloc((void**)&gpu_fixed_image, fixed->npix * fixed->pix_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for fixed image");
	if(hipMemcpy(gpu_fixed_image, fixed->img, fixed->npix * fixed->pix_size, hipMemcpyHostToDevice) != hipSuccess)
		checkCUDAError("Failed to copy fixed image to GPU");
	if(hipBindTexture(0, tex_fixed_image, gpu_fixed_image, fixed->npix * fixed->pix_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_fixed_image to linear memory");

	// Copy the moving image to the GPU.
	if(hipMalloc((void**)&gpu_moving_image, moving->npix * moving->pix_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for moving image");
	if(hipMemcpy(gpu_moving_image, moving->img, moving->npix * moving->pix_size, hipMemcpyHostToDevice) != hipSuccess)
		checkCUDAError("Failed to copy moving image to GPU");
	if(hipBindTexture(0, tex_moving_image, gpu_moving_image, moving->npix * moving->pix_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_moving_image to linear memory");

	// Copy the moving gradient to the GPU.
	if(hipMalloc((void**)&gpu_moving_grad, moving_grad->npix * moving_grad->pix_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for moving gradient");
	if(hipMemcpy(gpu_moving_grad, moving_grad->img, moving_grad->npix * moving_grad->pix_size, hipMemcpyHostToDevice) != hipSuccess)
		checkCUDAError("Failed to copy moving gradient to GPU");
	if(hipBindTexture(0, tex_moving_grad, gpu_moving_grad, moving_grad->npix * moving_grad->pix_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_moving_grad to linear memory");

	// Allocate memory for the coefficient LUT on the GPU.  The LUT will be copied to the
	// GPU each time bspline_cuda_run_kernels is called.
	coeff_mem_size = sizeof(float) * bxf->num_coeff;
	if(hipMalloc((void**)&gpu_coeff, coeff_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for coefficient LUT");
	if(hipBindTexture(0, tex_coeff, gpu_coeff, coeff_mem_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_coeff to linear memory");

	// Copy the multiplier LUT to the GPU.
	size_t q_lut_mem_size = sizeof(float)
		* bxf->vox_per_rgn[0]
		* bxf->vox_per_rgn[1]
		* bxf->vox_per_rgn[2]
		* 64;
	if(hipMalloc((void**)&gpu_q_lut, q_lut_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for multiplier LUT");
	if(hipMemcpy(gpu_q_lut, bxf->q_lut, q_lut_mem_size, hipMemcpyHostToDevice) != hipSuccess)
		checkCUDAError("Failed to copy multiplier LUT to GPU");
	if(hipBindTexture(0, tex_q_lut, gpu_q_lut, q_lut_mem_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_q_lut to linear memory");

	// Copy the index LUT to the GPU.
	size_t c_lut_mem_size = sizeof(int) 
		* bxf->rdims[0] 
		* bxf->rdims[1] 
		* bxf->rdims[2] 
		* 64;
	if(hipMalloc((void**)&gpu_c_lut, c_lut_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for index LUT");
	if(hipMemcpy(gpu_c_lut, bxf->c_lut, c_lut_mem_size, hipMemcpyHostToDevice) != hipSuccess)
		checkCUDAError("Failed to copy index LUT to GPU");
	if(hipBindTexture(0, tex_c_lut, gpu_c_lut, c_lut_mem_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_c_lut to linear memory");

	// Allocate memory to hold the voxel displacement values.
	size_t volume_mem_size = fixed->npix * fixed->pix_size;
	if(hipMalloc((void**)&gpu_dx, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for dy stream on GPU");
	if(hipMalloc((void**)&gpu_dy, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for dx stream on GPU");
	if(hipMalloc((void**)&gpu_dz, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for dz stream on GPU");

	if(hipBindTexture(0, tex_dx, gpu_dx, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_dx to linear memory");
	if(hipBindTexture(0, tex_dy, gpu_dy, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_dy to linear memory");
	if(hipBindTexture(0, tex_dz, gpu_dz, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to bind tex_dz to linear memory");

	// Allocate memory to hold the calculated intensity difference values.
	if(hipMalloc((void**)&gpu_diff, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for the diff stream on GPU");

	// Allocate memory to hold the array of valid voxels;
	if(hipMalloc((void**)&gpu_valid_voxels, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for the valid_voxel stream on GPU");

	// Allocate memory to hold the calculated dc_dv values.
	if(hipMalloc((void**)&gpu_dc_dv_x, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for the dc_dv_x stream on GPU");
	if(hipMalloc((void**)&gpu_dc_dv_y, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for the dc_dv_x stream on GPU");
	if(hipMalloc((void**)&gpu_dc_dv_z, volume_mem_size) != hipSuccess)
		checkCUDAError("Failed to allocate memory for the dc_dv_x stream on GPU");

	printf("DONE!\n");
	fflush(stdout);
}

void bspline_cuda_run_kernels(
	Volume *fixed,
	Volume *moving,
	Volume *moving_grad,
	BSPLINE_Xform *bxf,
	BSPLINE_Parms *parms,
	float *host_diff,
	float *host_dc_dv_x,
	float *host_dc_dv_y,
	float *host_dc_dv_z)
{
	// Read in the dimensions of the volume.
    int3 volume_dim;
    volume_dim.x = fixed->dim[0]; 
    volume_dim.y = fixed->dim[1];
    volume_dim.z = fixed->dim[2];

	// Read in the dimensions of the region.
    float3 rdims;
    rdims.x = (float)bxf->rdims[0];
    rdims.y = (float)bxf->rdims[1];
    rdims.z = (float)bxf->rdims[2];

	// Read in spacing between the control knots.
    int3 vox_per_rgn;
    vox_per_rgn.x = bxf->vox_per_rgn[0];
    vox_per_rgn.y = bxf->vox_per_rgn[1];
    vox_per_rgn.z = bxf->vox_per_rgn[2];

	// Read in the coordinates of the image origin.
	float3 img_origin;
	img_origin.x = (float)bxf->img_origin[0];
	img_origin.y = (float)bxf->img_origin[1];
	img_origin.z = (float)bxf->img_origin[2];

	// Read in image offset.
	float3 img_offset;
	img_offset.x = (float)moving->offset[0];
	img_offset.y = (float)moving->offset[1];
	img_offset.z = (float)moving->offset[2];

	// Read in the voxel dimensions.
	float3 pix_spacing;
	pix_spacing.x = (float)moving->pix_spacing[0];
	pix_spacing.y = (float)moving->pix_spacing[1];
	pix_spacing.z = (float)moving->pix_spacing[2];

	// Copy the coefficient LUT to the GPU.
	if(hipMemcpy(gpu_coeff, bxf->coeff, coeff_mem_size, hipMemcpyHostToDevice) != hipSuccess)
		checkCUDAError("Failed to copy coefficient LUT to GPU");

	// Configure the grid.
	int num_blocks = (int)ceil((volume_dim.x * volume_dim.y * volume_dim.z) / 512.0);
	dim3 dimGrid(num_blocks, 1, 1);
	dim3 dimBlock(128, 2, 2);
	printf("%d thread blocks will be created for each kernel.\n", num_blocks);
	fflush(stdout);

	// Start the clock.
	LARGE_INTEGER clock_count, clock_frequency;
    double clock_start, clock_end;
	QueryPerformanceFrequency(&clock_frequency);
    QueryPerformanceCounter(&clock_count);
    clock_start = (double)clock_count.QuadPart;

	printf("Launching bspline_cuda_compute_dxyz_kernel... ");
	bspline_cuda_compute_dxyz_kernel<<<dimGrid, dimBlock>>>(
		gpu_c_lut,
		gpu_q_lut,
		gpu_coeff,
		volume_dim,
		vox_per_rgn,
		rdims,
		gpu_dx,
		gpu_dy,
		gpu_dz
	);

	if(hipDeviceSynchronize() != hipSuccess)
		checkCUDAError("\nbspline_cuda_compute_dxyz_kernel failed");
	else
		printf("DONE!\n");

	printf("Launching bspline_cuda_compute_diff_kernel... ");
	fflush(stdout);
	bspline_cuda_compute_diff_kernel<<<dimGrid, dimBlock>>>(
		gpu_fixed_image,
		gpu_moving_image,
		gpu_dx,
		gpu_dy,
		gpu_dz,
		gpu_diff,
		gpu_valid_voxels,
		volume_dim,
		img_origin,
		pix_spacing,
		img_offset
	);
	
	if(hipDeviceSynchronize() != hipSuccess)
		checkCUDAError("bspline_cuda_compute_diff_kernel failed");
	else
		printf("DONE!\n");
	
	if(hipMemcpy(host_diff, gpu_diff, fixed->npix * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
		checkCUDAError("Failed to copy diff stream from GPU to host");

	printf("Launching bspline_cuda_compute_dc_dv_kernel... ");
	fflush(stdout);
	bspline_cuda_compute_dc_dv_kernel<<<dimGrid, dimBlock>>>(
		gpu_fixed_image,
		gpu_moving_image,
		gpu_moving_grad,
		gpu_c_lut, 
		gpu_q_lut,
		gpu_dx,
		gpu_dy,
		gpu_dz,
		gpu_diff,
		gpu_dc_dv_x,
		gpu_dc_dv_y,
		gpu_dc_dv_z,
		// gpu_grad,
		gpu_valid_voxels,
		volume_dim,
		vox_per_rgn,
		rdims,
		img_origin,
		pix_spacing,
		img_offset
	);
	
	if(hipDeviceSynchronize() != hipSuccess)
		checkCUDAError("bspline_cuda_compute_dc_dv_kernel failed");
	else
		printf("DONE!\n");

	// Stop the clock.
	QueryPerformanceCounter(&clock_count);
    clock_end = (double)clock_count.QuadPart;
	printf("CUDA kernels completed in %f seconds.\n", double(clock_end - clock_start)/(double)clock_frequency.QuadPart);
	fflush(stdout);

	// Copy results back from GPU.
	if(hipMemcpy(host_dc_dv_x, gpu_dc_dv_x, fixed->npix * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
		checkCUDAError("Failed to copy dc_dv stream from GPU to host");
	if(hipMemcpy(host_dc_dv_y, gpu_dc_dv_y, fixed->npix * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
		checkCUDAError("Failed to copy dc_dv stream from GPU to host");
	if(hipMemcpy(host_dc_dv_z, gpu_dc_dv_z, fixed->npix * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess)
		checkCUDAError("Failed to copy dc_dv stream from GPU to host");

}

void bspline_cuda_clean_up() {

	// Free memory on GPU.
	if(hipFree(gpu_fixed_image) != hipSuccess) 
		checkCUDAError("Failed to free memory for fixed_image");
	if(hipFree(gpu_moving_image) != hipSuccess) 
		checkCUDAError("Failed to free memory for moving_image");
	if(hipFree(gpu_moving_grad) != hipSuccess)
		checkCUDAError("Failed to free memory for moving_grad");
	if(hipFree(gpu_coeff) != hipSuccess) 
		checkCUDAError("Failed to free memory for coeff");
	if(hipFree(gpu_q_lut) != hipSuccess) 
		checkCUDAError("Failed to free memory for q_lut");
	if(hipFree(gpu_c_lut) != hipSuccess) 
		checkCUDAError("Failed to free memory for c_lut");
	if(hipFree(gpu_dx) != hipSuccess)
		checkCUDAError("Failed to free memory for dx");
	if(hipFree(gpu_dy) != hipSuccess) 
		checkCUDAError("Failed to free memory for dy");
	if(hipFree(gpu_dz) != hipSuccess) 
		checkCUDAError("Failed to free memory for dz");
	if(hipFree(gpu_diff) != hipSuccess)
		checkCUDAError("Failed to free memory for diff");
	if(hipFree(gpu_dc_dv_x) != hipSuccess)
		checkCUDAError("Failed to free memory for dc_dv_x");
	if(hipFree(gpu_dc_dv_y) != hipSuccess)
		checkCUDAError("Failed to free memory for dc_dv_y");
	if(hipFree(gpu_dc_dv_z) != hipSuccess)
		checkCUDAError("Failed to free memory for dc_dv_z");
	if(hipFree(gpu_valid_voxels) != hipSuccess)
		checkCUDAError("Failed to free memory for valid_voxels");

	fflush(stdout);
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if(hipSuccess != err) 
	{
		printf("CUDA Error -- %s: %s.\n", msg, hipGetErrorString(err));
		fflush(stdout);
		exit(-1);
	} 
}