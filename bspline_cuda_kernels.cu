#include "hip/hip_runtime.h"
#include "bspline_cuda.h"

// Declare texture references.
texture<float, 1, hipReadModeElementType> tex_fixed_image;
texture<float, 1, hipReadModeElementType> tex_moving_image;
texture<float, 1, hipReadModeElementType> tex_moving_grad;
texture<float, 1, hipReadModeElementType> tex_coeff;
texture<int, 1, hipReadModeElementType>   tex_c_lut;
texture<float, 1, hipReadModeElementType> tex_q_lut;
texture<float, 1, hipReadModeElementType> tex_score;

texture<float, 1> tex_dx;
texture<float, 1> tex_dy;
texture<float, 1> tex_dz;

texture<float, 1> tex_dc_dv;
texture<float, 1> tex_grad;

/***********************************************************************
 * test_kernel
 * A simple kernel used to ensure that CUDA is working correctly. 
 ***********************************************************************/
__global__ void test_kernel(
	int3 volume_dim,
	float *dx,
	float *dy,
	float *dz)
{
	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
	
	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	if (threadIdxInGrid < (volume_dim.x * volume_dim.y * volume_dim.z))
	{
		dx[threadIdxInGrid] = (float)threadIdxInGrid;
		dy[threadIdxInGrid] = (float)threadIdxInGrid;
		dz[threadIdxInGrid] = (float)threadIdxInGrid;
	}
}

/***********************************************************************
 * bspline_cuda_score_e_mse_kernel1a
 ***********************************************************************/
__global__ void bspline_cuda_score_e_mse_kernel1a (
	float  *dc_dv,
	float  *score,
	float3 rdims,			// Number of tiles/regions in x, y, and z
	int3   volume_dim,		// x, y, z dimensions of the volume in voxels
	float3 img_origin,		// Image origin (in mm)
    float3 img_spacing,     // Image spacing (in mm)
	float3 img_offset,		// Offset corresponding to the region of interest
    int3   roi_offset,	    // Position of first vox in ROI (in vox)
    int3   roi_dim,			// Dimension of ROI (in vox)
    int3   vox_per_rgn,	    // Knot spacing (in vox)
	float3 pix_spacing)		// Dimensions of a single voxel (in mm)
{
	int3   vox_coordinate;	// X, Y, Z coordinates for this voxel	
	int3   p;				// Offset of the tile in the volume (x, y and z)
	int3   q;				// Offset within the tile (measured in voxels).
	int3   coord_in_volume;	// Offset within the volume (measured in voxels).
	int    fv;				// Index of voxel in linear image array.
	float  fx, fy, fz;		// Physical coordinates within the volume.
	int    pidx;			// Index into c_lut.
	int    qidx;			// Index into q_lut.
	int    cidx;			// Index into the coefficient table.

	float  P;				
	float3 N;				// Multiplier values.		
	float3 d;				// B-spline deformation vector.
	float  diff;

	float3 distance_from_image_origin;
	float3 displacement_in_mm; 
	float3 displacement_in_vox;
	float3 displacement_in_vox_floor;
	float3 displacement_in_vox_round;
	float  fx1, fx2, fy1, fy2, fz1, fz2;
	int    mvf;
	float  m_val;
	float  m_x1y1z1, m_x2y1z1, m_x1y2z1, m_x2y2z1, m_x1y1z2, m_x2y1z2, m_x1y2z2, m_x2y2z2;

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// If the voxel lies outside the volume, do nothing.
	if(threadIdxInGrid < (volume_dim.x * volume_dim.y * volume_dim.z))
	{
		// Get the X, Y, Z position of the voxel.
		vox_coordinate.z = threadIdxInGrid / (volume_dim.x * volume_dim.y);
		vox_coordinate.y = (threadIdxInGrid - (vox_coordinate.z * volume_dim.x * volume_dim.y)) / volume_dim.x;
		vox_coordinate.x = threadIdxInGrid - vox_coordinate.z * volume_dim.x * volume_dim.y - (vox_coordinate.y * volume_dim.x);
	
		// Get the tile location of the voxel.
		p.x = vox_coordinate.x / vox_per_rgn.x;
		p.y = vox_coordinate.y / vox_per_rgn.y;
		p.z = vox_coordinate.z / vox_per_rgn.z;
	
		// Get the offset of the voxel within the tile.
		q.x = vox_coordinate.x - p.x * vox_per_rgn.x;
		q.y = vox_coordinate.y - p.y * vox_per_rgn.y;
		q.z = vox_coordinate.z - p.z * vox_per_rgn.z;

		// Calculate the x, y and z offsets of the voxel within the volume.
		coord_in_volume.x = roi_offset.x + p.x * vox_per_rgn.x + q.x;
		coord_in_volume.y = roi_offset.y + p.y * vox_per_rgn.y + q.y;
		coord_in_volume.z = roi_offset.z + p.z * vox_per_rgn.z + q.z;

		// If the voxel lies outside the image, do nothing.
		if(coord_in_volume.x <= (roi_offset.x + roi_dim.x) || 
			coord_in_volume.y <= (roi_offset.y + roi_dim.y) ||
			coord_in_volume.z <= (roi_offset.z + roi_dim.z)) {

			// Compute the physical coordinates of fixed image voxel.
			fx = img_origin.x + img_spacing.x * coord_in_volume.x;
			fy = img_origin.y + img_spacing.y * coord_in_volume.y;
			fz = img_origin.z + img_spacing.z * coord_in_volume.z;

			// Compute the linear index of fixed image voxel.
			fv = (coord_in_volume.z * volume_dim.x * volume_dim.y) + (coord_in_volume.y * volume_dim.x) + coord_in_volume.x;

			// ----------------------------------------------------------------
			// Calculate the B-Spline deformation vector.
			// ----------------------------------------------------------------

			// Use the offset of the voxel within the region to compute the index into the c_lut.
			pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
			pidx = pidx * 64;

			// Use the offset of the voxel to compute the index into the multiplier LUT or q_lut.
			qidx = ((q.z * vox_per_rgn.y + q.y) * vox_per_rgn.x) + q.x;
			qidx = qidx * 64;

			// Compute the deformation vector.
			d.x = 0.0;
			d.y = 0.0;
			d.z = 0.0;

			for(int k = 0; k < 64; k++)
			{
				// Calculate the index into the coefficients array.
				cidx = 3 * tex1Dfetch(tex_c_lut, pidx + k); 
				
				// Fetch the values for P, Ni, Nj, and Nk.
				P   = tex1Dfetch(tex_q_lut, qidx + k); 
				N.x = tex1Dfetch(tex_coeff, cidx + 0);  // x-value
				N.y = tex1Dfetch(tex_coeff, cidx + 1);  // y-value
				N.z = tex1Dfetch(tex_coeff, cidx + 2);  // z-value

				// Update the output (v) values.
				d.x += P * N.x;
				d.y += P * N.y;
				d.z += P * N.z;
			}

			// ----------------------------------------------------------------
			// Find correspondence in the moving image.
			// ----------------------------------------------------------------

			// Calculate the distance of the voxel from the origin (in mm) along the x, y and z axes.
			distance_from_image_origin.x = img_origin.x + (pix_spacing.x * coord_in_volume.x);
			distance_from_image_origin.y = img_origin.y + (pix_spacing.y * coord_in_volume.y);
			distance_from_image_origin.z = img_origin.z + (pix_spacing.z * coord_in_volume.z);
			
			// Calculate the displacement of the voxel (in mm) in the x, y, and z directions.
			displacement_in_mm.x = distance_from_image_origin.x + d.x;
			displacement_in_mm.y = distance_from_image_origin.y + d.y;
			displacement_in_mm.z = distance_from_image_origin.z + d.z;

			// Calculate the displacement value in terms of voxels.
			displacement_in_vox.x = (displacement_in_mm.x - img_offset.x) / pix_spacing.x;
			displacement_in_vox.y = (displacement_in_mm.y - img_offset.y) / pix_spacing.y;
			displacement_in_vox.z = (displacement_in_mm.z - img_offset.z) / pix_spacing.z;

			// Check if the displaced voxel lies outside the region of interest.
			if ((displacement_in_vox.x < -0.5) || (displacement_in_vox.x > (volume_dim.x - 0.5)) || 
				(displacement_in_vox.y < -0.5) || (displacement_in_vox.y > (volume_dim.y - 0.5)) || 
				(displacement_in_vox.z < -0.5) || (displacement_in_vox.z > (volume_dim.z - 0.5))) {
					// Do nothing.
			}
			else {

				// ----------------------------------------------------------------
				// Compute interpolation fractions.
				// ----------------------------------------------------------------

				// Clamp and interpolate along the X axis.
				displacement_in_vox_floor.x = floor(displacement_in_vox.x);
				displacement_in_vox_round.x = round(displacement_in_vox.x);
				fx2 = displacement_in_vox.x - displacement_in_vox_floor.x;
				if(displacement_in_vox_floor.x < 0){
					displacement_in_vox_floor.x = 0;
					displacement_in_vox_round.x = 0;
					fx2 = 0.0;
				}
				else if(displacement_in_vox_floor.x >= (volume_dim.x - 1)){
					displacement_in_vox_floor.x = volume_dim.x - 2;
					displacement_in_vox_round.x = volume_dim.x - 1;
					fx2 = 1.0;
				}
				fx1 = 1.0 - fx2;

				// Clamp and interpolate along the Y axis.
				displacement_in_vox_floor.y = floor(displacement_in_vox.y);
				displacement_in_vox_round.y = round(displacement_in_vox.y);
				fy2 = displacement_in_vox.y - displacement_in_vox_floor.y;
				if(displacement_in_vox_floor.y < 0){
					displacement_in_vox_floor.y = 0;
					displacement_in_vox_round.y = 0;
					fy2 = 0.0;
				}
				else if(displacement_in_vox_floor.y >= (volume_dim.y - 1)){
					displacement_in_vox_floor.y = volume_dim.y - 2;
					displacement_in_vox_round.y = volume_dim.y - 1;
					fy2 = 1.0;
				}
				fy1 = 1.0 - fy2;
				
				// Clamp and intepolate along the Z axis.
				displacement_in_vox_floor.z = floor(displacement_in_vox.z);
				displacement_in_vox_round.z = round(displacement_in_vox.z);
				fz2 = displacement_in_vox.z - displacement_in_vox_floor.z;
				if(displacement_in_vox_floor.z < 0){
					displacement_in_vox_floor.z = 0;
					displacement_in_vox_round.z = 0;
					fz2 = 0.0;
				}
				else if(displacement_in_vox_floor.z >= (volume_dim.z - 1)){
					displacement_in_vox_floor.z = volume_dim.z - 2;
					displacement_in_vox_round.z = volume_dim.z - 1;
					fz2 = 1.0;
				}
				fz1 = 1.0 - fz2;
				
				// ----------------------------------------------------------------
				// Compute moving image intensity using linear interpolation.
				// ----------------------------------------------------------------

				mvf = (displacement_in_vox_floor.z * volume_dim.y + displacement_in_vox_floor.y) * volume_dim.x + displacement_in_vox_floor.x;
				m_x1y1z1 = fx1 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf);
				m_x2y1z1 = fx2 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf + 1);
				m_x1y2z1 = fx1 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x);
				m_x2y2z1 = fx2 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x + 1);
				m_x1y1z2 = fx1 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x);
				m_x2y1z2 = fx2 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + 1);
				m_x1y2z2 = fx1 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x);
				m_x2y2z2 = fx2 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x + 1);
				m_val = m_x1y1z1 + m_x2y1z1 + m_x1y2z1 + m_x2y2z1 + m_x1y1z2 + m_x2y1z2 + m_x1y2z2 + m_x2y2z2;

				// ----------------------------------------------------------------
				// Compute intensity difference.
				// ----------------------------------------------------------------

				diff = tex1Dfetch(tex_fixed_image, fv) - m_val;
				score[threadIdxInGrid] = diff;

				// ----------------------------------------------------------------
				// Accumulate the score.
				// ----------------------------------------------------------------

				score[threadIdxInGrid] = tex1Dfetch(tex_score, threadIdxInGrid) + (diff * diff);
			}	
		}
	}
}

/***********************************************************************
 * bspline_cuda_score_e_mse_kernel1
 * This kernel is one of two used in the CUDA implementation of 
 * score_e_mse.  It calculates the dc_dv values for a region, as wellr
 * as accumulates the score for the entire volume.
 ***********************************************************************/
__global__ void bspline_cuda_score_e_mse_kernel1b (
	float  *dc_dv,
	float  *score,
	int3   sidx,			// Current "set index" given in x, y and z
	float3 rdims,			// Number of tiles/regions in x, y, and z
	int3   sdims,           // Dimensions of the set in tiles (x, y and z)
	int3   volume_dim,		// x, y, z dimensions of the volume in voxels
	float3 img_origin,		// Image origin (in mm)
    float3 img_spacing,     // Image spacing (in mm)
	float3 img_offset,		// Offset corresponding to the region of interest
    int3   roi_offset,	    // Position of first vox in ROI (in vox)
    int3   roi_dim,			// Dimension of ROI (in vox)
    int3   vox_per_rgn,	    // Knot spacing (in vox)
	int    total_vox_per_rgn,
	float3 pix_spacing)		// Dimensions of a single voxel (in mm)
{
	int3   s;				// Offset of the tile in the set (x, y and z)
	int3   p;				// Offset of the tile in the volume (x, y and z)
	int3   q;				// Offset within the tile (measured in voxels).
	int3   coord_in_volume;	// Offset within the volume (measured in voxels).
	int    fv;				// Index of voxel in linear image array.
	float  fx, fy, fz;		// Physical coordinates within the volume.
	int    pidx;			// Index into c_lut.
	int    qidx;			// Index into q_lut.
	int    cidx;			// Index into the coefficient table.

	float  P;				
	float3 N;				// Multiplier values.		
	float3 d;				// B-spline deformation vector.
	float  diff;

	float3 distance_from_image_origin;
	float3 displacement_in_mm; 
	float3 displacement_in_vox;
	float3 displacement_in_vox_floor;
	float3 displacement_in_vox_round;
	float  fx1, fx2, fy1, fy2, fz1, fz2;
	int    mvf;
	float  mvr;
	float  m_val;
	float  m_x1y1z1, m_x2y1z1, m_x1y2z1, m_x2y2z1, m_x1y1z2, m_x2y1z2, m_x1y2z2, m_x2y2z2;

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// Calculate the linear "set index," which is the index of the tile in the set that contains the 
	// voxel corresponding to this thread.
	int tileIdxInSet = threadIdxInGrid / total_vox_per_rgn;

	// If the voxel lies outside the volume, do nothing.
	if(threadIdxInGrid < (volume_dim.x * volume_dim.y * volume_dim.z))
	{
		// Calculate the offset of the tile within the set in the x, y, and z directions.
		s.x = tileIdxInSet % sdims.x;
		s.y = ((tileIdxInSet - s.x) / sdims.x) % sdims.y;
		s.z = ((((tileIdxInSet - s.x) / sdims.x) - s.y) / sdims.y) % sdims.z;

		// Calculate the offset of the tile in the volume, based on the set offset.
		p.x = (s.x * 4) + sidx.x;
		p.y = (s.y * 4) + sidx.y;
		p.z = (s.z * 4) + sidx.z;

		// Calculate the x, y and z offsets of the voxel within the tile.
		q.x = threadIdxInGrid % vox_per_rgn.x;
		q.y = ((threadIdxInGrid - q.x) / vox_per_rgn.x) % vox_per_rgn.y;
		q.z = ((((threadIdxInGrid - q.x) / vox_per_rgn.x) - q.y) / vox_per_rgn.y) % vox_per_rgn.z;

		// Calculate the x, y and z offsets of the voxel within the volume.
		coord_in_volume.x = roi_offset.x + p.x * vox_per_rgn.x + q.x;
		coord_in_volume.y = roi_offset.y + p.y * vox_per_rgn.y + q.y;
		coord_in_volume.z = roi_offset.z + p.z * vox_per_rgn.z + q.z;

		// If the voxel lies outside the image, do nothing.
		if(coord_in_volume.x <= (roi_offset.x + roi_dim.x) || 
			coord_in_volume.y <= (roi_offset.y + roi_dim.y) ||
			coord_in_volume.z <= (roi_offset.z + roi_dim.z)) {

			// Compute the physical coordinates of fixed image voxel.
			fx = img_origin.x + img_spacing.x * coord_in_volume.x;
			fy = img_origin.y + img_spacing.y * coord_in_volume.y;
			fz = img_origin.z + img_spacing.z * coord_in_volume.z;

			// Compute the linear index of fixed image voxel.
			fv = (coord_in_volume.z * volume_dim.x * volume_dim.y) + (coord_in_volume.y * volume_dim.x) + coord_in_volume.x;

			// ----------------------------------------------------------------
			// Calculate the B-Spline deformation vector.
			// ----------------------------------------------------------------

			// Use the offset of the voxel within the region to compute the index into the c_lut.
			pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
			pidx = pidx * 64;

			// Use the offset of the voxel to compute the index into the multiplier LUT or q_lut.
			qidx = ((q.z * vox_per_rgn.y + q.y) * vox_per_rgn.x) + q.x;
			qidx = qidx * 64;

			// Compute the deformation vector.
			d.x = 0.0;
			d.y = 0.0;
			d.z = 0.0;

			for(int k = 0; k < 64; k++)
			{
				// Calculate the index into the coefficients array.
				cidx = 3 * tex1Dfetch(tex_c_lut, pidx + k); 
				
				// Fetch the values for P, Ni, Nj, and Nk.
				P   = tex1Dfetch(tex_q_lut, qidx + k); 
				N.x = tex1Dfetch(tex_coeff, cidx + 0);  // x-value
				N.y = tex1Dfetch(tex_coeff, cidx + 1);  // y-value
				N.z = tex1Dfetch(tex_coeff, cidx + 2);  // z-value

				// Update the output (v) values.
				d.x += P * N.x;
				d.y += P * N.y;
				d.z += P * N.z;
			}
			
			// ----------------------------------------------------------------
			// Find correspondence in the moving image.
			// ----------------------------------------------------------------

			// Calculate the distance of the voxel from the origin (in mm) along the x, y and z axes.
			distance_from_image_origin.x = img_origin.x + (pix_spacing.x * coord_in_volume.x);
			distance_from_image_origin.y = img_origin.y + (pix_spacing.y * coord_in_volume.y);
			distance_from_image_origin.z = img_origin.z + (pix_spacing.z * coord_in_volume.z);
			
			// Calculate the displacement of the voxel (in mm) in the x, y, and z directions.
			displacement_in_mm.x = distance_from_image_origin.x + d.x;
			displacement_in_mm.y = distance_from_image_origin.y + d.y;
			displacement_in_mm.z = distance_from_image_origin.z + d.z;

			// Calculate the displacement value in terms of voxels.
			displacement_in_vox.x = (displacement_in_mm.x - img_offset.x) / pix_spacing.x;
			displacement_in_vox.y = (displacement_in_mm.y - img_offset.y) / pix_spacing.y;
			displacement_in_vox.z = (displacement_in_mm.z - img_offset.z) / pix_spacing.z;

			// Check if the displaced voxel lies outside the region of interest.
			if ((displacement_in_vox.x < -0.5) || (displacement_in_vox.x > (volume_dim.x - 0.5)) || 
				(displacement_in_vox.y < -0.5) || (displacement_in_vox.y > (volume_dim.y - 0.5)) || 
				(displacement_in_vox.z < -0.5) || (displacement_in_vox.z > (volume_dim.z - 0.5))) {
					// Do nothing.
			}
			else {

				// ----------------------------------------------------------------
				// Compute interpolation fractions.
				// ----------------------------------------------------------------

				// Clamp and interpolate along the X axis.
				displacement_in_vox_floor.x = floor(displacement_in_vox.x);
				displacement_in_vox_round.x = round(displacement_in_vox.x);
				fx2 = displacement_in_vox.x - displacement_in_vox_floor.x;
				if(displacement_in_vox_floor.x < 0){
					displacement_in_vox_floor.x = 0;
					displacement_in_vox_round.x = 0;
					fx2 = 0.0;
				}
				else if(displacement_in_vox_floor.x >= (volume_dim.x - 1)){
					displacement_in_vox_floor.x = volume_dim.x - 2;
					displacement_in_vox_round.x = volume_dim.x - 1;
					fx2 = 1.0;
				}
				fx1 = 1.0 - fx2;

				// Clamp and interpolate along the Y axis.
				displacement_in_vox_floor.y = floor(displacement_in_vox.y);
				displacement_in_vox_round.y = round(displacement_in_vox.y);
				fy2 = displacement_in_vox.y - displacement_in_vox_floor.y;
				if(displacement_in_vox_floor.y < 0){
					displacement_in_vox_floor.y = 0;
					displacement_in_vox_round.y = 0;
					fy2 = 0.0;
				}
				else if(displacement_in_vox_floor.y >= (volume_dim.y - 1)){
					displacement_in_vox_floor.y = volume_dim.y - 2;
					displacement_in_vox_round.y = volume_dim.y - 1;
					fy2 = 1.0;
				}
				fy1 = 1.0 - fy2;
				
				// Clamp and intepolate along the Z axis.
				displacement_in_vox_floor.z = floor(displacement_in_vox.z);
				displacement_in_vox_round.z = round(displacement_in_vox.z);
				fz2 = displacement_in_vox.z - displacement_in_vox_floor.z;
				if(displacement_in_vox_floor.z < 0){
					displacement_in_vox_floor.z = 0;
					displacement_in_vox_round.z = 0;
					fz2 = 0.0;
				}
				else if(displacement_in_vox_floor.z >= (volume_dim.z - 1)){
					displacement_in_vox_floor.z = volume_dim.z - 2;
					displacement_in_vox_round.z = volume_dim.z - 1;
					fz2 = 1.0;
				}
				fz1 = 1.0 - fz2;
				
				// ----------------------------------------------------------------
				// Compute moving image intensity using linear interpolation.
				// ----------------------------------------------------------------

				mvf = (displacement_in_vox_floor.z * volume_dim.y + displacement_in_vox_floor.y) * volume_dim.x + displacement_in_vox_floor.x;
				m_x1y1z1 = fx1 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf);
				m_x2y1z1 = fx2 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf + 1);
				m_x1y2z1 = fx1 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x);
				m_x2y2z1 = fx2 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x + 1);
				m_x1y1z2 = fx1 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x);
				m_x2y1z2 = fx2 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + 1);
				m_x1y2z2 = fx1 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x);
				m_x2y2z2 = fx2 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x + 1);
				m_val = m_x1y1z1 + m_x2y1z1 + m_x1y2z1 + m_x2y2z1 + m_x1y1z2 + m_x2y1z2 + m_x1y2z2 + m_x2y2z2;

				// ----------------------------------------------------------------
				// Compute intensity difference.
				// ----------------------------------------------------------------

				diff = tex1Dfetch(tex_fixed_image, fv) - m_val;
				
				// ----------------------------------------------------------------
				// Accumulate the score.
				// ----------------------------------------------------------------

				// The score calculation has been moved to bspline_cuda_score_e_kernel1a.
				// score[threadIdxInGrid] = tex1Dfetch(tex_score, threadIdxInGrid) + (diff * diff);

				// ----------------------------------------------------------------
				// Compute dc_dv for this offset
				// ----------------------------------------------------------------
				
				// Compute spatial gradient using nearest neighbors.
				mvr = (((displacement_in_vox_round.z * volume_dim.y) + displacement_in_vox_round.y) * volume_dim.x) + displacement_in_vox_round.x;
				dc_dv[3*(threadIdxInGrid)+0] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 0);
				dc_dv[3*(threadIdxInGrid)+1] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 1);
				dc_dv[3*(threadIdxInGrid)+2] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 2);
			}		
		}
	}
}

__global__ void bspline_cuda_score_e_mse_kernel1 (
	float  *dc_dv,
	float  *score,
	int3   sidx,			// Current "set index" given in x, y and z
	float3 rdims,			// Number of tiles/regions in x, y, and z
	int3   sdims,           // Dimensions of the set in tiles (x, y and z)
	int3   volume_dim,		// x, y, z dimensions of the volume in voxels
	float3 img_origin,		// Image origin (in mm)
    float3 img_spacing,     // Image spacing (in mm)
	float3 img_offset,		// Offset corresponding to the region of interest
    int3   roi_offset,	    // Position of first vox in ROI (in vox)
    int3   roi_dim,			// Dimension of ROI (in vox)
    int3   vox_per_rgn,	    // Knot spacing (in vox)
	int    total_vox_per_rgn,
	float3 pix_spacing)		// Dimensions of a single voxel (in mm)
{
	int3   s;				// Offset of the tile in the set (x, y and z)
	int3   p;				// Offset of the tile in the volume (x, y and z)
	int3   q;				// Offset within the tile (measured in voxels).
	int3   coord_in_volume;	// Offset within the volume (measured in voxels).
	int    fv;				// Index of voxel in linear image array.
	float  fx, fy, fz;		// Physical coordinates within the volume.
	int    pidx;			// Index into c_lut.
	int    qidx;			// Index into q_lut.
	int    cidx;			// Index into the coefficient table.

	float  P;				
	float3 N;				// Multiplier values.		
	float3 d;				// B-spline deformation vector.
	float  diff;

	float3 distance_from_image_origin;
	float3 displacement_in_mm; 
	float3 displacement_in_vox;
	float3 displacement_in_vox_floor;
	float3 displacement_in_vox_round;
	float  fx1, fx2, fy1, fy2, fz1, fz2;
	int    mvf;
	float  mvr;
	float  m_val;
	float  m_x1y1z1, m_x2y1z1, m_x1y2z1, m_x2y2z1, m_x1y1z2, m_x2y1z2, m_x1y2z2, m_x2y2z2;

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// Calculate the linear "set index," which is the index of the tile in the set that contains the 
	// voxel corresponding to this thread.
	int tileIdxInSet = threadIdxInGrid / total_vox_per_rgn;

	// If the voxel lies outside the volume, do nothing.
	if(threadIdxInGrid < (volume_dim.x * volume_dim.y * volume_dim.z))
	{
		// Calculate the offset of the tile within the set in the x, y, and z directions.
		s.x = tileIdxInSet % sdims.x;
		s.y = ((tileIdxInSet - s.x) / sdims.x) % sdims.y;
		s.z = ((((tileIdxInSet - s.x) / sdims.x) - s.y) / sdims.y) % sdims.z;

		// Calculate the offset of the tile in the volume, based on the set offset.
		p.x = (s.x * 4) + sidx.x;
		p.y = (s.y * 4) + sidx.y;
		p.z = (s.z * 4) + sidx.z;

		// Calculate the x, y and z offsets of the voxel within the tile.
		q.x = threadIdxInGrid % vox_per_rgn.x;
		q.y = ((threadIdxInGrid - q.x) / vox_per_rgn.x) % vox_per_rgn.y;
		q.z = ((((threadIdxInGrid - q.x) / vox_per_rgn.x) - q.y) / vox_per_rgn.y) % vox_per_rgn.z;

		// Calculate the x, y and z offsets of the voxel within the volume.
		coord_in_volume.x = roi_offset.x + p.x * vox_per_rgn.x + q.x;
		coord_in_volume.y = roi_offset.y + p.y * vox_per_rgn.y + q.y;
		coord_in_volume.z = roi_offset.z + p.z * vox_per_rgn.z + q.z;

		// If the voxel lies outside the image, do nothing.
		if(coord_in_volume.x <= (roi_offset.x + roi_dim.x) || 
			coord_in_volume.y <= (roi_offset.y + roi_dim.y) ||
			coord_in_volume.z <= (roi_offset.z + roi_dim.z)) {

			// Compute the physical coordinates of fixed image voxel.
			fx = img_origin.x + img_spacing.x * coord_in_volume.x;
			fy = img_origin.y + img_spacing.y * coord_in_volume.y;
			fz = img_origin.z + img_spacing.z * coord_in_volume.z;

			// Compute the linear index of fixed image voxel.
			fv = (coord_in_volume.z * volume_dim.x * volume_dim.y) + (coord_in_volume.y * volume_dim.x) + coord_in_volume.x;

			// ----------------------------------------------------------------
			// Calculate the B-Spline deformation vector.
			// ----------------------------------------------------------------

			// Use the offset of the voxel within the region to compute the index into the c_lut.
			pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
			pidx = pidx * 64;

			// Use the offset of the voxel to compute the index into the multiplier LUT or q_lut.
			qidx = ((q.z * vox_per_rgn.y + q.y) * vox_per_rgn.x) + q.x;
			qidx = qidx * 64;

			// Compute the deformation vector.
			d.x = 0.0;
			d.y = 0.0;
			d.z = 0.0;

			for(int k = 0; k < 64; k++)
			{
				// Calculate the index into the coefficients array.
				cidx = 3 * tex1Dfetch(tex_c_lut, pidx + k); 
				
				// Fetch the values for P, Ni, Nj, and Nk.
				P   = tex1Dfetch(tex_q_lut, qidx + k); 
				N.x = tex1Dfetch(tex_coeff, cidx + 0);  // x-value
				N.y = tex1Dfetch(tex_coeff, cidx + 1);  // y-value
				N.z = tex1Dfetch(tex_coeff, cidx + 2);  // z-value

				// Update the output (v) values.
				d.x += P * N.x;
				d.y += P * N.y;
				d.z += P * N.z;
			}
			
			// ----------------------------------------------------------------
			// Find correspondence in the moving image.
			// ----------------------------------------------------------------

			// Calculate the distance of the voxel from the origin (in mm) along the x, y and z axes.
			distance_from_image_origin.x = img_origin.x + (pix_spacing.x * coord_in_volume.x);
			distance_from_image_origin.y = img_origin.y + (pix_spacing.y * coord_in_volume.y);
			distance_from_image_origin.z = img_origin.z + (pix_spacing.z * coord_in_volume.z);
			
			// Calculate the displacement of the voxel (in mm) in the x, y, and z directions.
			displacement_in_mm.x = distance_from_image_origin.x + d.x;
			displacement_in_mm.y = distance_from_image_origin.y + d.y;
			displacement_in_mm.z = distance_from_image_origin.z + d.z;

			// Calculate the displacement value in terms of voxels.
			displacement_in_vox.x = (displacement_in_mm.x - img_offset.x) / pix_spacing.x;
			displacement_in_vox.y = (displacement_in_mm.y - img_offset.y) / pix_spacing.y;
			displacement_in_vox.z = (displacement_in_mm.z - img_offset.z) / pix_spacing.z;

			// Check if the displaced voxel lies outside the region of interest.
			if ((displacement_in_vox.x < -0.5) || (displacement_in_vox.x > (volume_dim.x - 0.5)) || 
				(displacement_in_vox.y < -0.5) || (displacement_in_vox.y > (volume_dim.y - 0.5)) || 
				(displacement_in_vox.z < -0.5) || (displacement_in_vox.z > (volume_dim.z - 0.5))) {
					// Do nothing.
			}
			else {

				// ----------------------------------------------------------------
				// Compute interpolation fractions.
				// ----------------------------------------------------------------

				// Clamp and interpolate along the X axis.
				displacement_in_vox_floor.x = floor(displacement_in_vox.x);
				displacement_in_vox_round.x = round(displacement_in_vox.x);
				fx2 = displacement_in_vox.x - displacement_in_vox_floor.x;
				if(displacement_in_vox_floor.x < 0){
					displacement_in_vox_floor.x = 0;
					displacement_in_vox_round.x = 0;
					fx2 = 0.0;
				}
				else if(displacement_in_vox_floor.x >= (volume_dim.x - 1)){
					displacement_in_vox_floor.x = volume_dim.x - 2;
					displacement_in_vox_round.x = volume_dim.x - 1;
					fx2 = 1.0;
				}
				fx1 = 1.0 - fx2;

				// Clamp and interpolate along the Y axis.
				displacement_in_vox_floor.y = floor(displacement_in_vox.y);
				displacement_in_vox_round.y = round(displacement_in_vox.y);
				fy2 = displacement_in_vox.y - displacement_in_vox_floor.y;
				if(displacement_in_vox_floor.y < 0){
					displacement_in_vox_floor.y = 0;
					displacement_in_vox_round.y = 0;
					fy2 = 0.0;
				}
				else if(displacement_in_vox_floor.y >= (volume_dim.y - 1)){
					displacement_in_vox_floor.y = volume_dim.y - 2;
					displacement_in_vox_round.y = volume_dim.y - 1;
					fy2 = 1.0;
				}
				fy1 = 1.0 - fy2;
				
				// Clamp and intepolate along the Z axis.
				displacement_in_vox_floor.z = floor(displacement_in_vox.z);
				displacement_in_vox_round.z = round(displacement_in_vox.z);
				fz2 = displacement_in_vox.z - displacement_in_vox_floor.z;
				if(displacement_in_vox_floor.z < 0){
					displacement_in_vox_floor.z = 0;
					displacement_in_vox_round.z = 0;
					fz2 = 0.0;
				}
				else if(displacement_in_vox_floor.z >= (volume_dim.z - 1)){
					displacement_in_vox_floor.z = volume_dim.z - 2;
					displacement_in_vox_round.z = volume_dim.z - 1;
					fz2 = 1.0;
				}
				fz1 = 1.0 - fz2;
				
				// ----------------------------------------------------------------
				// Compute moving image intensity using linear interpolation.
				// ----------------------------------------------------------------

				mvf = (displacement_in_vox_floor.z * volume_dim.y + displacement_in_vox_floor.y) * volume_dim.x + displacement_in_vox_floor.x;
				m_x1y1z1 = fx1 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf);
				m_x2y1z1 = fx2 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf + 1);
				m_x1y2z1 = fx1 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x);
				m_x2y2z1 = fx2 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x + 1);
				m_x1y1z2 = fx1 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x);
				m_x2y1z2 = fx2 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + 1);
				m_x1y2z2 = fx1 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x);
				m_x2y2z2 = fx2 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x + 1);
				m_val = m_x1y1z1 + m_x2y1z1 + m_x1y2z1 + m_x2y2z1 + m_x1y1z2 + m_x2y1z2 + m_x1y2z2 + m_x2y2z2;

				// ----------------------------------------------------------------
				// Compute intensity difference.
				// ----------------------------------------------------------------

				diff = tex1Dfetch(tex_fixed_image, fv) - m_val;
				
				// ----------------------------------------------------------------
				// Accumulate the score.
				// ----------------------------------------------------------------

				// The score calculation has been moved to bspline_cuda_score_e_kernel1a.
				score[threadIdxInGrid] = tex1Dfetch(tex_score, threadIdxInGrid) + (diff * diff);

				// ----------------------------------------------------------------
				// Compute dc_dv for this offset
				// ----------------------------------------------------------------
				
				// Compute spatial gradient using nearest neighbors.
				mvr = (((displacement_in_vox_round.z * volume_dim.y) + displacement_in_vox_round.y) * volume_dim.x) + displacement_in_vox_round.x;
				dc_dv[3*(threadIdxInGrid)+0] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 0);
				dc_dv[3*(threadIdxInGrid)+1] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 1);
				dc_dv[3*(threadIdxInGrid)+2] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 2);
			}		
		}
	}
}

/***********************************************************************
 * bspline_cuda_score_d_mse_kernel1
 * This kernel is one of two used in the CUDA implementation of 
 * score_d_mse.  It calculates the dc_dv values for a region, as well
 * as accumulates the score for the entire volume.
 ***********************************************************************/
__global__ void bspline_cuda_score_d_mse_kernel1 (
	float  *dc_dv,
	float  *score,			
	int3   p,				// Offset of the tile in the volume (x, y and z)
	int3   volume_dim,		// x, y, z dimensions of the volume in voxels
	float3 img_origin,		// Image origin (in mm)
    float3 img_spacing,     // Image spacing (in mm)
	float3 img_offset,		// Offset corresponding to the region of interest
    int3   roi_offset,	    // Position of first vox in ROI (in vox)
    int3   roi_dim,			// Dimension of ROI (in vox)
    int3   vox_per_rgn,	    // Knot spacing (in vox)
	float3 pix_spacing,		// Dimensions of a single voxel (in mm)
	float3 rdims)			// # of regions in (x,y,z)
{
	int3   q;				// Offset within the tile (measured in voxels).
	int3   coord_in_volume;	// Offset within the volume (measured in voxels).
	int    fv;				// Index of voxel in linear image array.
	float  fx, fy, fz;		// Physical coordinates within the volume.
	int    pidx;			// Index into c_lut.
	int    qidx;			// Index into q_lut.
	int    cidx;			// Index into the coefficient table.

	float  P;				
	float3 N;				// Multiplier values.		
	float3 d;				// B-spline deformation vector.
	float  diff;

	float3 distance_from_image_origin;
	float3 displacement_in_mm; 
	float3 displacement_in_vox;
	float3 displacement_in_vox_floor;
	float3 displacement_in_vox_round;
	float  fx1, fx2, fy1, fy2, fz1, fz2;
	int    mvf;
	float  mvr;
	float  m_val;
	float  m_x1y1z1, m_x2y1z1, m_x1y2z1, m_x2y2z1, m_x1y1z2, m_x2y1z2, m_x1y2z2, m_x2y2z2;

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// If the voxel lies outside the region, do nothing.
	if(threadIdxInGrid < (vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z))
	{	
		// Calculate the x, y and z offsets of the voxel within the tile.
		q.x = threadIdxInGrid % vox_per_rgn.x;
		q.y = ((threadIdxInGrid - q.x) / vox_per_rgn.x) % vox_per_rgn.y;
		q.z = ((((threadIdxInGrid - q.x) / vox_per_rgn.x) - q.y) / vox_per_rgn.y) % vox_per_rgn.z;

		// Calculate the x, y and z offsets of the voxel within the volume.
		coord_in_volume.x = roi_offset.x + p.x * vox_per_rgn.x + q.x;
		coord_in_volume.y = roi_offset.y + p.y * vox_per_rgn.y + q.y;
		coord_in_volume.z = roi_offset.z + p.z * vox_per_rgn.z + q.z;

		// If the voxel lies outside the image, do nothing.
		if(coord_in_volume.x <= (roi_offset.x + roi_dim.x) || 
			coord_in_volume.y <= (roi_offset.y + roi_dim.y) ||
			coord_in_volume.z <= (roi_offset.z + roi_dim.z)) {

			// Compute the physical coordinates of fixed image voxel.
			fx = img_origin.x + img_spacing.x * coord_in_volume.x;
			fy = img_origin.y + img_spacing.y * coord_in_volume.y;
			fz = img_origin.z + img_spacing.z * coord_in_volume.z;

			// Compute the linear index of fixed image voxel.
			fv = (coord_in_volume.z * volume_dim.x * volume_dim.y) + (coord_in_volume.y * volume_dim.x) + coord_in_volume.x;

			// ----------------------------------------------------------------
			// Calculate the B-Spline deformation vector.
			// ----------------------------------------------------------------

			// Use the offset of the voxel within the region to compute the index into the c_lut.
			pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
			pidx = pidx * 64;

			// Use the offset of the voxel to compute the index into the multiplier LUT or q_lut.
			// qidx = ((q.z * vox_per_rgn.y + q.y) * vox_per_rgn.x) + q.x;
			qidx = threadIdxInGrid * 64;

			// Compute the deformation vector.
			d.x = 0.0;
			d.y = 0.0;
			d.z = 0.0;

			for(int k = 0; k < 64; k++)
			{
				// Calculate the index into the coefficients array.
				cidx = 3 * tex1Dfetch(tex_c_lut, pidx + k); 
				
				// Fetch the values for P, Ni, Nj, and Nk.
				P   = tex1Dfetch(tex_q_lut, qidx + k); 
				N.x = tex1Dfetch(tex_coeff, cidx + 0);  // x-value
				N.y = tex1Dfetch(tex_coeff, cidx + 1);  // y-value
				N.z = tex1Dfetch(tex_coeff, cidx + 2);  // z-value

				// Update the output (v) values.
				d.x += P * N.x;
				d.y += P * N.y;
				d.z += P * N.z;
			}
			
			// ----------------------------------------------------------------
			// Find correspondence in the moving image.
			// ----------------------------------------------------------------

			// Calculate the distance of the voxel from the origin (in mm) along the x, y and z axes.
			distance_from_image_origin.x = img_origin.x + (pix_spacing.x * coord_in_volume.x);
			distance_from_image_origin.y = img_origin.y + (pix_spacing.y * coord_in_volume.y);
			distance_from_image_origin.z = img_origin.z + (pix_spacing.z * coord_in_volume.z);
			
			// Calculate the displacement of the voxel (in mm) in the x, y, and z directions.
			displacement_in_mm.x = distance_from_image_origin.x + d.x;
			displacement_in_mm.y = distance_from_image_origin.y + d.y;
			displacement_in_mm.z = distance_from_image_origin.z + d.z;

			// Calculate the displacement value in terms of voxels.
			displacement_in_vox.x = (displacement_in_mm.x - img_offset.x) / pix_spacing.x;
			displacement_in_vox.y = (displacement_in_mm.y - img_offset.y) / pix_spacing.y;
			displacement_in_vox.z = (displacement_in_mm.z - img_offset.z) / pix_spacing.z;

			// Check if the displaced voxel lies outside the region of interest.
			if ((displacement_in_vox.x < -0.5) || (displacement_in_vox.x > (volume_dim.x - 0.5)) || 
				(displacement_in_vox.y < -0.5) || (displacement_in_vox.y > (volume_dim.y - 0.5)) || 
				(displacement_in_vox.z < -0.5) || (displacement_in_vox.z > (volume_dim.z - 0.5))) {
				// diff = 0.0;
				// valid = 0;
			}
			else {

				// ----------------------------------------------------------------
				// Compute interpolation fractions.
				// ----------------------------------------------------------------

				// Clamp and interpolate along the X axis.
				displacement_in_vox_floor.x = floor(displacement_in_vox.x);
				displacement_in_vox_round.x = round(displacement_in_vox.x);
				fx2 = displacement_in_vox.x - displacement_in_vox_floor.x;
				if(displacement_in_vox_floor.x < 0){
					displacement_in_vox_floor.x = 0;
					displacement_in_vox_round.x = 0;
					fx2 = 0.0;
				}
				else if(displacement_in_vox_floor.x >= (volume_dim.x - 1)){
					displacement_in_vox_floor.x = volume_dim.x - 2;
					displacement_in_vox_round.x = volume_dim.x - 1;
					fx2 = 1.0;
				}
				fx1 = 1.0 - fx2;

				// Clamp and interpolate along the Y axis.
				displacement_in_vox_floor.y = floor(displacement_in_vox.y);
				displacement_in_vox_round.y = round(displacement_in_vox.y);
				fy2 = displacement_in_vox.y - displacement_in_vox_floor.y;
				if(displacement_in_vox_floor.y < 0){
					displacement_in_vox_floor.y = 0;
					displacement_in_vox_round.y = 0;
					fy2 = 0.0;
				}
				else if(displacement_in_vox_floor.y >= (volume_dim.y - 1)){
					displacement_in_vox_floor.y = volume_dim.y - 2;
					displacement_in_vox_round.y = volume_dim.y - 1;
					fy2 = 1.0;
				}
				fy1 = 1.0 - fy2;
				
				// Clamp and intepolate along the Z axis.
				displacement_in_vox_floor.z = floor(displacement_in_vox.z);
				displacement_in_vox_round.z = round(displacement_in_vox.z);
				fz2 = displacement_in_vox.z - displacement_in_vox_floor.z;
				if(displacement_in_vox_floor.z < 0){
					displacement_in_vox_floor.z = 0;
					displacement_in_vox_round.z = 0;
					fz2 = 0.0;
				}
				else if(displacement_in_vox_floor.z >= (volume_dim.z - 1)){
					displacement_in_vox_floor.z = volume_dim.z - 2;
					displacement_in_vox_round.z = volume_dim.z - 1;
					fz2 = 1.0;
				}
				fz1 = 1.0 - fz2;
				
				// ----------------------------------------------------------------
				// Compute moving image intensity using linear interpolation.
				// ----------------------------------------------------------------

				mvf = (displacement_in_vox_floor.z * volume_dim.y + displacement_in_vox_floor.y) * volume_dim.x + displacement_in_vox_floor.x;
				m_x1y1z1 = fx1 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf);
				m_x2y1z1 = fx2 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf + 1);
				m_x1y2z1 = fx1 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x);
				m_x2y2z1 = fx2 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x + 1);
				m_x1y1z2 = fx1 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x);
				m_x2y1z2 = fx2 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + 1);
				m_x1y2z2 = fx1 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x);
				m_x2y2z2 = fx2 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x + 1);
				m_val = m_x1y1z1 + m_x2y1z1 + m_x1y2z1 + m_x2y2z1 + m_x1y1z2 + m_x2y1z2 + m_x1y2z2 + m_x2y2z2;

				// ----------------------------------------------------------------
				// Compute intensity difference.
				// ----------------------------------------------------------------

				// diff[threadIdxInGrid] = fixed_image[threadIdxInGrid] - m_val;
				diff = tex1Dfetch(tex_fixed_image, fv) - m_val;
				
				// ----------------------------------------------------------------
				// Accumulate the score.
				// ----------------------------------------------------------------

				score[threadIdxInGrid] += diff * diff;
				//score[threadIdxInGrid] = tex1Dfetch(tex_score, threadIdxInGrid) + (diff * diff);

				// ----------------------------------------------------------------
				// Compute dc_dv for this offset
				// ----------------------------------------------------------------
				
				// Compute spatial gradient using nearest neighbors.
				mvr = (((displacement_in_vox_round.z * volume_dim.y) + displacement_in_vox_round.y) * volume_dim.x) + displacement_in_vox_round.x;
				dc_dv[3*(threadIdxInGrid)+0] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 0);
				dc_dv[3*(threadIdxInGrid)+1] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 1);
				dc_dv[3*(threadIdxInGrid)+2] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 2);
			}		
		}
	}
}

/***********************************************************************
 * bspline_cuda_score_d_mse_kernel1_v2
 * This kernel is one of two used in the CUDA implementation of 
 * score_d_mse.  It calculates the dc_dv values for a region, as well
 * as accumulates the score for the entire volume.  In comparison to 
 * bspline_cuda_score_d_mse_kernel1, this kernel computes the x, y, and
 * z portions of each value in separate threads for increased parallelism.
 ***********************************************************************/
__global__ void bspline_cuda_score_d_mse_kernel1_v2 (
	float  *dc_dv,
	float  *score,			
	int3   p,				// Offset of the tile in the volume (x, y and z)
	int3   volume_dim,		// x, y, z dimensions of the volume in voxels
	float3 img_origin,		// Image origin (in mm)
    float3 img_spacing,     // Image spacing (in mm)
	float3 img_offset,		// Offset corresponding to the region of interest
    int3   roi_offset,	    // Position of first vox in ROI (in vox)
    int3   roi_dim,			// Dimension of ROI (in vox)
    int3   vox_per_rgn,	    // Knot spacing (in vox)
	float3 pix_spacing,		// Dimensions of a single voxel (in mm)
	float3 rdims)			// # of regions in (x,y,z)
{
	int3   q;				// Offset within the tile (measured in voxels).
	int3   coord_in_volume;	// Offset within the volume (measured in voxels).
	int    fv;				// Index of voxel in linear image array.
	float  fx, fy, fz;		// Physical coordinates within the volume.
	int    pidx;			// Index into c_lut.
	int    qidx;			// Index into q_lut.
	int    cidx;			// Index into the coefficient table.

	float  P;				
	float3 N;				// Multiplier values.		
	float3 d;				// B-spline deformation vector.
	float  diff;

	float3 distance_from_image_origin;
	float3 displacement_in_mm; 
	float3 displacement_in_vox;
	float3 displacement_in_vox_floor;
	float3 displacement_in_vox_round;
	float  fx1, fx2, fy1, fy2, fz1, fz2;
	int    mvf;
	float  mvr;
	float  m_val;
	float  m_x1y1z1, m_x2y1z1, m_x1y2z1, m_x2y2z1, m_x1y1z2, m_x2y1z2, m_x1y2z2, m_x2y2z2;

	int lridx = 0;  // Linear index within the region
	int offset = 0; // x = 0, y = 1, z = 2

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// If the voxel lies outside the region, do nothing.
	if(threadIdxInGrid < (3 * vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z))
	{	
		// Calculate the linear index of the voxel in the region. Will be in the range
		// (0, vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z - 1).
		lridx = threadIdxInGrid / 3;

		// Calculate the coordinate offset (x = 0, y = 1, z = 2).
		offset = threadIdxInGrid - (lridx * 3);		

		// Calculate the x, y and z offsets of the voxel within the tile.
		q.x = lridx % vox_per_rgn.x;
		q.y = ((lridx - q.x) / vox_per_rgn.x) % vox_per_rgn.y;
		q.z = ((((lridx - q.x) / vox_per_rgn.x) - q.y) / vox_per_rgn.y) % vox_per_rgn.z;

		// Calculate the x, y and z offsets of the voxel within the volume.
		coord_in_volume.x = roi_offset.x + p.x * vox_per_rgn.x + q.x;
		coord_in_volume.y = roi_offset.y + p.y * vox_per_rgn.y + q.y;
		coord_in_volume.z = roi_offset.z + p.z * vox_per_rgn.z + q.z;

		// If the voxel lies outside the image, do nothing.
		if(coord_in_volume.x <= (roi_offset.x + roi_dim.x) || 
			coord_in_volume.y <= (roi_offset.y + roi_dim.y) ||
			coord_in_volume.z <= (roi_offset.z + roi_dim.z)) {

			// Compute the physical coordinates of fixed image voxel.
			fx = img_origin.x + img_spacing.x * coord_in_volume.x;
			fy = img_origin.y + img_spacing.y * coord_in_volume.y;
			fz = img_origin.z + img_spacing.z * coord_in_volume.z;

			// Compute the linear index of fixed image voxel in the volume.
			fv = (coord_in_volume.z * volume_dim.x * volume_dim.y) + (coord_in_volume.y * volume_dim.x) + coord_in_volume.x;
			
			// ----------------------------------------------------------------
			// Calculate the B-Spline deformation vector.
			// ----------------------------------------------------------------

			// Use the offset of the voxel within the region to compute the index into the c_lut.
			pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
			pidx = pidx * 64;

			// Use the offset of the voxel to compute the index into the multiplier LUT or q_lut.
			// qidx = ((q.z * vox_per_rgn.y + q.y) * vox_per_rgn.x) + q.x;
			qidx = lridx * 64;

			// Compute the deformation vector.
			d.x = 0.0;
			d.y = 0.0;
			d.z = 0.0;

			for(int k = 0; k < 64; k++)
			{
				// Calculate the index into the coefficients array.
				cidx = 3 * tex1Dfetch(tex_c_lut, pidx + k); 
				
				// Fetch the values for P, Ni, Nj, and Nk.
				P   = tex1Dfetch(tex_q_lut, qidx + k); 
				N.x = tex1Dfetch(tex_coeff, cidx + 0);  // x-value
				N.y = tex1Dfetch(tex_coeff, cidx + 1);  // y-value
				N.z = tex1Dfetch(tex_coeff, cidx + 2);  // z-value

				// Update the output (v) values.
				d.x += P * N.x;
				d.y += P * N.y;
				d.z += P * N.z;
			}
			
			// ----------------------------------------------------------------
			// Find correspondence in the moving image.
			// ----------------------------------------------------------------

			// Calculate the distance of the voxel from the origin (in mm) along the x, y and z axes.
			distance_from_image_origin.x = img_origin.x + (pix_spacing.x * coord_in_volume.x);
			distance_from_image_origin.y = img_origin.y + (pix_spacing.y * coord_in_volume.y);
			distance_from_image_origin.z = img_origin.z + (pix_spacing.z * coord_in_volume.z);
			
			// Calculate the displacement of the voxel (in mm) in the x, y, and z directions.
			displacement_in_mm.x = distance_from_image_origin.x + d.x;
			displacement_in_mm.y = distance_from_image_origin.y + d.y;
			displacement_in_mm.z = distance_from_image_origin.z + d.z;

			// Calculate the displacement value in terms of voxels.
			displacement_in_vox.x = (displacement_in_mm.x - img_offset.x) / pix_spacing.x;
			displacement_in_vox.y = (displacement_in_mm.y - img_offset.y) / pix_spacing.y;
			displacement_in_vox.z = (displacement_in_mm.z - img_offset.z) / pix_spacing.z;

			// Check if the displaced voxel lies outside the region of interest.
			if ((displacement_in_vox.x < -0.5) || (displacement_in_vox.x > (volume_dim.x - 0.5)) || 
				(displacement_in_vox.y < -0.5) || (displacement_in_vox.y > (volume_dim.y - 0.5)) || 
				(displacement_in_vox.z < -0.5) || (displacement_in_vox.z > (volume_dim.z - 0.5))) {
				// Do nothing.
			}
			else {

				// ----------------------------------------------------------------
				// Compute interpolation fractions.
				// ----------------------------------------------------------------

				// Clamp and interpolate along the X axis.
				displacement_in_vox_floor.x = floor(displacement_in_vox.x);
				displacement_in_vox_round.x = round(displacement_in_vox.x);
				fx2 = displacement_in_vox.x - displacement_in_vox_floor.x;
				if(displacement_in_vox_floor.x < 0){
					displacement_in_vox_floor.x = 0;
					displacement_in_vox_round.x = 0;
					fx2 = 0.0;
				}
				else if(displacement_in_vox_floor.x >= (volume_dim.x - 1)){
					displacement_in_vox_floor.x = volume_dim.x - 2;
					displacement_in_vox_round.x = volume_dim.x - 1;
					fx2 = 1.0;
				}
				fx1 = 1.0 - fx2;

				// Clamp and interpolate along the Y axis.
				displacement_in_vox_floor.y = floor(displacement_in_vox.y);
				displacement_in_vox_round.y = round(displacement_in_vox.y);
				fy2 = displacement_in_vox.y - displacement_in_vox_floor.y;
				if(displacement_in_vox_floor.y < 0){
					displacement_in_vox_floor.y = 0;
					displacement_in_vox_round.y = 0;
					fy2 = 0.0;
				}
				else if(displacement_in_vox_floor.y >= (volume_dim.y - 1)){
					displacement_in_vox_floor.y = volume_dim.y - 2;
					displacement_in_vox_round.y = volume_dim.y - 1;
					fy2 = 1.0;
				}
				fy1 = 1.0 - fy2;
				
				// Clamp and intepolate along the Z axis.
				displacement_in_vox_floor.z = floor(displacement_in_vox.z);
				displacement_in_vox_round.z = round(displacement_in_vox.z);
				fz2 = displacement_in_vox.z - displacement_in_vox_floor.z;
				if(displacement_in_vox_floor.z < 0){
					displacement_in_vox_floor.z = 0;
					displacement_in_vox_round.z = 0;
					fz2 = 0.0;
				}
				else if(displacement_in_vox_floor.z >= (volume_dim.z - 1)){
					displacement_in_vox_floor.z = volume_dim.z - 2;
					displacement_in_vox_round.z = volume_dim.z - 1;
					fz2 = 1.0;
				}
				fz1 = 1.0 - fz2;
				
				// ----------------------------------------------------------------
				// Compute moving image intensity using linear interpolation.
				// ----------------------------------------------------------------

				mvf = (displacement_in_vox_floor.z * volume_dim.y + displacement_in_vox_floor.y) * volume_dim.x + displacement_in_vox_floor.x;
				m_x1y1z1 = fx1 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf);
				m_x2y1z1 = fx2 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf + 1);
				m_x1y2z1 = fx1 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x);
				m_x2y2z1 = fx2 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x + 1);
				m_x1y1z2 = fx1 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x);
				m_x2y1z2 = fx2 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + 1);
				m_x1y2z2 = fx1 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x);
				m_x2y2z2 = fx2 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x + 1);
				m_val = m_x1y1z1 + m_x2y1z1 + m_x1y2z1 + m_x2y2z1 + m_x1y1z2 + m_x2y1z2 + m_x1y2z2 + m_x2y2z2;

				// ----------------------------------------------------------------
				// Compute intensity difference.
				// ----------------------------------------------------------------

				// diff[threadIdxInGrid] = fixed_image[threadIdxInGrid] - m_val;
				diff = tex1Dfetch(tex_fixed_image, fv) - m_val;
				
				// ----------------------------------------------------------------
				// Accumulate the score.
				// ----------------------------------------------------------------

				if(offset == 0)
					score[lridx] = tex1Dfetch(tex_score, lridx) + (diff * diff);

				// ----------------------------------------------------------------
				// Compute dc_dv for this offset
				// ----------------------------------------------------------------
				
				// Compute spatial gradient using nearest neighbors.
				mvr = (((displacement_in_vox_round.z * volume_dim.y) + displacement_in_vox_round.y) * volume_dim.x) + displacement_in_vox_round.x;
				dc_dv[threadIdxInGrid] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + offset);
			}		
		}
	}
}

/***********************************************************************
 * bspline_cuda_score_d_mse_kernel1_v3
 * This kernel is one of two used in the CUDA implementation of 
 * score_d_mse.  It calculates the dc_dv values for a region, as well
 * as accumulates the score for the entire volume.  In comparison to 
 * bspline_cuda_score_d_mse_kernel2, this kernel uses some shared
 * memory to exchange data between threads to reduce memory accesses.
 ***********************************************************************/
__global__ void bspline_cuda_score_d_mse_kernel1_v3 (
	float  *dc_dv,
	float  *score,			
	int3   p,				// Offset of the tile in the volume (x, y and z)
	int3   volume_dim,		// x, y, z dimensions of the volume in voxels
	float3 img_origin,		// Image origin (in mm)
    float3 img_spacing,     // Image spacing (in mm)
	float3 img_offset,		// Offset corresponding to the region of interest
    int3   roi_offset,	    // Position of first vox in ROI (in vox)
    int3   roi_dim,			// Dimension of ROI (in vox)
    int3   vox_per_rgn,	    // Knot spacing (in vox)
	float3 pix_spacing,		// Dimensions of a single voxel (in mm)
	float3 rdims)			// # of regions in (x,y,z)
{
	// Shared memory is allocated on a per block basis.  Therefore, only allocate 
	// (sizeof(data) * blocksize) memory when calling the kernel.
	extern __shared__ float sdata[]; 

	int lridx = 0;  // Linear index within the region
	int offset = 0; // x = 0, y = 1, z = 2

	int3   q;				// Offset within the tile (measured in voxels).
	int3   coord_in_volume;	// Offset within the volume (measured in voxels).
	int    fv;				// Index of voxel in linear image array.
	float  fx, fy, fz;		// Physical coordinates within the volume.
	int    pidx;			// Index into c_lut.
	int    qidx;			// Index into q_lut.
	int    cidx;			// Index into the coefficient table.
	float  P;				
	float3 N;				// Multiplier values.		
	float3 d;				// B-spline deformation vector.
	float  diff;
	float3 distance_from_image_origin;
	float3 displacement_in_mm; 
	float3 displacement_in_vox;
	float3 displacement_in_vox_floor;
	float3 displacement_in_vox_round;
	float  fx1, fx2, fy1, fy2, fz1, fz2;
	int    mvf;
	float  mvr;
	float  m_val;
	float  m_x1y1z1, m_x2y1z1, m_x1y2z1, m_x2y2z1, m_x1y1z2, m_x2y1z2, m_x1y2z2, m_x2y2z2;

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Calculate the number of unusable threads in each block.
	int threadsLostPerBlock = threadsPerBlock - (threadsPerBlock / 3) * 3;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * (threadsPerBlock - threadsLostPerBlock)) + threadIdxInBlock;

	// Set the "write flag" to 0.
	sdata[2*(threadIdxInBlock/3)+2] = 0.0;

	// If the voxel lies outside the region, do nothing.
	if(threadIdxInBlock < (threadsPerBlock - threadsLostPerBlock) &&
		threadIdxInGrid < (3 * vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z))
	{	
		// Calculate the linear index of the voxel in the region. Will be in the range
		// (0, vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z - 1).
		lridx = threadIdxInGrid / 3;

		// Calculate the coordinate offset (x = 0, y = 1, z = 2).
		offset = threadIdxInGrid - (lridx * 3);		

		// Only one out of every three threads needs to calculate the following information.
		// All other threads get the data from shared memory.
		if(offset ==  0) {

		// Calculate the x, y and z offsets of the voxel within the tile.
		q.x = lridx % vox_per_rgn.x;
		q.y = ((lridx - q.x) / vox_per_rgn.x) % vox_per_rgn.y;
		q.z = ((((lridx - q.x) / vox_per_rgn.x) - q.y) / vox_per_rgn.y) % vox_per_rgn.z;

		// Calculate the x, y and z offsets of the voxel within the volume.
		coord_in_volume.x = roi_offset.x + p.x * vox_per_rgn.x + q.x;
		coord_in_volume.y = roi_offset.y + p.y * vox_per_rgn.y + q.y;
		coord_in_volume.z = roi_offset.z + p.z * vox_per_rgn.z + q.z;

		// If the voxel lies outside the image, do nothing.
		if(coord_in_volume.x <= (roi_offset.x + roi_dim.x) || 
			coord_in_volume.y <= (roi_offset.y + roi_dim.y) ||
			coord_in_volume.z <= (roi_offset.z + roi_dim.z)) {

			// Compute the physical coordinates of fixed image voxel.
			fx = img_origin.x + img_spacing.x * coord_in_volume.x;
			fy = img_origin.y + img_spacing.y * coord_in_volume.y;
			fz = img_origin.z + img_spacing.z * coord_in_volume.z;

			// Compute the linear index of fixed image voxel in the volume.
			fv = (coord_in_volume.z * volume_dim.x * volume_dim.y) + (coord_in_volume.y * volume_dim.x) + coord_in_volume.x;
			
			// ----------------------------------------------------------------
			// Calculate the B-Spline deformation vector.
			// ----------------------------------------------------------------

			// Use the offset of the voxel within the region to compute the index into the c_lut.
			pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
			pidx = pidx * 64;

			// Use the offset of the voxel to compute the index into the multiplier LUT or q_lut.
			// qidx = ((q.z * vox_per_rgn.y + q.y) * vox_per_rgn.x) + q.x;
			qidx = lridx * 64;

			// Compute the deformation vector.
			d.x = 0.0;
			d.y = 0.0;
			d.z = 0.0;

			for(int k = 0; k < 64; k++)
			{
				// Calculate the index into the coefficients array.
				cidx = 3 * tex1Dfetch(tex_c_lut, pidx + k); 
				
				// Fetch the values for P, Ni, Nj, and Nk.
				P   = tex1Dfetch(tex_q_lut, qidx + k); 
				N.x = tex1Dfetch(tex_coeff, cidx + 0);  // x-value
				N.y = tex1Dfetch(tex_coeff, cidx + 1);  // y-value
				N.z = tex1Dfetch(tex_coeff, cidx + 2);  // z-value

				// Update the output (v) values.
				d.x += P * N.x;
				d.y += P * N.y;
				d.z += P * N.z;
			}
			
			// ----------------------------------------------------------------
			// Find correspondence in the moving image.
			// ----------------------------------------------------------------

			// Calculate the distance of the voxel from the origin (in mm) along the x, y and z axes.
			distance_from_image_origin.x = img_origin.x + (pix_spacing.x * coord_in_volume.x);
			distance_from_image_origin.y = img_origin.y + (pix_spacing.y * coord_in_volume.y);
			distance_from_image_origin.z = img_origin.z + (pix_spacing.z * coord_in_volume.z);
			
			// Calculate the displacement of the voxel (in mm) in the x, y, and z directions.
			displacement_in_mm.x = distance_from_image_origin.x + d.x;
			displacement_in_mm.y = distance_from_image_origin.y + d.y;
			displacement_in_mm.z = distance_from_image_origin.z + d.z;

			// Calculate the displacement value in terms of voxels.
			displacement_in_vox.x = (displacement_in_mm.x - img_offset.x) / pix_spacing.x;
			displacement_in_vox.y = (displacement_in_mm.y - img_offset.y) / pix_spacing.y;
			displacement_in_vox.z = (displacement_in_mm.z - img_offset.z) / pix_spacing.z;

			// Check if the displaced voxel lies outside the region of interest.
			if ((displacement_in_vox.x < -0.5) || (displacement_in_vox.x > (volume_dim.x - 0.5)) || 
				(displacement_in_vox.y < -0.5) || (displacement_in_vox.y > (volume_dim.y - 0.5)) || 
				(displacement_in_vox.z < -0.5) || (displacement_in_vox.z > (volume_dim.z - 0.5))) {
				
				if(offset == 0) {
					sdata[2*(threadIdxInBlock/3)] = 0.0;
					sdata[2*(threadIdxInBlock/3)+1] = 0.0;
				}
			}
			else {
					
					// ----------------------------------------------------------------
					// Compute interpolation fractions.
					// ----------------------------------------------------------------

					// Clamp and interpolate along the X axis.
					displacement_in_vox_floor.x = floor(displacement_in_vox.x);
					displacement_in_vox_round.x = round(displacement_in_vox.x);
					fx2 = displacement_in_vox.x - displacement_in_vox_floor.x;
					if(displacement_in_vox_floor.x < 0){
						displacement_in_vox_floor.x = 0;
						displacement_in_vox_round.x = 0;
						fx2 = 0.0;
					}
					else if(displacement_in_vox_floor.x >= (volume_dim.x - 1)){
						displacement_in_vox_floor.x = volume_dim.x - 2;
						displacement_in_vox_round.x = volume_dim.x - 1;
						fx2 = 1.0;
					}
					fx1 = 1.0 - fx2;

					// Clamp and interpolate along the Y axis.
					displacement_in_vox_floor.y = floor(displacement_in_vox.y);
					displacement_in_vox_round.y = round(displacement_in_vox.y);
					fy2 = displacement_in_vox.y - displacement_in_vox_floor.y;
					if(displacement_in_vox_floor.y < 0){
						displacement_in_vox_floor.y = 0;
						displacement_in_vox_round.y = 0;
						fy2 = 0.0;
					}
					else if(displacement_in_vox_floor.y >= (volume_dim.y - 1)){
						displacement_in_vox_floor.y = volume_dim.y - 2;
						displacement_in_vox_round.y = volume_dim.y - 1;
						fy2 = 1.0;
					}
					fy1 = 1.0 - fy2;
					
					// Clamp and intepolate along the Z axis.
					displacement_in_vox_floor.z = floor(displacement_in_vox.z);
					displacement_in_vox_round.z = round(displacement_in_vox.z);
					fz2 = displacement_in_vox.z - displacement_in_vox_floor.z;
					if(displacement_in_vox_floor.z < 0){
						displacement_in_vox_floor.z = 0;
						displacement_in_vox_round.z = 0;
						fz2 = 0.0;
					}
					else if(displacement_in_vox_floor.z >= (volume_dim.z - 1)){
						displacement_in_vox_floor.z = volume_dim.z - 2;
						displacement_in_vox_round.z = volume_dim.z - 1;
						fz2 = 1.0;
					}
					fz1 = 1.0 - fz2;
					
					// ----------------------------------------------------------------
					// Compute moving image intensity using linear interpolation.
					// ----------------------------------------------------------------

					mvf = (displacement_in_vox_floor.z * volume_dim.y + displacement_in_vox_floor.y) * volume_dim.x + displacement_in_vox_floor.x;
					m_x1y1z1 = fx1 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf);
					m_x2y1z1 = fx2 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf + 1);
					m_x1y2z1 = fx1 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x);
					m_x2y2z1 = fx2 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x + 1);
					m_x1y1z2 = fx1 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x);
					m_x2y1z2 = fx2 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + 1);
					m_x1y2z2 = fx1 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x);
					m_x2y2z2 = fx2 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x + 1);
					m_val = m_x1y1z1 + m_x2y1z1 + m_x1y2z1 + m_x2y2z1 + m_x1y1z2 + m_x2y1z2 + m_x1y2z2 + m_x2y2z2;

					// ----------------------------------------------------------------
					// Compute intensity difference.
					// ----------------------------------------------------------------

					// diff[threadIdxInGrid] = fixed_image[threadIdxInGrid] - m_val;
					diff = tex1Dfetch(tex_fixed_image, fv) - m_val;
					
					// ----------------------------------------------------------------
					// Accumulate the score.
					// ----------------------------------------------------------------
				
					score[lridx] = tex1Dfetch(tex_score, lridx) + (diff * diff);

					// ----------------------------------------------------------------
					// Compute dc_dv for this offset
					// ----------------------------------------------------------------
					
					// Compute spatial gradient using nearest neighbors.
					mvr = (((displacement_in_vox_round.z * volume_dim.y) + displacement_in_vox_round.y) * volume_dim.x) + displacement_in_vox_round.x;

					// Store this data in shared memory.
					sdata[2*(threadIdxInBlock/3)] = diff;
					sdata[2*(threadIdxInBlock/3)+1] = mvr;
					sdata[2*(threadIdxInBlock/3)+2] = 1.0;
				}				
			}
		}
	}

	// Wait until all the threads in this thread block reach this point.
	__syncthreads();

	// dc_dv[threadIdxInGrid] = diff * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + offset);

	if(sdata[2*(threadIdxInBlock/3)+2] == 1.0)
		dc_dv[threadIdxInGrid] = sdata[2*(threadIdxInBlock/3)] * 
			tex1Dfetch(tex_moving_grad, (3 * (int)sdata[2*(threadIdxInBlock/3)+1]) + offset);
}

__global__ void bspline_cuda_score_e_mse_kernel2_v2 (
	float  *dc_dv,
	float  *grad,
	float  *gpu_q_lut,
	int    num_threads,
	int3   p,
	float3 rdims,
	int    offset,
	int3   vox_per_rgn,
	int    threadsPerControlPoint)
{
	// Shared memory is allocated on a per block basis.  Therefore, only allocate 
	// (sizeof(data) * blocksize) memory when calling the kernel.
	extern __shared__ float sdata[]; 

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// Set the number of threads being used to work on each control point.
	int tpcp = threadsPerControlPoint;

	// If the thread does not correspond to a control point, do nothing.
	if(threadIdxInGrid < num_threads)
	{
		int qidx;
		float result = 0.0;
		float temp0, temp1, temp2, temp3, temp4, temp5, temp6, temp7;

		// Calculate the linear index of the control point.
		int m = threadIdxInGrid / (threadsPerControlPoint * 3);

		// Use the offset of the voxel within the region to compute the index into the c_lut.
		int pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;

		// Calculate the coordinate offset (x = 0, y = 1, z = 2).
		int xyzOffset = (threadIdxInGrid / threadsPerControlPoint) - (m * 3);

		// Determine the thread offset for this control point, in the range [0, threadsPerControlPoint).
		int cpThreadOffset = threadIdxInGrid % threadsPerControlPoint;

		// Calculate index into coefficient texture.
		int cidx = tex1Dfetch(tex_c_lut, 64 * pidx + m) * 3;

		// Calculate the number of voxels in the region.
		int num_vox = vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z;

		// Calculate the offset of this tile in the dc_dv array.
		int tileOffset = 3 * num_vox * offset;

		for(qidx = cpThreadOffset; qidx < num_vox - (8*tpcp); qidx = qidx + (8*tpcp)) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)          + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+(1*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(1*tpcp)) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+(2*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(2*tpcp)) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+(3*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(3*tpcp)) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+(4*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(4*tpcp)) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+(5*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(5*tpcp)) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+(6*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(6*tpcp)) + m);
			temp7 = tex1Dfetch(tex_dc_dv, 3*(qidx+(7*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(7*tpcp)) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6 + temp7;
		}
		
		if(qidx+(7*tpcp) < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)          + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+(1*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(1*tpcp)) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+(2*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(2*tpcp)) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+(3*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(3*tpcp)) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+(4*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(4*tpcp)) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+(5*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(5*tpcp)) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+(6*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(6*tpcp)) + m);
			temp7 = tex1Dfetch(tex_dc_dv, 3*(qidx+(7*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(7*tpcp)) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6 + temp7;
		}
		else if(qidx+(6*tpcp) < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)          + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+(1*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(1*tpcp)) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+(2*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(2*tpcp)) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+(3*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(3*tpcp)) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+(4*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(4*tpcp)) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+(5*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(5*tpcp)) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+(6*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(6*tpcp)) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6;
		}
		else if(qidx+(5*tpcp) < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)          + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+(1*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(1*tpcp)) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+(2*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(2*tpcp)) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+(3*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(3*tpcp)) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+(4*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(4*tpcp)) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+(5*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(5*tpcp)) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5;
		}
		else if(qidx+(4*tpcp) < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)          + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+(1*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(1*tpcp)) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+(2*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(2*tpcp)) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+(3*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(3*tpcp)) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+(4*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(4*tpcp)) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4;
		}
		else if(qidx+(3*tpcp) < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)          + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+(1*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(1*tpcp)) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+(2*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(2*tpcp)) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+(3*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(3*tpcp)) + m);
			result += temp0 + temp1 + temp2 + temp3;
		}
		else if(qidx+(2*tpcp) < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)          + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+(1*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(1*tpcp)) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+(2*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(2*tpcp)) + m);
			result += temp0 + temp1 + temp2;
		}
		else if(qidx+(1*tpcp) < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)          + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+(1*tpcp)) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+(1*tpcp)) + m);
			result += temp0 + temp1;
		}
		else if(qidx < num_vox)
			result += tex1Dfetch(tex_dc_dv, 3*(qidx) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);

		sdata[(tpcp * threadIdxInBlock) + cpThreadOffset] = result;
		
		// Wait for the other threads in the thread block to reach this point.
		__syncthreads();

		if(cpThreadOffset == 0) {
			temp0 = 0.0;

			// Accumulate all the partial results for this control point.
			for(int i = 0; i < tpcp; i++) {
				temp0 += sdata[(tpcp * threadIdxInBlock) + i];
			}

			// Update the gradient stream.
			grad[cidx + xyzOffset] = tex1Dfetch(tex_grad, cidx + xyzOffset) + temp0;
		}			
	}
}

/***********************************************************************
 * bspline_cuda_score_e_mse_kernel2
 * This kernel is the second of two used in the CUDA implementation of 
 * score_e_mse.  It calculates the gradient values for the control knots.
 ***********************************************************************/
__global__ void bspline_cuda_score_e_mse_kernel2 (
	float  *dc_dv,
	float  *grad,
	float  *gpu_q_lut,
	int    num_threads,
	int3   p,
	float3 rdims,
	int    offset,
	int3   vox_per_rgn,
	int    total_vox_per_rgn) // Volume of a tile in voxels)
{
	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// If the thread does not correspond to a control point, do nothing.
	if(threadIdxInGrid < num_threads)
	{
		int m;
		int num_vox;
		int xyzOffset;
		int tileOffset;
		int cidx;
		int qidx;
		float result = 0.0;
		float temp0, temp1, temp2, temp3, temp4, temp5, temp6, temp7;

		int q[3];

		// Use the offset of the voxel within the region to compute the index into the c_lut.
		int pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
		
		// Calculate the linear index of the control point.
		m = threadIdxInGrid / 3;

		// Calculate the coordinate offset (x = 0, y = 1, z = 2).
		xyzOffset = threadIdxInGrid - (m * 3);

		// Calculate index into coefficient texture.
		cidx = tex1Dfetch(tex_c_lut, 64 * pidx + m) * 3;

		// Calculate the number of voxels in the region.
		num_vox = vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z;

		// Calculate the offset of this tile in the dc_dv array.
		tileOffset = 3 * num_vox * offset;

		/* ORIGINAL CODE: Looked at each offset serially.
		// Serial across offsets.
		for(int qidx = 0; qidx < (vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z); qidx++) {
			result += tex1Dfetch(tex_dc_dv, 3*qidx + offset) * tex1Dfetch(tex_q_lut, 64*qidx + m);
		}
		*/

		// NAGA: Unrolling the loop 8 times; 4 seems to work as well as 8
		// FOR_CHRIS: FIX to make sure the unrolling works with an arbitrary loop index
		for(qidx = 0; qidx < num_vox - 8; qidx = qidx + 8) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+5) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+5) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+6) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+6) + m);
			temp7 = tex1Dfetch(tex_dc_dv, 3*(qidx+7) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+7) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6 + temp7;
		}
		
		if(qidx+7 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+5) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+5) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+6) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+6) + m);
			temp7 = tex1Dfetch(tex_dc_dv, 3*(qidx+7) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+7) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6 + temp7;
		}
		else if(qidx+6 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+5) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+5) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+6) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+6) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6;
		}
		else if(qidx+5 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+5) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+5) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5;
		}
		else if(qidx+4 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4;
		}
		else if(qidx+3 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			result += temp0 + temp1 + temp2 + temp3;
		}
		else if(qidx+2 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			result += temp0 + temp1 + temp2;
		}
		else if(qidx+1 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			result += temp0 + temp1;
		}
		else if(qidx < num_vox)
			result += tex1Dfetch(tex_dc_dv, 3*(qidx) + tileOffset + xyzOffset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);

		grad[cidx + xyzOffset] = tex1Dfetch(tex_grad, cidx + xyzOffset) + result;
	}
}

/***********************************************************************
 * bspline_cuda_score_d_mse_kernel2
 * This kernel is the second of two used in the CUDA implementation of 
 * score_d_mse.  It calculates the gradient values for the control knots.
 ***********************************************************************/
__global__ void bspline_cuda_score_d_mse_kernel2 (
	float  *dc_dv,
	float  *grad,
	float  *gpu_q_lut,
	int    num_threads,
	int3   p,
	float3 rdims,
	int3   vox_per_rgn)
{
	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// If the thread does not correspond to a control point, do nothing.
	if(threadIdxInGrid < num_threads)
	{	
		int m;
		int offset;
		int cidx;
		int qidx;
		int num_vox;
		float result = 0.0;
		float temp0, temp1, temp2, temp3, temp4, temp5, temp6, temp7;

		int q[3];

		// Use the offset of the voxel within the region to compute the index into the c_lut.
		int pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
		
		// Calculate the linear index of the control point.
		m = threadIdxInGrid / 3;

		// Calculate the coordinate offset (x = 0, y = 1, z = 2).
		offset = threadIdxInGrid - (m * 3);

		// Calculate index into coefficient texture.
		cidx = tex1Dfetch(tex_c_lut, 64*pidx + m) * 3;

		// Calculate the number of voxels in the region.
		num_vox = vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z;

		/* ORIGINAL CODE: Looked at each offset serially.
		// Serial across offsets.
		for(int qidx = 0; qidx < (vox_per_rgn.x * vox_per_rgn.y * vox_per_rgn.z); qidx++) {
			result += tex1Dfetch(tex_dc_dv, 3*qidx + offset) * tex1Dfetch(tex_q_lut, 64*qidx + m);
		}
		*/

		// NAGA: Unrolling the loop 8 times; 4 seems to work as well as 8
		// FOR_CHRIS: FIX to make sure the unrolling works with an arbitrary loop index
		for(qidx = 0; qidx < num_vox - 8; qidx = qidx + 8) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + offset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+5) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+5) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+6) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+6) + m);
			temp7 = tex1Dfetch(tex_dc_dv, 3*(qidx+7) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+7) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6 + temp7;
		}
		
		if(qidx+7 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + offset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+5) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+5) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+6) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+6) + m);
			temp7 = tex1Dfetch(tex_dc_dv, 3*(qidx+7) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+7) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6 + temp7;
		}
		else if(qidx+6 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + offset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+5) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+5) + m);
			temp6 = tex1Dfetch(tex_dc_dv, 3*(qidx+6) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+6) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5 + temp6;
		}
		else if(qidx+5 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + offset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			temp5 = tex1Dfetch(tex_dc_dv, 3*(qidx+5) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+5) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4 + temp5;
		}
		else if(qidx+4 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + offset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			temp4 = tex1Dfetch(tex_dc_dv, 3*(qidx+4) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+4) + m);
			result += temp0 + temp1 + temp2 + temp3 + temp4;
		}
		else if(qidx+3 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + offset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			temp3 = tex1Dfetch(tex_dc_dv, 3*(qidx+3) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+3) + m);
			result += temp0 + temp1 + temp2 + temp3;
		}
		else if(qidx+2 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + offset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			temp2 = tex1Dfetch(tex_dc_dv, 3*(qidx+2) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+2) + m);
			result += temp0 + temp1 + temp2;
		}
		else if(qidx+1 < num_vox) {
			temp0 = tex1Dfetch(tex_dc_dv, 3*(qidx)   + offset) * tex1Dfetch(tex_q_lut, 64*(qidx)   + m);
			temp1 = tex1Dfetch(tex_dc_dv, 3*(qidx+1) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx+1) + m);
			result += temp0 + temp1;
		}
		else if(qidx < num_vox)
			result += tex1Dfetch(tex_dc_dv, 3*(qidx) + offset) * tex1Dfetch(tex_q_lut, 64*(qidx) + m);

		grad[cidx + offset] = tex1Dfetch(tex_grad, cidx + offset) + result;
	}
}

/* Kernel to compute the displacement values in the X, Y, and Z directions. */
__global__ void bspline_cuda_compute_dxyz_kernel(
	int   *c_lut,
	float *q_lut,
	float *coeff,
	int3 volume_dim,
	int3 vox_per_rgn,
	float3 rdims,
	float *dx,
	float *dy,
	float *dz
	)
{
	int3 vox_coordinate;	// X, Y, Z coordinates for this voxel	
	int3 p;				    // Tile index.
	int3 q;				    // Offset within tile.
	int pidx;				// Index into c_lut.
	int qidx;				// Index into q_lut.
	int cidx;				// Index into the coefficient table.
	int* prow;				// First element in the correct row in c_lut.
	float* qrow;			// First element in the correct row in q_lut.
	float P;				
	float3 N;				// Multiplier values.		
	float3 output;			// Output values.

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// If the voxel lies outside the volume, do nothing.
	if(threadIdxInGrid < (volume_dim.x * volume_dim.y * volume_dim.z))
	{
		// Get the X, Y, Z position of the voxel.
		// vox_coordinate.z = floor(threadIdxInGrid / (volume_dim.x * volume_dim.y));
		// vox_coordinate.y = floor((threadIdxInGrid - vox_coordinate.z * (volume_dim.x * volume_dim.y)) / volume_dim.x);
		vox_coordinate.z = threadIdxInGrid / (volume_dim.x * volume_dim.y);
		vox_coordinate.y = (threadIdxInGrid - (vox_coordinate.z * volume_dim.x * volume_dim.y)) / volume_dim.x;
		vox_coordinate.x = threadIdxInGrid - vox_coordinate.z * volume_dim.x * volume_dim.y - (vox_coordinate.y * volume_dim.x);
			
		// Get the tile location of the voxel.
		p.x = vox_coordinate.x / vox_per_rgn.x;
		p.y = vox_coordinate.y / vox_per_rgn.y;
		p.z = vox_coordinate.z / vox_per_rgn.z;
				
		// Get the offset of the voxel within the tile.
		q.x = vox_coordinate.x - p.x * vox_per_rgn.x;
		q.y = vox_coordinate.y - p.y * vox_per_rgn.y;
		q.z = vox_coordinate.z - p.z * vox_per_rgn.z;
				
		// Use the tile location of the voxel to compute the index into the c_lut.
		pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
		prow = &c_lut[pidx*64];
		pidx = pidx * 64;

		// Use the offset of the voxel to compute the index into the multiplier LUT or q_lut.
		qidx = ((q.z * vox_per_rgn.y + q.y) * vox_per_rgn.x) + q.x;
		// qrow = &q_lut[qidx*64];
		qidx = qidx * 64;

		// Initialize output values.
		output.x = 0.0;
		output.y = 0.0;
		output.z = 0.0;

		for(int k = 0; k < 64; k++)
		{
			// Calculate the index into the coefficients array.
			cidx = 3 * prow[k];
			// cidx = 3 * tex1Dfetch(tex_c_lut, pidx + k); 
			
			// Fetch the values for P, Ni, Nj, and Nk.
			// P = qrow[k];
			P  = tex1Dfetch(tex_q_lut, qidx + k); 
			N.x = tex1Dfetch(tex_coeff, cidx + 0);  // x-value
			N.y = tex1Dfetch(tex_coeff, cidx + 1);  // y-value
			N.z = tex1Dfetch(tex_coeff, cidx + 2);  // z-value

			// Update the output (v) values.
			output.x += P * N.x;
			output.y += P * N.y;
			output.z += P * N.z;
		}

		// Save the calculated values to the output streams.
		dx[threadIdxInGrid] = output.x;
		dy[threadIdxInGrid] = output.y;
		dz[threadIdxInGrid] = output.z;
	}
}

/* Kernel to compute the intensity difference between the voxels in the moving and fixed images. */
__global__ void bspline_cuda_compute_diff_kernel (
	float* fixed_image,
	float* moving_image,
	float* dx,
	float* dy,
	float* dz,
	float* diff,
	int*   valid_voxels,
	int3   volume_dim,		// x, y, z dimensions of the volume in voxels
	float3 img_origin,		// x, y, z coordinates for the image origin
	float3 pix_spacing,		// Dimensions of a single voxel in millimeters
	float3 img_offset)		// Offset corresponding to the region of interest
{	

	int3   vox_coordinate;
	float3 distance_from_image_origin;
	float3 displacement_in_mm; 
	float3 displacement_in_vox;
	int3   displacement_in_vox_floor;
	float  fx1, fx2, fy1, fy2, fz1, fz2;
	int    mvf;
	float  m_val;
	float  m_x1y1z1, m_x2y1z1, m_x1y2z1, m_x2y2z1, m_x1y1z2, m_x2y1z2, m_x1y2z2, m_x2y2z2;

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// Ensure that the thread index corresponds to a voxel in the volume before continuing.
	if(threadIdxInGrid < (volume_dim.x * volume_dim.y * volume_dim.z))
	{ 
		// Get the x, y, z position of the voxel.
		vox_coordinate.z = threadIdxInGrid / (volume_dim.x * volume_dim.y);
		vox_coordinate.y = (threadIdxInGrid - (vox_coordinate.z * volume_dim.x * volume_dim.y)) / volume_dim.x;
		vox_coordinate.x = threadIdxInGrid - vox_coordinate.z * volume_dim.x * volume_dim.y - (vox_coordinate.y * volume_dim.x);

		// Calculate the distance of the voxel from the origin (in mm) along the x, y and z axes.
		distance_from_image_origin.x = img_origin.x + (pix_spacing.x * vox_coordinate.x);
		distance_from_image_origin.y = img_origin.y + (pix_spacing.y * vox_coordinate.y);
		distance_from_image_origin.z = img_origin.z + (pix_spacing.z * vox_coordinate.z);
		
		// Calculate the displacement of the voxel (in mm) in the x, y, and z directions.
		displacement_in_mm.x = distance_from_image_origin.x + tex1Dfetch(tex_dx, threadIdxInGrid); //dx[threadIdxInGrid];
		displacement_in_mm.y = distance_from_image_origin.y + tex1Dfetch(tex_dy, threadIdxInGrid); //dy[threadIdxInGrid];
		displacement_in_mm.z = distance_from_image_origin.z + tex1Dfetch(tex_dz, threadIdxInGrid); //dz[threadIdxInGrid];

		// Calculate the displacement value in terms of voxels.
		displacement_in_vox.x = (displacement_in_mm.x - img_offset.x) / pix_spacing.x;
		displacement_in_vox.y = (displacement_in_mm.y - img_offset.y) / pix_spacing.y;
		displacement_in_vox.z = (displacement_in_mm.z - img_offset.z) / pix_spacing.z;

		// Check if the displaced voxel lies outside the region of interest.
		if ((displacement_in_vox.x < -0.5) || (displacement_in_vox.x > (volume_dim.x - 0.5)) || 
			(displacement_in_vox.y < -0.5) || (displacement_in_vox.y > (volume_dim.y - 0.5)) || 
			(displacement_in_vox.z < -0.5) || (displacement_in_vox.z > (volume_dim.z - 0.5))) {
			diff[threadIdxInGrid] = 0.0;
			valid_voxels[threadIdxInGrid] = 0;
		}
		else {

			// Clamp and interpolate along the X axis.
			displacement_in_vox_floor.x = (int)floor(displacement_in_vox.x);
			fx2 = displacement_in_vox.x - displacement_in_vox_floor.x;
			if(displacement_in_vox_floor.x < 0){
				displacement_in_vox_floor.x = 0;
				fx2 = 0.0;
			}
			else if(displacement_in_vox_floor.x >= (volume_dim.x - 1)){
				displacement_in_vox_floor.x = volume_dim.x - 2;
				fx2 = 1.0;
			}
			fx1 = 1.0 - fx2;
			
			// Clamp and interpolate along the Y axis.
			displacement_in_vox_floor.y = (int)floor(displacement_in_vox.y);
			fy2 = displacement_in_vox.y - displacement_in_vox_floor.y;
			if(displacement_in_vox_floor.y < 0){
				displacement_in_vox_floor.y = 0;
				fy2 = 0.0;
			}
			else if(displacement_in_vox_floor.y >= (volume_dim.y - 1)){
				displacement_in_vox_floor.y = volume_dim.y - 2;
				fy2 = 1.0;
			}
			fy1 = 1.0 - fy2;
			
			// Clamp and intepolate along the Z axis.
			displacement_in_vox_floor.z = (int)floor(displacement_in_vox.z);
			fz2 = displacement_in_vox.z - displacement_in_vox_floor.z;
			if(displacement_in_vox_floor.z < 0){
				displacement_in_vox_floor.z = 0;
				fz2 = 0.0;
			}
			else if(displacement_in_vox_floor.z >= (volume_dim.z - 1)){
				displacement_in_vox_floor.z = volume_dim.z - 2;
				fz2 = 1.0;
			}
			fz1 = 1.0 - fz2;
			
			// Compute moving image intensity using linear interpolation.
			mvf = (displacement_in_vox_floor.z * volume_dim.y + displacement_in_vox_floor.y) * volume_dim.x + displacement_in_vox_floor.x;
			/*
			m_x1y1z1 = fx1 * fy1 * fz1 * moving_image[mvf];
			m_x2y1z1 = fx2 * fy1 * fz1 * moving_image[mvf + 1];
			m_x1y2z1 = fx1 * fy2 * fz1 * moving_image[mvf + volume_dim.x];
			m_x2y2z1 = fx2 * fy2 * fz1 * moving_image[mvf + volume_dim.x + 1];
			m_x1y1z2 = fx1 * fy1 * fz2 * moving_image[mvf + volume_dim.y * volume_dim.x];
			m_x2y1z2 = fx2 * fy1 * fz2 * moving_image[mvf + volume_dim.y * volume_dim.x + 1];
			m_x1y2z2 = fx1 * fy2 * fz2 * moving_image[mvf + volume_dim.y * volume_dim.x + volume_dim.x];
			m_x2y2z2 = fx2 * fy2 * fz2 * moving_image[mvf + volume_dim.y * volume_dim.x + volume_dim.x + 1];
			*/
			m_x1y1z1 = fx1 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf);
			m_x2y1z1 = fx2 * fy1 * fz1 * tex1Dfetch(tex_moving_image, mvf + 1);
			m_x1y2z1 = fx1 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x);
			m_x2y2z1 = fx2 * fy2 * fz1 * tex1Dfetch(tex_moving_image, mvf + volume_dim.x + 1);
			m_x1y1z2 = fx1 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x);
			m_x2y1z2 = fx2 * fy1 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + 1);
			m_x1y2z2 = fx1 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x);
			m_x2y2z2 = fx2 * fy2 * fz2 * tex1Dfetch(tex_moving_image, mvf + volume_dim.y * volume_dim.x + volume_dim.x + 1);
			m_val = m_x1y1z1 + m_x2y1z1 + m_x1y2z1 + m_x2y2z1 + m_x1y1z2 + m_x2y1z2 + m_x1y2z2 + m_x2y2z2;

			// Compute intensity difference.
			// diff[threadIdxInGrid] = fixed_image[threadIdxInGrid] - m_val;
			diff[threadIdxInGrid] = tex1Dfetch(tex_fixed_image, threadIdxInGrid) - m_val;
			valid_voxels[threadIdxInGrid] = 1;
		}
	}
}

/* Kernel to compute the dc_dv values used to update the control-knot coefficients. */
__global__ void bspline_cuda_compute_dc_dv_kernel (
	float  *fixed_image,
	float  *moving_image,
	float  *moving_grad,
	int    *c_lut,
	float  *q_lut,
	float  *dx,
	float  *dy,
	float  *dz,
	float  *diff,
	float  *dc_dv_x,
	float  *dc_dv_y,
	float  *dc_dv_z,
	// float  *grad,
	int    *valid_voxels,
	int3   volume_dim,		// x, y, z dimensions of the volume in voxels
	int3   vox_per_rgn,
	float3 rdims,
	float3 img_origin,		// x, y, z coordinates for the image origin
	float3 pix_spacing,		// Dimensions of a single voxel in millimeters
	float3 img_offset)		// Offset corresponding to the region of interest
{	
	int3   vox_coordinate;
	float3 distance_from_image_origin;
	float3 displacement_in_mm; 
	float3 displacement_in_vox;
	float3 displacement_in_vox_floor;
	float3 displacement_in_vox_round;
	int3   p;		// Tile index.
	int3   q;		// Offset within tile.
	int    pidx;	// Index into c_lut.
	int    qidx;	// Index into q_lut.
	int    cidx;	// Index into the coefficient table.
	int*   prow;	// First element in the correct row in c_lut.
	float* qrow;	// First element in the correct row in q_lut.
	float  mvr;

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// Ensure that the thread index corresponds to a voxel in the volume before continuing.
	if(threadIdxInGrid < (volume_dim.x * volume_dim.y * volume_dim.z))
	{ 
		// Get the x, y, z position of the voxel.
		vox_coordinate.z = threadIdxInGrid / (volume_dim.x * volume_dim.y);
		vox_coordinate.y = (threadIdxInGrid - (vox_coordinate.z * volume_dim.x * volume_dim.y)) / volume_dim.x;
		vox_coordinate.x = threadIdxInGrid - vox_coordinate.z * volume_dim.x * volume_dim.y - (vox_coordinate.y * volume_dim.x);

		// Calculate the distance of the voxel from the origin (in mm) along the x, y and z axes.
		distance_from_image_origin.x = img_origin.x + (pix_spacing.x * vox_coordinate.x);
		distance_from_image_origin.y = img_origin.y + (pix_spacing.y * vox_coordinate.y);
		distance_from_image_origin.z = img_origin.z + (pix_spacing.z * vox_coordinate.z);
		
		// Calculate the displacement of the voxel (in mm) in the x, y, and z directions.
		displacement_in_mm.x = distance_from_image_origin.x + tex1Dfetch(tex_dx, threadIdxInGrid); //dx[threadIdxInGrid];
		displacement_in_mm.y = distance_from_image_origin.y + tex1Dfetch(tex_dy, threadIdxInGrid); //dy[threadIdxInGrid];
		displacement_in_mm.z = distance_from_image_origin.z + tex1Dfetch(tex_dz, threadIdxInGrid); //dz[threadIdxInGrid];

		// Calculate the displacement value in terms of voxels.
		displacement_in_vox.x = (displacement_in_mm.x - img_offset.x) / pix_spacing.x;
		displacement_in_vox.y = (displacement_in_mm.y - img_offset.y) / pix_spacing.y;
		displacement_in_vox.z = (displacement_in_mm.z - img_offset.z) / pix_spacing.z;

		/*
		// Get the tile location of the voxel.
		p.x = vox_coordinate.x / vox_per_rgn.x;
		p.y = vox_coordinate.y / vox_per_rgn.y;
		p.z = vox_coordinate.z / vox_per_rgn.z;
				
		// Get the offset of the voxel within the tile.
		q.x = vox_coordinate.x - p.x * vox_per_rgn.x;
		q.y = vox_coordinate.y - p.y * vox_per_rgn.y;
		q.z = vox_coordinate.z - p.z * vox_per_rgn.z;
				
		// Use the tile location of the voxel to compute the index into the c_lut.
		pidx = ((p.z * rdims.y + p.y) * rdims.x) + p.x;
		prow = &c_lut[pidx*64];

		// Use the offset if the voxel to compute the index into the multiplier LUT or q_lut.
		qidx = ((q.z * vox_per_rgn.y + q.y) * vox_per_rgn.x) + q.x;
		qrow = &q_lut[qidx*64];
		*/

		// Check if the displaced voxel lies outside the region of interest.
		if ((displacement_in_vox.x < -0.5) || (displacement_in_vox.x > (volume_dim.x - 0.5)) || 
			(displacement_in_vox.y < -0.5) || (displacement_in_vox.y > (volume_dim.y - 0.5)) || 
			(displacement_in_vox.z < -0.5) || (displacement_in_vox.z > (volume_dim.z - 0.5))) {
			dc_dv_x[threadIdxInGrid] = 0.0;
			dc_dv_y[threadIdxInGrid] = 0.0;
			dc_dv_z[threadIdxInGrid] = 0.0;
		}
		else {

			// Clamp and interpolate along the X axis.
			displacement_in_vox_floor.x = floor(displacement_in_vox.x);
			displacement_in_vox_round.x = round(displacement_in_vox.x);
			if(displacement_in_vox_floor.x < 0){
				displacement_in_vox_floor.x = 0;
				displacement_in_vox_round.x = 0;
			}
			else if(displacement_in_vox_floor.x >= (volume_dim.x - 1)){
				displacement_in_vox_floor.x = volume_dim.x - 2;
				displacement_in_vox_round.x = volume_dim.x - 1;
			}
			
			// Clamp and interpolate along the Y axis.
			displacement_in_vox_floor.y = floor(displacement_in_vox.y);
			displacement_in_vox_round.y = round(displacement_in_vox.y);
			if(displacement_in_vox_floor.y < 0){
				displacement_in_vox_floor.y = 0;
				displacement_in_vox_round.y = 0;
			}
			else if(displacement_in_vox_floor.y >= (volume_dim.y - 1)){
				displacement_in_vox_floor.y = volume_dim.y - 2;
				displacement_in_vox_round.y = volume_dim.y - 1;
			}
			
			// Clamp and intepolate along the Z axis.
			displacement_in_vox_floor.z = floor(displacement_in_vox.z);
			displacement_in_vox_round.z = round(displacement_in_vox.z);
			if(displacement_in_vox_floor.z < 0){
				displacement_in_vox_floor.z = 0;
				displacement_in_vox_round.z = 0;
			}
			else if(displacement_in_vox_floor.z >= (volume_dim.z - 1)){
				displacement_in_vox_floor.z = volume_dim.z - 2;
				displacement_in_vox_round.z = volume_dim.z - 1;
			}

			// Compute spatial gradient using nearest neighbors.
			mvr = (((displacement_in_vox_round.z * volume_dim.y) + displacement_in_vox_round.y) * volume_dim.x) + displacement_in_vox_round.x;
			dc_dv_x[threadIdxInGrid] = diff[threadIdxInGrid] * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 0); //moving_grad[(3 * (int)mvr) + 0];
			dc_dv_y[threadIdxInGrid] = diff[threadIdxInGrid] * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 1); //moving_grad[(3 * (int)mvr) + 1];
			dc_dv_z[threadIdxInGrid] = diff[threadIdxInGrid] * tex1Dfetch(tex_moving_grad, (3 * (int)mvr) + 2); //moving_grad[(3 * (int)mvr) + 2];
			
			/*
		    for (int i = 0; i < 64; i++) {
				cidx = 3 * prow[i];
				grad[cidx+0] += dc_dv.x * qrow[i];
				grad[cidx+1] += dc_dv.y * qrow[i];
				grad[cidx+2] += dc_dv.z * qrow[i];
			}
			*/
		}
	}
}

// This reduce function will work for any size array.  It is the same as 
// bspline_cuda_compute_score_kernel, with the exception that it assumes all values are valid.
__global__ void sum_reduction_kernel(
  float *idata, 
  float *odata, 
  int   num_elems)
{
  // Shared memory is allocated on a per block basis.  Therefore, only allocate 
  // (sizeof(data) * blocksize) memory when calling the kernel.
  extern __shared__ float sdata[];
  
  // Calculate the index of the thread block in the grid.
  int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
  
  // Calculate the total number of threads in each thread block.
  int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);
  
  // Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
  int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;
  
  // Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
  int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

  // Load data into shared memory.
  if(threadIdxInGrid >= num_elems)
    sdata[threadIdxInBlock] = 0.0;
  else 
    sdata[threadIdxInBlock] = idata[threadIdxInGrid];

  // Wait for all threads in the block to reach this point.
  __syncthreads();
  
  // Perform the reduction in shared memory.  Stride over the block and reduce
  // parts until it is down to a single value (stored in sdata[0]).
  for(unsigned int s = threadsPerBlock / 2; s > 0; s >>= 1) {
    if (threadIdxInBlock < s) {
      sdata[threadIdxInBlock] += sdata[threadIdxInBlock + s];
    }

    // Wait for all threads to complete this stride.
    __syncthreads();
  }
  
  // Write the result for this block back to global memory.
  if(threadIdxInBlock == 0) {
	  odata[threadIdxInGrid] = sdata[0];
  }
}

// This reduce function will work for any size array, and also checks a flag for each voxel
// to determine whether or not it is valid before adding it to the final sum.
__global__ void bspline_cuda_compute_score_kernel(
  float *idata, 
  float *odata, 
  int   *valid_voxels, 
  int   num_elems)
{
  // Shared memory is allocated on a per block basis.  Therefore, only allocate 
  // (sizeof(data) * blocksize) memory when calling the kernel.
  extern __shared__ float sdata[]; 
  
  // Calculate the index of the thread block in the grid.
  int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;
  
  // Calculate the total number of threads in each thread block.
  int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);
  
  // Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
  int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;
  
  // Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
  int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

  // Load data into shared memory.
  if(threadIdxInGrid >= num_elems || valid_voxels[threadIdxInGrid] == 0)
    sdata[threadIdxInBlock] = 0.0;
  else 
    sdata[threadIdxInBlock] = idata[threadIdxInGrid] * idata[threadIdxInGrid];

  // Wait for all threads in the block to reach this point.
  __syncthreads();
  
  // Perform the reduction in shared memory.  Stride over the block and reduce
  // parts until it is down to a single value (stored in sdata[0]).
  for(unsigned int s = threadsPerBlock / 2; s > 0; s >>= 1) {
    if (threadIdxInBlock < s) {
      sdata[threadIdxInBlock] += sdata[threadIdxInBlock + s];
    }

    // Wait for all threads to complete this stride.
    __syncthreads();
  }
  
  // Write the result for this block back to global memory.
  if(threadIdxInBlock == 0) {
	  odata[threadIdxInGrid] = sdata[0];
  }
}

__global__ void sum_reduction_last_step_kernel(
	float *idata,
	float *odata,
	int   num_elems)
{
	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	if(threadIdxInGrid == 0) {
	
		float sum = 0.0;
		
		for(int i = 0; i < num_elems; i += threadsPerBlock) {
			sum += idata[i];
		}

		odata[0] = sum;
	}
}

__global__ void bspline_cuda_update_grad_kernel(
	float *grad,
	int num_vox,
	int num_elems)
{
	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	if(threadIdxInGrid < num_elems) {
		//grad[threadIdxInGrid] = 2 * grad[threadIdxInGrid] / num_vox;
		grad[threadIdxInGrid] = 2 * tex1Dfetch(tex_grad, threadIdxInGrid) / num_vox;
	}
}

__global__ void bspline_cuda_compute_grad_mean_kernel(
	float *idata,
	float *odata,
	int num_elems)
{
	// Shared memory is allocated on a per block basis.  Therefore, only allocate 
	// (sizeof(data) * blocksize) memory when calling the kernel.
	extern __shared__ float sdata[]; 

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// Load data into shared memory.
	if(threadIdxInGrid >= num_elems)
		sdata[threadIdxInBlock] = 0.0;
	else 
		sdata[threadIdxInBlock] = idata[threadIdxInGrid];

	// Wait for all threads in the block to reach this point.
	__syncthreads();

	// Perform the reduction in shared memory.  Stride over the block and reduce
	// parts until it is down to a single value (stored in sdata[0]).
	for(unsigned int s = threadsPerBlock / 2; s > 0; s >>= 1) {
		if (threadIdxInBlock < s) {
			sdata[threadIdxInBlock] += sdata[threadIdxInBlock + s];
		}

		// Wait for all threads to complete this stride.
		__syncthreads();
	}

	// Write the result for this block back to global memory.
	if(threadIdxInBlock == 0) {
		odata[threadIdxInGrid] = sdata[0];
	}
}

__global__ void bspline_cuda_compute_grad_norm_kernel(
	float *idata,
	float *odata,
	int num_elems)
{
	// Shared memory is allocated on a per block basis.  Therefore, only allocate 
	// (sizeof(data) * blocksize) memory when calling the kernel.
	extern __shared__ float sdata[]; 

	// Calculate the index of the thread block in the grid.
	int blockIdxInGrid  = (gridDim.x * blockIdx.y) + blockIdx.x;

	// Calculate the total number of threads in each thread block.
	int threadsPerBlock  = (blockDim.x * blockDim.y * blockDim.z);

	// Next, calculate the index of the thread in its thread block, in the range 0 to threadsPerBlock.
	int threadIdxInBlock = (blockDim.x * blockDim.y * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

	// Finally, calculate the index of the thread in the grid, based on the location of the block in the grid.
	int threadIdxInGrid = (blockIdxInGrid * threadsPerBlock) + threadIdxInBlock;

	// Load data into shared memory.
	if(threadIdxInGrid >= num_elems)
		sdata[threadIdxInBlock] = 0.0;
	else 
		sdata[threadIdxInBlock] = fabs(idata[threadIdxInGrid]);

	// Wait for all threads in the block to reach this point.
	__syncthreads();

	// Perform the reduction in shared memory.  Stride over the block and reduce
	// parts until it is down to a single value (stored in sdata[0]).
	for(unsigned int s = threadsPerBlock / 2; s > 0; s >>= 1) {
		if (threadIdxInBlock < s) {
			sdata[threadIdxInBlock] += sdata[threadIdxInBlock + s];
		}

		// Wait for all threads to complete this stride.
		__syncthreads();
	}

	// Write the result for this block back to global memory.
	if(threadIdxInBlock == 0) {
		odata[threadIdxInGrid] = sdata[0];
	}
}