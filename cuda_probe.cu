#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cutil.h>

extern "C" int
cuda_probe (void)
{
    int devicecount;
    printf ("Testing for CUDA...\n");
    hipGetDeviceCount (&devicecount);

    if (devicecount == 0)
    {
	printf("Suitable CUDA environment not detected!\n");
	return 0;
    }

    // It is possible at this point that devicecount = 1 and still be
    // without an actual CUDA device.  CUDA 2.0 and 2.1 exhibit this
    // behavior.  Apparently 2.x will detect an emulator device and
    // throw a 1 by reference when you call hipGetDeviceCount().
    // You are apparently able to distinguish between an actual
    // CUDA device and the emulator by checking the major and minor
    // revision numbers on the compute capability.  Emulated devices
    // are supposed to return 9999 for both major and minor revision
    // numbers.  Some, however, report that while this is the behavior
    // for CUDA 2.0, CUDA 2.1 returns different nonsensical numbers
    // when the detected device is emulated.  Therefore, the best
    // solution (until the behavior is standardised across releases)
    // is to specifically check for compute capabilities we KNOW are
    // working with Plastimatch.
    //

    // Get CUDA device properties.
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);

    if (props.major == 1)
    {
	/* GCS: Cuda 2.2 with Tesla returns compatibility 1.3 */
	printf ("Detected CUDA!  Compute Capability %d.%d\n",
		props.major, props.minor);
	return 1;
    }

    // Unless proven otherwise, we assume no CUDA.
    printf ("No cuda information found\n");
    return 0;
}
