#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cutil.h>

extern "C" int
cuda_probe (void)
{
    int devicecount;
    printf ("Testing for CUDA...\n");
    hipGetDeviceCount (&devicecount);

    if (devicecount == 0)
    {
	printf("Suitable CUDA environment not detected!\n");
	return 0;
    }

    // It is possible at this point that devicecount = 1 and still be
    // without an actual CUDA device.  CUDA 2.0 and 2.1 exhibit this
    // behavior.  Apparently 2.x will detect an emulator device and
    // throw a 1 by reference when you call hipGetDeviceCount().
    // You are apparently able to distinguish between an actual
    // CUDA device and the emulator by checking the major and minor
    // revision numbers on the compute capability.  Emulated devices
    // are supposed to return 9999 for both major and minor revision
    // numbers.  Some, however, report that while this is the behavior
    // for CUDA 2.0, CUDA 2.1 returns different nonsensical numbers
    // when the detected device is emulated.  Therefore, the best
    // solution (until the behavior is standardised across releases)
    // is to specifically check for compute capabilities we KNOW are
    // working with Plastimatch.
    //

    // Get CUDA device properties.
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);

    if(props.major == 1)
    {
	if(props.minor == 0)
	    printf ("Detected CUDA!  Compute Capability 1.0");
	else if(props.minor == 1)
	    printf ("Detected CUDA!  Compute Capability 1.1");
	else if(props.minor == 2)
	    printf ("Detected CUDA!  Compute Capability 1.2");
	else
	{
	    printf ("Suitable CUDA environment not detected!\n");
	    return 0;
	}
	return 1;
    }

    // Unless proven otherwise, we assume no CUDA.
    return 0;
}
