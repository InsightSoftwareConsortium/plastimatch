#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError (const char *msg);

// Part 3 of 5: implement the kernel
__global__ void 
myFirstKernel(int *d_a)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_a[idx] = idx;  
}

__global__ void 
reduce(float *idata, float *odata) 
{
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = idata[i];

    __syncthreads();

    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
	if(tid < s) {
	    sdata[tid] += sdata[tid + s];
	}
	__syncthreads();
    }

    if(tid == 0) {
	odata[blockIdx.x] = sdata[0];
    }
}

int 
cuda_test_1 (int argc, char** argv)
{
    // pointer for host memory
    int *h_a;

    // pointer for device memory
    int *d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc((void **) &d_a, memSize);

    checkCUDAError("hipMalloc");

    // Part 2 of 5: launch kernel
    dim3 dimGrid(numBlocks, 1, 1);
    dim3 dimBlock(numThreadsPerBlock, 1, 1);
    myFirstKernel<<<dimGrid, dimBlock>>>(d_a);

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy( h_a, d_a, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct
    for (int i = 0; i < numBlocks; i++)	{
	for (int j = 0; j < numThreadsPerBlock; j++) {
	    assert (h_a[i * numThreadsPerBlock + j] == i * numThreadsPerBlock + j);
	}
    }

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void 
checkCUDAError (const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)  {
        fprintf (stderr, "Cuda error: %s: %s.\n", 
		 msg, hipGetErrorString (err));
        exit(-1);
    }                         
}

int
cuda_mem_test (int argc, char** argv)
{
    void *test[4];
    int alloc_size;

    for (alloc_size = 1024; alloc_size <= 1024*1024*1024; alloc_size *= 2) {
	printf ("Alloc = %d\n", alloc_size);
	hipMalloc ((void**) &test[0], alloc_size);
	checkCUDAError ("hipMalloc");
	hipMalloc ((void**) &test[1], alloc_size);
	checkCUDAError ("hipMalloc");
	hipMalloc ((void**) &test[2], alloc_size);
	checkCUDAError ("hipMalloc");
	hipMalloc ((void**) &test[3], alloc_size);
	checkCUDAError ("hipMalloc");
	hipFree (test[0]);
	checkCUDAError ("hipFree");
	hipFree (test[1]);
	checkCUDAError ("hipFree");
	hipFree (test[2]);
	checkCUDAError ("hipFree");
	hipFree (test[3]);
	checkCUDAError ("hipFree");
    }
    return 0;
}

int 
main (int argc, char** argv)
{
    //cuda_test_1 (argc, argv);
    return cuda_mem_test (argc, argv);
}
