/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
#include "plm_config.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_util.h"

void cuda_utils_check_error (const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf (stderr, "CUDA ERROR: %s (%s).\n", 
	    msg, hipGetErrorString(err));
        exit (EXIT_FAILURE);
    }                         
}
