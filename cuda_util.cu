/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
#include "plm_config.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_util.h"

void
cuda_utils_check_error (const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf (stderr, "CUDA ERROR: %s (%s).\n", 
	    msg, hipGetErrorString(err));
        exit (EXIT_FAILURE);
    }                         
}


int
cuda_utils_return_error (const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
//        printf ("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
        return 1;
    }                         
    return 0;
}
