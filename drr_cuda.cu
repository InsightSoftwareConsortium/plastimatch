#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------
   See COPYRIGHT.TXT and LICENSE.TXT for copyright and license information
   ----------------------------------------------------------------------- */
#include "plm_config.h"

/****************************************************\
* Uncomment the line below to enable verbose output. *
* Enabling this should not nerf performance.         *
\****************************************************/
#define VERBOSE 1

/**********************************************************\
* Uncomment the line below to enable detailed performance  *
* reporting.  This measurement alters the system, however, *
* resulting in significantly slower kernel execution.      *
\**********************************************************/
#define TIME_KERNEL
#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

/*****************
*  C   #includes *
*****************/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "cuda_utils.h"
#include "drr_cuda.h"
#include "drr_cuda_p.h"
#include "drr_opts.h"
#include "file_util.h"
#include "math_util.h"
#include "plm_cuda_math.h"
#include "proj_image.h"
#include "ray_trace_exact.h"
#include "volume.h"
#include "timer.h"

/* Textures */
texture<float, 1, hipReadModeElementType> tex_img;
texture<float, 1, hipReadModeElementType> tex_matrix;
texture<float, 3, hipReadModeElementType> tex_3Dvol;

#define DRR_LEN_TOLERANCE 1e-6

/* From volume_limit.c */
__device__ int
volume_limit_clip_segment (
    float3 lower_limit,         /* INPUT:  The bounding box to clip to */
    float3 upper_limit,         /* INPUT:  The bounding box to clip to */
    float3 *ip1,                /* OUTPUT: Intersection point 1 */
    float3 *ip2,                /* OUTPUT: Intersection point 2 */
    float3 p1,                  /* INPUT:  Line segment point 1 */
    float3 p2                   /* INPUT:  Line segment point 2 */
)
{
    float3 ray, inv_ray;
    float alpha_in, alpha_out;
    float3 alpha_low, alpha_high;
    int3 ploc;
    int3 is_parallel;

    ray = p2 - p1;
    inv_ray = 1.0f / ray;

    /* Find intersection configuration of ray base */
    /* -1 is POINTLOC_LEFT, 0 is POINTLOC_INSIDE, 1 is POINTLOC_RIGHT */
    ploc = make_int3 (-1, -1, -1);
    if (p1.x > upper_limit.x) {
	ploc.x = 1;
    } else if (p1.x > lower_limit.x) {
	ploc.x = 0;
    }
    if (p1.y > upper_limit.y) {
	ploc.y = 1;
    } else if (p1.y > lower_limit.y) {
	ploc.y = 0;
    }
    if (p1.z > upper_limit.z) {
	ploc.z = 1;
    } else if (p1.z > lower_limit.z) {
	ploc.z = 0;
    }

    /* Check if ray parallel to grid */
    is_parallel = fabsf3(ray) < DRR_LEN_TOLERANCE;

    /* Compute alphas for general configuration */
    alpha_low = (lower_limit - p1) * inv_ray;
    alpha_high = (upper_limit - p1) * inv_ray;

    /* Check case where ray is parallel to grid.  If any dimension is 
       parallel to grid, then p1 must be inside slap, otherwise there 
       is no intersection of segment and cube. */
    if (is_parallel.x) {
	if (!ploc.x) return 0;
	alpha_low.x = - FLT_MAX;
	alpha_high.x = + FLT_MAX;
    }
    if (is_parallel.y) {
	if (!ploc.y) return 0;
	alpha_low.y = - FLT_MAX;
	alpha_high.y = + FLT_MAX;
    }
    if (is_parallel.z) {
	if (!ploc.z) return 0;
	alpha_low.z = - FLT_MAX;
	alpha_high.z = + FLT_MAX;
    }

    /* Sort alpha */
    sortf3 (&alpha_low, &alpha_high);

    /* Check if alpha values overlap in all three dimensions.
       alpha_in is the minimum alpha, where the ray enters the volume.
       alpha_out is where it exits the volume. */
    alpha_in = fmaxf(alpha_low.x, fmaxf (alpha_low.y, alpha_low.z));
    alpha_out = fminf(alpha_high.x, fminf (alpha_high.y, alpha_high.z));

    /* If exit is before entrance, the segment does not intersect the volume */
    if (alpha_out - alpha_in < DRR_LEN_TOLERANCE) {
	return 0;
    }

    /* Compute the volume intersection points */
    *ip1 = p1 + alpha_in * ray;
    *ip2 = p2 + alpha_out * ray;

    return 1;
}

/* From volume_limit.c */
__device__ 
float
ray_trace_uniform (
    float3 ip1,                /* INPUT: Intersection point 1 */
    float3 ip2                 /* INPUT: Intersection point 2 */
)
{
    float3 ray = normalize (ip2 - ip1);
    float step_length = 0.1;
    float3 p;
    int step;

#define MAX_STEPS 100

    //ray = normalize (ray);
    for (step = 0; step < MAX_STEPS; step++) {
	p = ip1 + step * step_length * ray;
	
    }
    return 2.5;
}

/* Main DRR function */
__global__ void
kernel_drr (
    float * dev_img, 
    int2 img_dim, 
    float2 ic, 
    float3 nrm, 
    float sad, 
    float scale, 
    float3 p1, 
    float3 ul_room, 
    float3 incr_r, 
    float3 incr_c, 
    int4 image_window, 
    float3 lower_limit, 
    float3 upper_limit, 
    float3 vol_offset, 
    int3 vol_dim, 
    float3 vol_pix_spacing
)
{
    extern __shared__ float sdata[];

    float3 p2;
    float3 ip1, ip2;
    int r, c;
    int idx;
    float outval;
    float3 r_tgt, tmp;
    int cols;

    /* Get coordinates of this image pixel */
    c = blockIdx.x * blockDim.x + threadIdx.x;
    r = blockIdx.y * blockDim.y + threadIdx.y;

    /* Compute ray */
    r_tgt = ul_room;
    tmp = r * incr_r;
    r_tgt = r_tgt + tmp;
    tmp = c * incr_c;
    p2 = r_tgt + tmp;

    /* Compute output location */
    cols = image_window.w - image_window.z + 1;
    idx = (c - image_window.z) + (r - image_window.x) * cols;

    /* Clip ray to volume */
    if (volume_limit_clip_segment (lower_limit, upper_limit, 
	    &ip1, &ip2, p1, p2) == 0)
    {
	outval = 0;
    } else {
	outval = ray_trace_uniform (ip1, ip2);
    }

    /* Write output pixel value */
    if (r < img_dim.x && c < img_dim.y) {
	dev_img[r*img_dim.x + c] = outval;
    }
}

void*
drr_cuda_state_create (
    Proj_image *proj,
    Volume *vol,
    Drr_options *options
)
{
    Drr_cuda_state *state;
    Drr_kernel_args *kargs;

    state = (Drr_cuda_state *) malloc (sizeof(Drr_cuda_state));
    memset (state, 0, sizeof(Drr_cuda_state));

    state->kargs = kargs = (Drr_kernel_args*) malloc (sizeof(Drr_kernel_args));
    hipMalloc ((void**) &state->dev_matrix, 12 * sizeof(float));
    hipMalloc ((void**) &state->dev_kargs, sizeof(Drr_kernel_args));

    printf ("printf state = %p\n", state);
    printf ("printf state->kargs = %p\n", state->kargs);

    kargs->vol_offset = make_float3 (vol->offset);
    kargs->vol_dim = make_int3 (vol->dim);
    kargs->vol_spacing = make_float3 (vol->pix_spacing);

#if defined (commentout)
    /* The below code is Junan's.  Presumably this way can be better 
       for using hardware linear interpolation, but for now I'm going 
       to follow Tony's method. */
    // prepare texture
    hipChannelFormatDesc ca_descriptor;
    hipExtent ca_extent;
    hipArray *dev_3Dvol=0;

    ca_descriptor = hipCreateChannelDesc<float>();
    ca_extent.width  = vol->dim[0];
    ca_extent.height = vol->dim[1];
    ca_extent.depth  = vol->dim[2];
    hipMalloc3DArray (&dev_3Dvol, &ca_descriptor, ca_extent);
    hipBindTextureToArray (tex_3Dvol, dev_3Dvol, ca_descriptor);

    hipMemcpy3DParms cpy_params = {0};
    cpy_params.extent   = ca_extent;
    cpy_params.kind     = hipMemcpyHostToDevice;
    cpy_params.dstArray = dev_3Dvol;

    //http://sites.google.com/site/cudaiap2009/cookbook-1#TOC-CUDA-3D-Texture-Example-Gerald-Dall
    // The pitched pointer is really tricky to get right. We give the
    // pitch of a row, then the number of elements in a row, then the
    // height, and we omit the 3rd dimension.
    cpy_params.srcPtr = make_hipPitchedPtr ((void*)vol->img, 
	ca_extent.width * sizeof(float), ca_extent.width , ca_extent.height);

    hipMemcpy3D (&cpy_params);
#endif

    hipMalloc ((void**) &state->dev_img, 
	options->image_resolution[0] * options->image_resolution[1] 
	* sizeof(float));
    cuda_utils_check_error ("Unable to allocate dev_img\n");
    printf ("dev_img = %p (%d %d)\n", state->dev_img, 
	options->image_resolution[0], options->image_resolution[1]);

    return (void*) state;
}

void
drr_cuda_state_destroy (
    void *void_state
)
{
    Drr_cuda_state *state = (Drr_cuda_state*) void_state;
    
    hipFree (state->dev_img);
    hipFree (state->dev_kargs);
    hipFree (state->dev_matrix);
    free (state->kargs);
}

void
drr_cuda_ray_trace_image (
    Proj_image *proj, 
    Volume *vol, 
    Volume_limit *vol_limit, 
    double p1[3], 
    double ul_room[3], 
    double incr_r[3], 
    double incr_c[3], 
    void *dev_state, 
    Drr_options *options
)
{
    Timer timer, total_timer;
    double time_kernel = 0;
    int i;

    // CUDA device pointers
    Drr_cuda_state *state = (Drr_cuda_state*) dev_state;
    Drr_kernel_args *kargs = state->kargs;

    // Start the timer
    plm_timer_start (&total_timer);
    plm_timer_start (&timer);

    // Load dynamic kernel arguments (different for each projection)
    kargs->img_dim.x = proj->dim[0];
    kargs->img_dim.y = proj->dim[1];
    kargs->ic.x = proj->pmat->ic[0];
    kargs->ic.y = proj->pmat->ic[1];
    kargs->nrm.x = proj->pmat->nrm[0];
    kargs->nrm.y = proj->pmat->nrm[1];
    kargs->nrm.z = proj->pmat->nrm[2];
    kargs->sad = proj->pmat->sad;
    kargs->sid = proj->pmat->sid;
    for (i = 0; i < 12; i++) {
	kargs->matrix[i] = (float) proj->pmat->matrix[i];
    }
    kargs->p1.x = p1[0];
    kargs->p1.y = p1[1];
    kargs->p1.z = p1[2];
    kargs->ul_room.x = ul_room[0];
    kargs->ul_room.y = ul_room[1];
    kargs->ul_room.z = ul_room[2];
    kargs->incr_r = make_float3 (incr_r);
    kargs->incr_c = make_float3 (incr_c);
    kargs->image_window = make_int4 (options->image_window);
    kargs->lower_limit = make_float3 (vol_limit->lower_limit);
    kargs->upper_limit = make_float3 (vol_limit->upper_limit);

    printf ("ul_room = %f %f %f\n", ul_room[0], ul_room[1], ul_room[2]);

    hipMemcpy (state->dev_matrix, kargs->matrix, sizeof(kargs->matrix), 
	hipMemcpyHostToDevice);
    hipBindTexture (0, tex_matrix, state->dev_matrix, sizeof(kargs->matrix));

    // Thread Block Dimensions
    int tBlock_x = 16;
    int tBlock_y = 16;

    // Each element in the image gets 1 thread
    int blocksInX = (vol->dim[0]+tBlock_x-1)/tBlock_x;
    int blocksInY = (vol->dim[1]+tBlock_y-1)/tBlock_y;
    dim3 dimGrid  = dim3(blocksInX, blocksInY);
    dim3 dimBlock = dim3(tBlock_x, tBlock_y);

    // Note: proj->img AND proj->matrix are passed via texture memory

    //int smemSize = vol->dim[0]  * sizeof(float);

    printf ("Preprocessing time: %f secs\n", plm_timer_report (&timer));
    plm_timer_start (&timer);

    // Invoke ze kernel  \(^_^)/
    kernel_drr<<< dimGrid, dimBlock>>> (
	state->dev_img, 
	kargs->img_dim,
	kargs->ic,
	kargs->nrm,
	kargs->sad,
	kargs->scale,
	kargs->p1, 
	kargs->ul_room, 
	kargs->incr_r, 
	kargs->incr_c, 
	kargs->image_window, 
	kargs->lower_limit,
	kargs->upper_limit,
	kargs->vol_offset,
	kargs->vol_dim,
	kargs->vol_spacing);

    printf ("Kernel time: %f secs\n", plm_timer_report (&timer));
    plm_timer_start (&timer);

    cuda_utils_check_error("Kernel Panic!");

#if defined (TIME_KERNEL)
    // CUDA kernel calls are asynchronous...
    // In order to accurately time the kernel
    // execution time we need to set a thread
    // barrier here after its execution.
    hipDeviceSynchronize();
#endif

    time_kernel += plm_timer_report (&timer);

    // Unbind the image and projection matrix textures
    //hipUnbindTexture (tex_img);
    hipUnbindTexture (tex_matrix);

    // Copy reconstructed volume from device to host
    printf ("dev_img = %p (%d %d)\n", state->dev_img, 
	proj->dim[0], proj->dim[1]);
    hipMemcpy (proj->img, state->dev_img, 
	proj->dim[0] * proj->dim[1] * sizeof(float), 
	hipMemcpyDeviceToHost);
    cuda_utils_check_error("Error: Unable to retrieve data volume.");
}
