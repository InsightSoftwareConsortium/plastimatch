#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------------
   see copyright.txt and license.txt for copyright and license information
   ----------------------------------------------------------------------- */
#include "plmregister_config.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cuda_kernel_util.h"
#include "cuda_texture.h"
#include "cuda_util.h"
#include "demons.h"
#include "demons_cuda.h"
#include "demons_state.h"
#include "gaussian.h"
#include "plm_cuda_math.h"
#include "plm_timer.h"
#include "volume.h"

/* GCS 2023-05-09.  Why is this redefined from cuda_kernel_util.h ? */
#undef block_size
#define block_size 256
#define BLOCK_SIZE 256

/* Texture Memory */
class Demons_cuda_state {
public:
    Cuda_texture fixed;
    Cuda_texture moving;
    Cuda_texture grad_x;
    Cuda_texture grad_y;
    Cuda_texture grad_z;
    Cuda_texture grad_mag;
    Cuda_texture vf_est_x;
    Cuda_texture vf_est_y;
    Cuda_texture vf_est_z;
    Cuda_texture vf_smooth_x;
    Cuda_texture vf_smooth_y;
    Cuda_texture vf_smooth_z;
};

/*
Constant Memory
*/
__constant__ int c_dim[3];
__constant__ int c_moving_dim[3];
__constant__ float c_spacing_div2[3];
__constant__ float c_f2mo[3];
__constant__ float c_f2ms[3];
__constant__ float c_invmps[3];


/*
Constant Memory Functions
*/
void 
setConstantDimension (plm_long *h_dim)
{
    int i_dim[3] = { (int) h_dim[0], (int) h_dim[1], (int) h_dim[2] };
    hipMemcpyToSymbol(HIP_SYMBOL(c_dim), i_dim, sizeof(int3));
    //hipMemcpyToSymbol(HIP_SYMBOL(c_dim), h_dim, sizeof(int3));
}

void 
setConstantMovingDimension (plm_long *h_dim)
{
    int i_dim[3] = { (int) h_dim[0], (int) h_dim[1], (int) h_dim[2] };
    hipMemcpyToSymbol(HIP_SYMBOL(c_moving_dim), i_dim, sizeof(int3));
}

void setConstantPixelSpacing(float *h_spacing_div2)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_spacing_div2), h_spacing_div2, sizeof(float3));
}

void setConstantF2mo(float *h_f2mo)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_f2mo), h_f2mo, sizeof(float3));
}

void setConstantF2ms(float *h_f2ms)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_f2ms), h_f2ms, sizeof(float3));
}

void setConstantInvmps(float *h_invmps)
{
    hipMemcpyToSymbol(HIP_SYMBOL(c_invmps), h_invmps, sizeof(float3));
}

/*
Device Functions
*/
__device__ int volume_index_cuda (int *dims, int i, int j, int k)
{
    return i + (dims[0] * (j + dims[1] * k));
}

/*
Kernels
*/
__global__ void
calculate_gradient_magnitude_image_kernel (
    hipTextureObject_t grad_x,
    hipTextureObject_t grad_y,
    hipTextureObject_t grad_z,
    hipSurfaceObject_t grad_mag)
{
    // calculate surface coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= c_dim[0] || y >= c_dim[1] || z >= c_dim[2])
        return;

    long v = (z * c_dim[1] * c_dim[0]) + (y * c_dim[0]) + x;
    long v3 = v * 3;

    float vox_grad_x = tex3D<float> (grad_x, x, y, z);
    float vox_grad_y = tex3D<float> (grad_y, x, y, z);
    float vox_grad_z = tex3D<float> (grad_z, x, y, z);
    float val = vox_grad_x * vox_grad_x
        + vox_grad_y * vox_grad_y + vox_grad_z * vox_grad_z;

    surf3Dwrite (val, grad_mag, x * 4, y, z);
}

__global__ void 
estimate_displacements_kernel (
    hipSurfaceObject_t vf_est_x,
    hipSurfaceObject_t vf_est_y,
    hipSurfaceObject_t vf_est_z,
    hipTextureObject_t vf_smooth_x,
    hipTextureObject_t vf_smooth_y,
    hipTextureObject_t vf_smooth_z,
    hipTextureObject_t fixed,
    hipTextureObject_t moving,
    hipTextureObject_t grad_x,
    hipTextureObject_t grad_y,
    hipTextureObject_t grad_z,
    hipTextureObject_t grad_mag,
    float *ssd, 
    int *inliers, 
    float homog, 
    float denominator_eps, 
    float accel, 
    int blockY, 
    float invBlockY
)
{
    // calculate surface coordinates
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z*blockDim.z + threadIdx.z;

    if (i >= c_dim[0] || j >= c_dim[1] || k >= c_dim[2])
	return;

    long fv = (k * c_dim[1] * c_dim[0]) + (j * c_dim[0]) + i;
    long f3v = 3 * fv;

    float mi = c_f2mo[0] + i * c_f2ms[0];
    float mj = c_f2mo[1] + j * c_f2ms[1];
    float mk = c_f2mo[2] + k * c_f2ms[2];

    /* Find correspondence with nearest neighbor interpolation 
       and boundary checking */
    int mz = __float2int_rn (mk + c_invmps[2] 
	* tex3D<float> (vf_smooth_z, i, j, k));
    if (mz < 0 || mz >= c_moving_dim[2])
	return;

    int my = __float2int_rn (mj + c_invmps[1] 
	* tex3D<float> (vf_smooth_y, i, j, k));
    if (my < 0 || my >= c_moving_dim[1])
	return;

    int mx = __float2int_rn (mi + c_invmps[0] 
	* tex3D<float> (vf_smooth_x, i, j, k));
    if (mx < 0 || mx >= c_moving_dim[0])
	return;

    int mv = (mz * c_moving_dim[1] + my) * c_moving_dim[0] + mx;
    int m3v = 3 * mv;

    /* Find image difference at this correspondence */
    float diff = tex3D<float> (fixed, i, j, k)
        - tex3D<float> (moving, mx, my, mz);

    /* Compute denominator */
    float denom = tex3D<float> (grad_mag, mx, my, mz) + homog * diff * diff;

    /* Compute SSD for statistics */
    inliers[fv] = 1;
    ssd[fv] = diff * diff;

    /* Threshold the denominator to stabilize estimation */
    if (denom < denominator_eps) 
	return;

    /* Compute new estimate of displacement */
    float mult = accel * diff / denom;
    float data;
    surf3Dread (&data, vf_est_x, mx * 4, my, mz);
    data += mult * tex3D<float>(grad_x, mx, my, mz);
    surf3Dwrite (data, vf_est_x, mx * 4, my, mz);
    surf3Dread (&data, vf_est_y, mx * 4, my, mz);
    data += mult * tex3D<float>(grad_y, mx, my, mz);
    surf3Dwrite (data, vf_est_y, mx * 4, my, mz);
    surf3Dread (&data, vf_est_z, mx * 4, my, mz);
    data += mult * tex3D<float>(grad_z, mx, my, mz);
    surf3Dwrite (data, vf_est_z, mx * 4, my, mz);
}

template <class T> __global__ void
reduction(T *vectorData, int totalElements)
{
    __shared__ T vector[BLOCK_SIZE * 2];

    /* Find position in vector */
    int threadID = threadIdx.x;
    int blockID = blockIdx.x;
    int xInVector = BLOCK_SIZE * blockID * 2 + threadID;

    vector[threadID] = (xInVector < totalElements) ? vectorData[xInVector] : 0;
    vector[threadID + BLOCK_SIZE] = (xInVector + BLOCK_SIZE < totalElements) ? vectorData[xInVector + BLOCK_SIZE] : 0;
    __syncthreads();

    /* Calculate partial sum */
    for (int stride = BLOCK_SIZE; stride > 0; stride >>= 1) {
        if (threadID < stride)
            vector[threadID] += vector[threadID + stride];
        __syncthreads();
    }
    __syncthreads();

    if (threadID == 0)
        vectorData[blockID] = vector[0];
}

__device__ void
vf_conv_x (
    hipSurfaceObject_t vf_out,
    hipTextureObject_t vf_in,
    float *ker,
    int x,
    int y,
    int z,
    int i1,
    int j1,
    int j2
)
{
    float sum = 0.0;
    for (int i = i1, j = j1; j <= j2; i++, j++) {
        float data = tex3D<float> (vf_in, i, y, z);
        sum += ker[j] * data;
    }
    surf3Dwrite (sum, vf_out, x * 4, y, z);
}

__device__ void
vf_conv_y (
    hipSurfaceObject_t vf_out,
    hipTextureObject_t vf_in,
    float *ker,
    int x,
    int y,
    int z,
    int i1,
    int j1,
    int j2
)
{
    float sum = 0.0;
    for (int i = i1, j = j1; j <= j2; i++, j++) {
        float data = tex3D<float> (vf_in, x, i, z);
        sum += ker[j] * data;
    }
    surf3Dwrite (sum, vf_out, x * 4, y, z);
}

__device__ void
vf_conv_z (
    hipSurfaceObject_t vf_out,
    hipTextureObject_t vf_in,
    float *ker,
    int x,
    int y,
    int z,
    int i1,
    int j1,
    int j2
)
{
    float sum = 0.0;
    for (int i = i1, j = j1; j <= j2; i++, j++) {
        float data = tex3D<float> (vf_in, x, y, i);
        sum += ker[j] * data;
    }
    surf3Dwrite (sum, vf_out, x * 4, y, z);
}

__global__ void
vf_convolve_x_kernel (
    hipSurfaceObject_t vf_out_x,
    hipSurfaceObject_t vf_out_y,
    hipSurfaceObject_t vf_out_z,
    hipTextureObject_t vf_in_x,
    hipTextureObject_t vf_in_y,
    hipTextureObject_t vf_in_z,
    float *ker, int half_width, int blockY, float invBlockY)
{
    int i, i1;		/* i is the offset in the vf */
    int j, j1, j2;	/* j is the index of the kernel */
    int d;			/* d is the vector field direction */

    // calculate surface coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= c_dim[0] || y >= c_dim[1] || z >= c_dim[2])
        return;

    long v3 = 3 * ((z * c_dim[1] * c_dim[0]) + (y * c_dim[0]) + x);

    j1 = x - half_width;
    j2 = x + half_width;
    if (j1 < 0) j1 = 0;
    if (j2 >= c_dim[0]) {
        j2 = c_dim[0] - 1;
    }
    i1 = j1 - x;
    j1 = j1 - x + half_width;
    j2 = j2 - x + half_width;

    vf_conv_x (vf_out_x, vf_in_x, ker, x, y, z, i1, j1, j2);
    vf_conv_x (vf_out_y, vf_in_y, ker, x, y, z, i1, j1, j2);
    vf_conv_x (vf_out_z, vf_in_z, ker, x, y, z, i1, j1, j2);
}

__global__ void
vf_convolve_y_kernel (
    hipSurfaceObject_t vf_out_x,
    hipSurfaceObject_t vf_out_y,
    hipSurfaceObject_t vf_out_z,
    hipTextureObject_t vf_in_x,
    hipTextureObject_t vf_in_y,
    hipTextureObject_t vf_in_z,
    float *ker, int half_width, int blockY, float invBlockY)
{
    int i, i1;		/* i is the offset in the vf */
    int j, j1, j2;	/* j is the index of the kernel */
    int d;			/* d is the vector field direction */

    // calculate surface coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= c_dim[0] || y >= c_dim[1] || z >= c_dim[2])
        return;

    long v3 = 3 * ((z * c_dim[1] * c_dim[0]) + (y * c_dim[0]) + x);

    j1 = y - half_width;
    j2 = y + half_width;
    if (j1 < 0) j1 = 0;
    if (j2 >= c_dim[1]) {
        j2 = c_dim[1] - 1;
    }
    i1 = j1 - y;
    j1 = j1 - y + half_width;
    j2 = j2 - y + half_width;

    vf_conv_y (vf_out_x, vf_in_x, ker, x, y, z, i1, j1, j2);
    vf_conv_y (vf_out_y, vf_in_y, ker, x, y, z, i1, j1, j2);
    vf_conv_y (vf_out_z, vf_in_z, ker, x, y, z, i1, j1, j2);
}

__global__ void
vf_convolve_z_kernel (
    hipSurfaceObject_t vf_out_x,
    hipSurfaceObject_t vf_out_y,
    hipSurfaceObject_t vf_out_z,
    hipTextureObject_t vf_in_x,
    hipTextureObject_t vf_in_y,
    hipTextureObject_t vf_in_z,
    float *ker, int half_width, int blockY, float invBlockY)
{
    int i, i1;		/* i is the offset in the vf */
    int j, j1, j2;	/* j is the index of the kernel */
    int d;			/* d is the vector field direction */

    // calculate surface coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= c_dim[0] || y >= c_dim[1] || z >= c_dim[2])
        return;

    long v3 = 3 * ((z * c_dim[1] * c_dim[0]) + (y * c_dim[0]) + x);

    j1 = z - half_width;
    j2 = z + half_width;
    if (j1 < 0) j1 = 0;
    if (j2 >= c_dim[2]) {
        j2 = c_dim[2] - 1;
    }
    i1 = j1 - z;
    j1 = j1 - z + half_width;
    j2 = j2 - z + half_width;

    vf_conv_z (vf_out_x, vf_in_x, ker, x, y, z, i1, j1, j2);
    vf_conv_z (vf_out_y, vf_in_y, ker, x, y, z, i1, j1, j2);
    vf_conv_z (vf_out_z, vf_in_z, ker, x, y, z, i1, j1, j2);
}

__global__ void
volume_calc_grad_kernel (
    hipTextureObject_t moving,
    hipSurfaceObject_t grad_x,
    hipSurfaceObject_t grad_y,
    hipSurfaceObject_t grad_z
)
{
    // calculate surface coordinates
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z*blockDim.z + threadIdx.z;

    if (i >= c_dim[0] || j >= c_dim[1] || k >= c_dim[2])
        return;

    /* p is prev, n is next */
    int i_p = (i == 0) ? 0 : i - 1;
    int i_n = (i == c_dim[0] - 1) ? c_dim[0] - 1 : i + 1;
    int j_p = (j == 0) ? 0 : j - 1;
    int j_n = (j == c_dim[1] - 1) ? c_dim[1] - 1 : j + 1;
    int k_p = (k == 0) ? 0 : k - 1;
    int k_n = (k == c_dim[2] - 1) ? c_dim[2] - 1 : k + 1;

    long v3 = 3 * ((k * c_dim[1] * c_dim[0]) + (j * c_dim[0]) + i);

    long gi = v3;
    long gj = v3 + 1;
    long gk = v3 + 2;

    float val = 0;
    val = c_spacing_div2[0] *
        (tex3D<float>(moving, i_p, j, k) - tex3D<float>(moving, i_n, j, k));
    surf3Dwrite (val, grad_x, i * 4, j, k);

    val = c_spacing_div2[1] * 
        (tex3D<float>(moving, i, j_p, k) - tex3D<float>(moving, i, j_p, k));
    surf3Dwrite (val, grad_y, i * 4, j, k);

    val = c_spacing_div2[2] *
        (tex3D<float>(moving, i, j, k_p) - tex3D<float>(moving, i, j, k_n));
    surf3Dwrite (val, grad_z, i * 4, j, k);
}

void
demons_cuda (
    Demons_state *demons_state,
    Volume* fixed, 
    Volume* moving, 
    Volume* moving_grad, 
    Volume* vf_init, 
    Demons_parms* parms
)
{
    int i;
    int	it;		/* Iterations */
    float f2mo[3];	/* Origin difference (in cm) from fixed to moving */
    float f2ms[3];	/* Slope to convert fixed to moving */
    float invmps[3];	/* 1/pixel spacing of moving image */
    float *kerx, *kery, *kerz;
    int fw[3];
    double diff_run;
    //Volume *vf_est, *vf_smooth;
    int inliers;
    float ssd;

    Plm_timer* timer = new Plm_timer;
    Plm_timer* gpu_timer = new Plm_timer;
    Plm_timer* kernel_timer = new Plm_timer;

    int num_elements, half_num_elements, reductionBlocks;
    size_t vol_size, interleaved_vol_size, inlier_size;
    int *d_inliers;
    float total_runtime, spacing_div2[3];
    float *d_kerx, *d_kery, *d_kerz, *d_swap, *d_ssd;
    dim3 reductionGrid;

    Demons_cuda_state dcstate;
    
    printf ("Hello from demons_cuda()\n");

    // This code uses planar format
    vf_convert_to_planar (demons_state->vf_smooth);
    
    /* Initialize GPU timers */
    double gpu_time = 0;
    double kernel_time = 0;
	
    /* Determine GPU execution environment */
    int threadX = BLOCK_SIZE;
    int threadY = 1;
    int threadZ = 1;
    int blockX = (fixed->dim[0] + threadX - 1) / threadX;
    int blockY = (fixed->dim[1] + threadY - 1) / threadY;
    int blockZ = (fixed->dim[2] + threadZ - 1) / threadZ;
    dim3 block_dim = dim3(threadX, threadY, threadZ);
    dim3 grid_dim = dim3(blockX, blockY, blockZ);

    for (i = 0; i < 3; i++)
	spacing_div2[i] = 0.5 / moving->spacing[i];

    /* Determine size of device memory */
    vol_size = moving->dim[0] * moving->dim[1] * moving->dim[2] * sizeof(float);
    interleaved_vol_size = 3 * fixed->dim[0] * fixed->dim[1] * fixed->dim[2] * sizeof(float);
    inlier_size = moving->dim[0] * moving->dim[1] * moving->dim[2] * sizeof(int);

    /* Allocate device memory */
    gpu_timer->start ();
    hipMalloc((void**)&d_ssd, vol_size);
    hipMalloc((void**)&d_inliers, inlier_size);

    // Allocate device memory and bind to textures
    dcstate.fixed.make_and_bind (fixed->dim, (float*) fixed->img);
    dcstate.moving.make_and_bind (moving->dim, (float*) moving->img);
    dcstate.grad_x.make_and_bind (moving->dim, 0);
    dcstate.grad_y.make_and_bind (moving->dim, 0);
    dcstate.grad_z.make_and_bind (moving->dim, 0);
    dcstate.grad_mag.make_and_bind (moving->dim, 0);
    dcstate.vf_est_x.make_and_bind (moving->dim, 0);
    dcstate.vf_est_y.make_and_bind (moving->dim, 0);
    dcstate.vf_est_z.make_and_bind (moving->dim, 0);
    /* GCS FIX: initialize vf_smooth to initial guess if supplied */
    dcstate.vf_smooth_x.make_and_bind (moving->dim, 0);
    dcstate.vf_smooth_y.make_and_bind (moving->dim, 0);
    dcstate.vf_smooth_z.make_and_bind (moving->dim, 0);
    
    /* Copy/Initialize device memory */
    gpu_time += gpu_timer->report ();

    /* Set device constant memory */
    setConstantDimension(fixed->dim);
    setConstantMovingDimension(moving->dim);
    setConstantPixelSpacing(spacing_div2);

    /* Bind device texture memory */
    //hipBindTexture(0, tex_fixed, d_fixed, vol_size);
    //hipBindTexture(0, tex_moving, d_moving, vol_size);
    gpu_time += gpu_timer->report ();

    /* Check for any errors prekernel execution */
    CUDA_check_error("Error before kernel execution");

    /* Call kernel */
    kernel_timer->start ();
    volume_calc_grad_kernel<<< grid_dim, block_dim >>>(
        dcstate.moving.tex, dcstate.grad_x.surf,
        dcstate.grad_y.surf, dcstate.grad_z.surf);

    hipDeviceSynchronize();
    kernel_time += kernel_timer->report ();

    /* Check for any errors postkernel execution */
    CUDA_check_error("Kernel execution failed");

    /* Call kernel */
    kernel_timer->start ();
    calculate_gradient_magnitude_image_kernel<<< grid_dim, block_dim >>> (
        dcstate.grad_x.tex, dcstate.grad_y.tex, dcstate.grad_z.tex,
        dcstate.grad_mag.surf);
    hipDeviceSynchronize();
    kernel_time += kernel_timer->report ();

    /* Check for any errors postkernel execution */
    CUDA_check_error("Kernel execution failed");

    /* Validate filter widths */
    validate_filter_widths (fw, parms->filter_width);

    /* Create the seperable smoothing kernels for the x, y, and z directions */
    kerx = create_ker (parms->filter_std / fixed->spacing[0], fw[0]/2);
    kery = create_ker (parms->filter_std / fixed->spacing[1], fw[1]/2);
    kerz = create_ker (parms->filter_std / fixed->spacing[2], fw[2]/2);
    kernel_stats (kerx, kery, kerz, fw);

    /* Compute some variables for converting pixel sizes / origins */
    for (i = 0; i < 3; i++) {
	invmps[i] = 1 / moving->spacing[i];
	f2mo[i] = (fixed->origin[i] - moving->origin[i]) / moving->spacing[i];
	f2ms[i] = fixed->spacing[i] / moving->spacing[i];
    }

    /* Allocate device memory */
    gpu_timer->start ();
    printf ("Doing hipMalloc\n");
    hipMalloc ((void**)&d_kerx, fw[0] * sizeof(float));
    hipMalloc ((void**)&d_kery, fw[1] * sizeof(float));
    hipMalloc ((void**)&d_kerz, fw[2] * sizeof(float));

    /* Copy/Initialize device memory */
    printf ("Doing hipMemcpy\n");
    hipMemcpy (d_kerx, kerx, fw[0] * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy (d_kery, kery, fw[1] * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy (d_kerz, kerz, fw[2] * sizeof(float), hipMemcpyHostToDevice);

    /* Set device constant memory */
    setConstantF2ms (f2mo);
    setConstantF2ms (f2ms);
    setConstantInvmps (invmps);

    /* Bind device texture memory */
    printf ("Doing hipBindTexture\n");
    gpu_time += gpu_timer->report ();

    timer->start ();

    /* Main loop through iterations.  At the start of each iteration, 
       the current displacement field will be in vf_smooth. */
    for (it = 0; it < parms->max_its; it++) {
	printf ("Looping...\n");
	inliers = 0; ssd = 0.0;

	/* Check for any errors prekernel execution */
	CUDA_check_error ("Error before kernel execution");

	gpu_timer->start ();
	hipMemset(d_ssd, 0, vol_size);
	hipMemset(d_inliers, 0, inlier_size);
	gpu_time += gpu_timer->report ();

	// Call kernel, new displacement estimates will be stored into vf_est.
	kernel_timer->start ();
	estimate_displacements_kernel<<< grid_dim, block_dim >>> (
            dcstate.vf_est_x.surf,
            dcstate.vf_est_y.surf,
            dcstate.vf_est_z.surf,
            dcstate.vf_smooth_x.tex,
            dcstate.vf_smooth_y.tex,
            dcstate.vf_smooth_z.tex,
            dcstate.fixed.tex,
            dcstate.moving.tex,
            dcstate.grad_x.tex,
            dcstate.grad_y.tex,
            dcstate.grad_z.tex,
            dcstate.grad_mag.tex,
	    d_ssd, 
	    d_inliers, 
	    parms->homog, 
	    parms->denominator_eps, 
	    parms->accel, 
	    blockY, 
	    1.0f / (float)blockY);
	hipDeviceSynchronize ();
	kernel_time += kernel_timer->report ();

	/* Check for any errors postkernel execution */
	CUDA_check_error ("Kernel execution failed");

	// Calculate statistics for display
	num_elements = moving->dim[0] * moving->dim[1] * moving->dim[2];
	while (num_elements > 1) {
	    half_num_elements = num_elements / 2;
	    reductionBlocks = (half_num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;

	    /* Invoke kernels */
	    dim3 reductionGrid(reductionBlocks, 1);
	    kernel_timer->start ();
	    reduction<float><<< reductionGrid, block_dim >>>(d_ssd, num_elements);
	    hipDeviceSynchronize();
	    reduction<int><<< reductionGrid, block_dim >>>(d_inliers, num_elements);
	    hipDeviceSynchronize();
	    kernel_time += kernel_timer->report ();

	    /* Check for any errors postkernel execution */
	    CUDA_check_error("Kernel execution failed");

	    num_elements = reductionBlocks;
	}

	gpu_timer->start ();
	hipMemcpy(&ssd, d_ssd, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&inliers, d_inliers, sizeof(int), hipMemcpyDeviceToHost);
	gpu_time += gpu_timer->report ();

	/* Print statistics */
	printf ("----- SSD = %.01f (%d/%d)\n", ssd/inliers, inliers, fixed->npix);

	/* Check for any errors prekernel execution */
	CUDA_check_error("Error before kernel execution");

	/* Smooth vf_est into vf_smooth.  The volumes are ping-ponged. */
	kernel_timer->start ();
	vf_convolve_x_kernel<<< grid_dim, block_dim >>> (
            dcstate.vf_smooth_x.surf,
            dcstate.vf_smooth_y.surf,
            dcstate.vf_smooth_z.surf,
            dcstate.vf_est_x.tex,
            dcstate.vf_est_y.tex,
            dcstate.vf_est_z.tex,
            d_kerx, fw[0] / 2, blockY, 1.0f / (float)blockY);
	hipDeviceSynchronize();
	kernel_time += kernel_timer->report ();

	/* Check for any errors postkernel execution */
	CUDA_check_error("Kernel execution failed");

	/* Smooth vf_smooth into vf_est.  The volumes are ping-ponged. */
	kernel_timer->start ();
	vf_convolve_y_kernel<<< grid_dim, block_dim >>> (
            dcstate.vf_est_x.surf,
            dcstate.vf_est_y.surf,
            dcstate.vf_est_z.surf,
            dcstate.vf_smooth_x.tex,
            dcstate.vf_smooth_y.tex,
            dcstate.vf_smooth_z.tex,
            d_kery, fw[1] / 2, blockY, 1.0f / (float)blockY);
	hipDeviceSynchronize();
	kernel_time += kernel_timer->report ();

	/* Check for any errors postkernel execution */
	CUDA_check_error("Kernel execution failed");

	/* Smooth vf_est into vf_smooth.  The volumes are ping-ponged. */
	vf_convolve_z_kernel<<< grid_dim, block_dim >>> (
            dcstate.vf_smooth_x.surf,
            dcstate.vf_smooth_y.surf,
            dcstate.vf_smooth_z.surf,
            dcstate.vf_est_x.tex,
            dcstate.vf_est_y.tex,
            dcstate.vf_est_z.tex,
            d_kerz, fw[2] / 2, blockY, 1.0f / (float)blockY);
	hipDeviceSynchronize();
	kernel_time += kernel_timer->report ();

	/* Check for any errors postkernel execution */
	CUDA_check_error("Kernel execution failed");
    }

    /* Copy final output from device to host */
    float **img = (float**) demons_state->vf_smooth->img;
    gpu_timer->start ();
    CUDA_memcpy_from_3d_array (img[0], fixed->dim, &(dcstate.vf_smooth_x.dev));
    CUDA_memcpy_from_3d_array (img[1], fixed->dim, &(dcstate.vf_smooth_y.dev));
    CUDA_memcpy_from_3d_array (img[2], fixed->dim, &(dcstate.vf_smooth_z.dev));
    gpu_time += gpu_timer->report ();

    // Host expects interleaved
    vf_convert_to_interleaved (demons_state->vf_smooth);

    /* Print statistics */
    diff_run = timer->report ();
    printf("Time for %d iterations = %f (%f sec / it)\n", parms->max_its, diff_run, diff_run / parms->max_its);
    total_runtime = gpu_time + kernel_time;
    printf("\nTransfer run time: %f ms\n", gpu_time * 1000);
    printf("Kernel run time: %f ms\n", kernel_time * 1000);
    printf("Total CUDA run time: %f s\n\n", total_runtime);

    // Clean up
    free(kerx);
    free(kery);
    free(kerz);

    delete timer;
    delete kernel_timer;
    delete gpu_timer;

    /* Free device global memory */
    hipFree(d_ssd);
    hipFree(d_inliers);
}
